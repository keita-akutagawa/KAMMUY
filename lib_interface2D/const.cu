#include "const.hpp"


void initializeDeviceConstants()
{
    hipMemcpyToSymbol(HIP_SYMBOL(device_PI), &PI, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(device_interfaceLength), &interfaceLength, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(device_indexOfInterfaceStartInMHD), &indexOfInterfaceStartInMHD, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(device_indexOfInterfaceStartInPIC), &indexOfInterfaceStartInPIC, sizeof(int));

    hipMemcpyToSymbol(HIP_SYMBOL(device_windowSizeForRemoveNoiseByConvolution), &windowSizeForRemoveNoiseByConvolution, sizeof(int));

    hipMemcpyToSymbol(HIP_SYMBOL(device_reloadParticlesTotalNumIon), &reloadParticlesTotalNumIon, sizeof(unsigned long long));
    hipMemcpyToSymbol(HIP_SYMBOL(device_reloadParticlesTotalNumElectron), &reloadParticlesTotalNumElectron, sizeof(unsigned long long));
}


