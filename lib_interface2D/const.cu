#include "const.hpp"


void initializeDeviceConstants()
{
    hipMemcpyToSymbol(HIP_SYMBOL(device_PI), &PI, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(device_interfaceLength), &interfaceLength, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(device_indexOfInterfaceStartInMHD), &indexOfInterfaceStartInMHD, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(device_indexOfInterfaceStartInPIC), &indexOfInterfaceStartInPIC, sizeof(int));

    hipMemcpyToSymbol(HIP_SYMBOL(device_windowSizeForRemoveNoiseByConvolution), &windowSizeForRemoveNoiseByConvolution, sizeof(int));
}


