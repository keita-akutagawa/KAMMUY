#include "hip/hip_runtime.h"
#include "remove_noise.hpp"


using namespace IdealMHD2DConst;
using namespace PIC2DConst;
using namespace Interface2DConst;


InterfaceNoiseRemover2D::InterfaceNoiseRemover2D(
    int indexStartMHD, 
    int indexStartPIC, 
    int length, 
    int windowSizeForConvolution
)
    : indexOfInterfaceStartInMHD(indexStartMHD), 
      indexOfInterfaceStartInPIC(indexStartPIC), 
      interfaceLength(length), 
      indexOfInterfaceEndInMHD(indexStartMHD + length), 
      indexOfInterfaceEndInPIC(indexStartPIC + length), 

      windowSize(windowSizeForConvolution), 

      tmpB(PIC2DConst::nx_PIC * (interfaceLength + windowSize)), 
      tmpE(PIC2DConst::nx_PIC * (interfaceLength + windowSize)), 
      tmpCurrent(PIC2DConst::nx_PIC * (interfaceLength + windowSize)), 
      tmpZerothMoment(PIC2DConst::nx_PIC * (interfaceLength + windowSize)), 
      tmpFirstMoment(PIC2DConst::nx_PIC * (interfaceLength + windowSize))
{
}


__global__ void copyFields_kernel(
    const MagneticField* B, 
    const ElectricField* E, 
    const CurrentField* current, 
    MagneticField* tmpB, 
    ElectricField* tmpE, 
    CurrentField* tmpCurrent, 
    int indexOfInterfaceStartInPIC, 
    int interfaceLength, 
    int windowSize
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < PIC2DConst::device_nx_PIC && j < interfaceLength + windowSize) {
        int ny_PIC = PIC2DConst::device_ny_PIC;
        int indexPIC = indexOfInterfaceStartInPIC + j + i * ny_PIC;
        int indexForCopy = 

        tmpB[indexPIC] = B[indexPIC];
        tmpE[indexPIC] = E[indexPIC];
        tmpCurrent[indexPIC] = current[indexPIC];
    }
}


__global__ void convolveFields_kernel(
    MagneticField* B, 
    ElectricField* E, 
    CurrentField* current, 
    int indexOfInterfaceStartInMHD, 
    int indexOfInterfaceStartInPIC, 
    int interfaceLength
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (0 < i && i < PIC2DConst::device_nx_PIC - 1 && 0 < j && j < interfaceLength - 1) {
        int ny_PIC = PIC2DConst::device_ny_PIC;
        int indexPIC = indexOfInterfaceStartInPIC + j + i * ny_PIC;
        int 


    }
}

void InterfaceNoiseRemover2D::convolveFields(
    thrust::device_vector<MagneticField>& B, 
    thrust::device_vector<ElectricField>& E, 
    thrust::device_vector<CurrentField>& current
)
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((IdealMHD2DConst::nx_MHD + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (interfaceLength + windowSize + threadsPerBlock.y - 1) / threadsPerBlock.y);

    copyFields_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(B.data()), 
        thrust::raw_pointer_cast(E.data()), 
        thrust::raw_pointer_cast(current.data()), 
        thrust::raw_pointer_cast(tmpB.data()), 
        thrust::raw_pointer_cast(tmpE.data()), 
        thrust::raw_pointer_cast(tmpCurrent.data()), 
        indexOfInterfaceStartInPIC, 
        interfaceLength, 
        windowSize
    );

    hipDeviceSynchronize();
    

    convolveFields_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(B.data()), 
        thrust::raw_pointer_cast(E.data()), 
        thrust::raw_pointer_cast(current.data()), 
        indexOfInterfaceStartInMHD, 
        indexOfInterfaceStartInPIC, 
        interfaceLength
    );

    hipDeviceSynchronize();
}

