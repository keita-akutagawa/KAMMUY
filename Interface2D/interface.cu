#include "hip/hip_runtime.h"
#include "interface.hpp"
#include <cmath>
#include <hiprand/hiprand_kernel.h>
#include <random>
#include <algorithm>
#include <thrust/fill.h>
#include <thrust/partition.h>


using namespace IdealMHD2DConst;
using namespace PIC2DConst;
using namespace Interface2DConst;


__global__ void initializeReloadParticlesSource_kernel(
    Particle* reloadParticlesSourceSpecies, 
    unsigned long long reloadParticlesNumSpecies, 
    int seed
)
{
    unsigned long long i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < reloadParticlesNumSpecies) {
        hiprandState stateX; 
        hiprandState stateY;
        hiprandState stateVx; 
        hiprandState stateVy; 
        hiprandState stateVz;  
        hiprand_init(seed, i, 0, &stateX);
        hiprand_init(seed + 1, i, 0, &stateY);
        hiprand_init(seed + 2, i, 0, &stateVx);
        hiprand_init(seed + 3, i, 0, &stateVy);
        hiprand_init(seed + 4, i, 0, &stateVz);

        reloadParticlesSourceSpecies[i].x = hiprand_uniform(&stateX);
        reloadParticlesSourceSpecies[i].y = hiprand_uniform(&stateY);
        reloadParticlesSourceSpecies[i].vx = hiprand_normal(&stateVx);
        reloadParticlesSourceSpecies[i].vy = hiprand_normal(&stateVy);
        reloadParticlesSourceSpecies[i].vz = hiprand_normal(&stateVz);
    }
}

Interface2D::Interface2D(
    int indexStartMHD, 
    int indexStartPIC, 
    int length
)
    :  indexOfInterfaceStartInMHD(indexStartMHD), 
       indexOfInterfaceStartInPIC(indexStartPIC), 
       interfaceLength(length), 
       indexOfInterfaceEndInMHD(indexStartMHD + length), 
       indexOfInterfaceEndInPIC(indexStartPIC + length), 

       interlockingFunctionY(interfaceLength, 0.0f), 
       interlockingFunctionYHalf(interfaceLength - 1, 0.0f),

       host_interlockingFunctionY(interfaceLength, 0.0f), 
       host_interlockingFunctionYHalf(interfaceLength - 1, 0.0f),

       zerothMomentIon(PIC2DConst::nx_PIC * PIC2DConst::ny_PIC), 
       zerothMomentElectron(PIC2DConst::nx_PIC * PIC2DConst::ny_PIC), 
       firstMomentIon(PIC2DConst::nx_PIC * PIC2DConst::ny_PIC), 
       firstMomentElectron(PIC2DConst::nx_PIC * PIC2DConst::ny_PIC),

       restartParticlesIndexIon(0), 
       restartParticlesIndexElectron(0), 

       reloadParticlesDataIon(PIC2DConst::nx_PIC * interfaceLength), 
       reloadParticlesDataElectron(PIC2DConst::nx_PIC * interfaceLength), 
       reloadParticlesSourceIon(Interface2DConst::reloadParticlesTotalNumIon), 
       reloadParticlesSourceElectron(Interface2DConst::reloadParticlesTotalNumElectron), 

       reloadParticlesIndexIon(PIC2DConst::nx_PIC * interfaceLength, 0), 
       reloadParticlesIndexElectron(PIC2DConst::nx_PIC * interfaceLength, 0), 
       host_reloadParticlesIndexIon(PIC2DConst::nx_PIC * interfaceLength, 0), 
       host_reloadParticlesIndexElectron(PIC2DConst::nx_PIC * interfaceLength, 0), 

       B_timeAve(PIC2DConst::nx_PIC * PIC2DConst::ny_PIC), 
       zerothMomentIon_timeAve(PIC2DConst::nx_PIC * PIC2DConst::ny_PIC), 
       zerothMomentElectron_timeAve(PIC2DConst::nx_PIC * PIC2DConst::ny_PIC), 
       firstMomentIon_timeAve(PIC2DConst::nx_PIC * PIC2DConst::ny_PIC), 
       firstMomentElectron_timeAve(PIC2DConst::nx_PIC * PIC2DConst::ny_PIC), 

       USub(IdealMHD2DConst::nx_MHD * IdealMHD2DConst::ny_MHD), 
       UHalf(IdealMHD2DConst::nx_MHD * IdealMHD2DConst::ny_MHD)  
{

    for(int i = 0; i < interfaceLength; i++) {
        host_interlockingFunctionY[i] = min(
            0.5f * (1.0f + cos(Interface2DConst::PI * (i - 0.0f) / (interfaceLength - 0.0f))), 
            1e-20f
        );
    }
    for(int i = 0; i < interfaceLength - 1; i++) {
        host_interlockingFunctionYHalf[i] = min(
            0.5f * (1.0f + cos(Interface2DConst::PI * (i + 0.5f - 0.0f) / (interfaceLength - 0.0f))), 
            1e-20f
        );
    }

    interlockingFunctionY = host_interlockingFunctionY;
    interlockingFunctionYHalf = host_interlockingFunctionYHalf;


    dim3 threadsPerBlockForIon(256);
    dim3 blocksPerGridForIon((Interface2DConst::reloadParticlesTotalNumIon + threadsPerBlockForIon.x - 1) / threadsPerBlockForIon.x);

    initializeReloadParticlesSource_kernel<<<blocksPerGridForIon, threadsPerBlockForIon>>>(
        thrust::raw_pointer_cast(reloadParticlesSourceIon.data()),
        Interface2DConst::reloadParticlesTotalNumIon, 
        10000
    );

    hipDeviceSynchronize();

    dim3 threadsPerBlockForElectron(256);
    dim3 blocksPerGridForElectron((Interface2DConst::reloadParticlesTotalNumElectron + threadsPerBlockForElectron.x - 1) / threadsPerBlockForElectron.x);

    initializeReloadParticlesSource_kernel<<<blocksPerGridForElectron, threadsPerBlockForElectron>>>(
        thrust::raw_pointer_cast(reloadParticlesSourceElectron.data()),
        Interface2DConst::reloadParticlesTotalNumElectron, 
        20000
    );

    hipDeviceSynchronize();
}


__global__ void sendMHDtoPIC_magneticField_yDirection_kernel(
    const float* interlockingFunctionY, 
    const float* interlockingFunctionYHalf, 
    const ConservationParameter* U, 
    MagneticField* B, 
    int indexOfInterfaceStartInMHD, 
    int indexOfInterfaceStartInPIC, 
    int interfaceLength
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (0 < i && i < PIC2DConst::device_nx_PIC && 0 < j && j < interfaceLength - 1) {
        float bXPIC, bYPIC, bZPIC;
        float bXMHD, bYMHD, bZMHD;
        float bXInterface, bYInterface, bZInterface;

        int indexPIC = indexOfInterfaceStartInPIC +  j + i * PIC2DConst::device_ny_PIC;
        int indexMHD = indexOfInterfaceStartInMHD +  j + i * IdealMHD2DConst::device_ny_MHD;

        //PICのグリッドにMHDを合わせる
        bXPIC = B[indexPIC].bX;
        bYPIC = B[indexPIC].bY;
        bZPIC = B[indexPIC].bZ;
        bXMHD = 0.25f * (U[indexMHD].bX + U[indexMHD - IdealMHD2DConst::device_ny_MHD].bX + U[indexMHD + 1].bX + U[indexMHD + 1 - IdealMHD2DConst::device_ny_MHD].bX);
        bYMHD = 0.25f * (U[indexMHD].bY + U[indexMHD + IdealMHD2DConst::device_ny_MHD].bY + U[indexMHD - 1].bY + U[indexMHD - 1 + IdealMHD2DConst::device_ny_MHD].bY);
        bZMHD = 0.25f * (U[indexMHD].bZ + U[indexMHD + IdealMHD2DConst::device_ny_MHD].bZ + U[indexMHD + 1].bZ + U[indexMHD + 1 + IdealMHD2DConst::device_ny_MHD].bZ);

        bXInterface = interlockingFunctionYHalf[j] * bXMHD + (1.0f - interlockingFunctionYHalf[j]) * bXPIC;
        bYInterface = interlockingFunctionY[j]     * bYMHD + (1.0f - interlockingFunctionY[j])     * bYPIC;
        bZInterface = interlockingFunctionYHalf[j] * bZMHD + (1.0f - interlockingFunctionYHalf[j]) * bZPIC;
        
        B[indexPIC].bX = bXInterface;
        B[indexPIC].bY = bYInterface;
        B[indexPIC].bZ = bZInterface;
    }
}

void Interface2D::sendMHDtoPIC_magneticField_yDirection(
    const thrust::device_vector<ConservationParameter>& U, 
    thrust::device_vector<MagneticField>& B
)
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((PIC2DConst::nx_PIC + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (interfaceLength + threadsPerBlock.y - 1) / threadsPerBlock.y);

    sendMHDtoPIC_magneticField_yDirection_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(interlockingFunctionY.data()),
        thrust::raw_pointer_cast(interlockingFunctionYHalf.data()),
        thrust::raw_pointer_cast(U.data()), 
        thrust::raw_pointer_cast(B.data()), 
        indexOfInterfaceStartInMHD, 
        indexOfInterfaceStartInPIC, 
        interfaceLength
    );

    hipDeviceSynchronize();
}


__global__ void sendMHDtoPIC_electricField_yDirection_kernel(
    const float* interlockingFunctionY, 
    const float* interlockingFunctionYHalf, 
    const ConservationParameter* U, 
    ElectricField* E, 
    int indexOfInterfaceStartInMHD, 
    int indexOfInterfaceStartInPIC, 
    int interfaceLength
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (0 < i && i < PIC2DConst::device_nx_PIC && 0 < j &&  j < interfaceLength - 1) {
        float eXPIC, eYPIC, eZPIC;
        float eXMHD, eYMHD, eZMHD;
        float eXPlusX1MHD;
        float eYPlusY1MHD;
        float rho, u, v, w;
        float bXMHD, bYMHD, bZMHD;
        float eXInterface, eYInterface, eZInterface;

        int indexPIC = indexOfInterfaceStartInPIC + j + i * PIC2DConst::device_ny_PIC;
        int indexMHD = indexOfInterfaceStartInMHD + j + i * IdealMHD2DConst::device_ny_MHD;

        //PICのグリッドにMHDを合わせる
        eXPIC = E[indexPIC].eX;
        eYPIC = E[indexPIC].eY;
        eZPIC = E[indexPIC].eZ;

        rho = U[indexMHD].rho;
        u = U[indexMHD].rhoU / rho;
        v = U[indexMHD].rhoV / rho;
        w = U[indexMHD].rhoW / rho; 
        bXMHD = 0.5f * (U[indexMHD].bX + U[indexMHD - IdealMHD2DConst::device_ny_MHD].bX);
        bYMHD = 0.5f * (U[indexMHD].bY + U[indexMHD - 1].bY);
        bZMHD = U[indexMHD].bZ;
        eXMHD = -(v * bZMHD - w * bYMHD);
        eYMHD = -(w * bXMHD - u * bZMHD);
        eZMHD = -(u * bYMHD - v * bXMHD);

        rho = U[indexMHD + IdealMHD2DConst::device_ny_MHD].rho;
        u = U[indexMHD + IdealMHD2DConst::device_ny_MHD].rhoU / rho;
        v = U[indexMHD + IdealMHD2DConst::device_ny_MHD].rhoV / rho;
        w = U[indexMHD + IdealMHD2DConst::device_ny_MHD].rhoW / rho; 
        bXMHD = 0.5f * (U[indexMHD + IdealMHD2DConst::device_ny_MHD].bX + U[indexMHD].bX);
        bYMHD = 0.5f * (U[indexMHD + IdealMHD2DConst::device_ny_MHD].bY + U[indexMHD - 1 + IdealMHD2DConst::device_ny_MHD].bY);
        bZMHD = U[indexMHD + IdealMHD2DConst::device_ny_MHD].bZ;
        eXPlusX1MHD = -(v * bZMHD - w * bYMHD);

        rho = U[indexMHD + 1].rho;
        u = U[indexMHD + 1].rhoU / rho;
        v = U[indexMHD + 1].rhoV / rho;
        w = U[indexMHD + 1].rhoW / rho; 
        bXMHD = 0.5f * (U[indexMHD + 1].bX + U[indexMHD + 1 - IdealMHD2DConst::device_ny_MHD].bX);
        bYMHD = 0.5f * (U[indexMHD + 1].bY + U[indexMHD].bY);
        bZMHD = U[indexMHD + 1].bZ;
        eYPlusY1MHD = -(w * bXMHD - u * bZMHD);


        eXInterface = interlockingFunctionY[j]     * 0.5f * (eXMHD + eXPlusX1MHD) + (1.0f - interlockingFunctionY[j])     * eXPIC;
        eYInterface = interlockingFunctionYHalf[j] * 0.5f * (eYMHD + eYPlusY1MHD) + (1.0f - interlockingFunctionYHalf[j]) * eYPIC;
        eZInterface = interlockingFunctionY[j]     * eZMHD                        + (1.0f - interlockingFunctionY[j])     * eZPIC;
         
        E[indexPIC].eX = eXInterface;
        E[indexPIC].eY = eYInterface;
        E[indexPIC].eZ = eZInterface;
    }
}

void Interface2D::sendMHDtoPIC_electricField_yDirection(
    const thrust::device_vector<ConservationParameter>& U, 
    thrust::device_vector<ElectricField>& E
)
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((PIC2DConst::nx_PIC + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (interfaceLength + threadsPerBlock.y - 1) / threadsPerBlock.y);

    sendMHDtoPIC_electricField_yDirection_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(interlockingFunctionY.data()), 
        thrust::raw_pointer_cast(interlockingFunctionYHalf.data()), 
        thrust::raw_pointer_cast(U.data()), 
        thrust::raw_pointer_cast(E.data()), 
        indexOfInterfaceStartInMHD, 
        indexOfInterfaceStartInPIC, 
        interfaceLength
    );

    hipDeviceSynchronize();
}


__global__ void sendMHDtoPIC_currentField_yDirection_kernel(
    const float* interlockingFunctionY, 
    const float* interlockingFunctionYHalf, 
    const ConservationParameter* U, 
    CurrentField* current, 
    int indexOfInterfaceStartInMHD, 
    int indexOfInterfaceStartInPIC, 
    int interfaceLength
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (0 < i && i < PIC2DConst::device_nx_PIC && 0 < j && j < interfaceLength - 1) {
        float jXPIC, jYPIC, jZPIC;
        float jXMHD, jYMHD, jZMHD;
        float jXPlusX1MHD; 
        float jYPlusY1MHD; 
        float jXInterface, jYInterface, jZInterface;
        int nx = IdealMHD2DConst::device_nx_MHD;
        float dx = IdealMHD2DConst::device_dx_MHD, dy = IdealMHD2DConst::device_dy_MHD;

        int indexPIC = indexOfInterfaceStartInPIC + j + i * PIC2DConst::device_ny_PIC;
        int indexMHD = indexOfInterfaceStartInMHD + j + i * IdealMHD2DConst::device_ny_MHD;

        //PICのグリッドにMHDを合わせる
        jXPIC = current[indexPIC].jX;
        jYPIC = current[indexPIC].jY;
        jZPIC = current[indexPIC].jZ;
        jXMHD = (U[indexMHD + 1].bZ - U[indexMHD - 1].bZ) / (2.0f * dy);
        jYMHD = -(U[indexMHD + nx].bZ - U[indexMHD - nx].bZ) / (2.0f * dx);
        jZMHD = 0.25f * ((U[indexMHD + nx].bY - U[indexMHD].bY) / dx - (U[indexMHD + 1].bX - U[indexMHD].bX) / dy 
                       + (U[indexMHD].bY - U[indexMHD - nx].bY) / dx - (U[indexMHD + 1 - nx].bX - U[indexMHD - nx].bX) / dy
                       + (U[indexMHD - 1 + nx].bY - U[indexMHD - 1].bY) / dx - (U[indexMHD].bX - U[indexMHD - 1].bX) / dy
                       + (U[indexMHD - 1].bY - U[indexMHD - 1 - nx].bY) / dx - (U[indexMHD - nx].bX - U[indexMHD - 1 - nx].bX) / dy);

        jXPlusX1MHD = (U[indexMHD + 2].bZ - U[indexMHD].bZ) / (2.0f * dy);
        jYPlusY1MHD = -(U[indexMHD + 2 * nx].bZ - U[indexMHD].bZ) / (2.0f * dx);

        jXInterface = interlockingFunctionY[j]     * 0.5f * (jXMHD + jXPlusX1MHD) + (1.0f - interlockingFunctionY[j])     * jXPIC;
        jYInterface = interlockingFunctionYHalf[j] * 0.5f * (jYMHD + jYPlusY1MHD) + (1.0f - interlockingFunctionYHalf[j]) * jYPIC;
        jZInterface = interlockingFunctionY[j]     * jZMHD                        + (1.0f - interlockingFunctionY[j])     * jZPIC;
        
        current[indexPIC].jX = jXInterface;
        current[indexPIC].jY = jYInterface;
        current[indexPIC].jZ = jZInterface;
    }
}

void Interface2D::sendMHDtoPIC_currentField_yDirection(
    const thrust::device_vector<ConservationParameter>& U, 
    thrust::device_vector<CurrentField>& current
)
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((PIC2DConst::nx_PIC + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (interfaceLength + threadsPerBlock.y - 1) / threadsPerBlock.y);

    sendMHDtoPIC_currentField_yDirection_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(interlockingFunctionY.data()), 
        thrust::raw_pointer_cast(interlockingFunctionYHalf.data()), 
        thrust::raw_pointer_cast(U.data()), 
        thrust::raw_pointer_cast(current.data()), 
        indexOfInterfaceStartInMHD, 
        indexOfInterfaceStartInPIC, 
        interfaceLength
    );

    hipDeviceSynchronize();
}


__device__ void cudaAssert(bool condition, int value1, float value2) {
    if (!condition) {
        printf("%d : %f \n", value1, value2);
    }
}


__global__ void sendMHDtoPIC_particle_yDirection_kernel(
    const float* interlockingFunctionY, 
    const ZerothMoment* zerothMomentIon, 
    const ZerothMoment* zerothMomentElectron, 
    const FirstMoment* firstMomentIon, 
    const FirstMoment* firstMomentElectron, 
    const ConservationParameter* U, 
    ReloadParticlesData* reloadParticlesDataIon, 
    ReloadParticlesData* reloadParticlesDataElectron, 
    unsigned long long* reloadParticlesIndexIon, 
    unsigned long long* reloadParticlesIndexElectron, 
    int indexOfInterfaceStartInMHD, 
    int indexOfInterfaceStartInPIC, 
    int interfaceLength
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (0 < i && i < PIC2DConst::device_nx_PIC && 0 < j && j < interfaceLength - 1) {
        int indexPIC = indexOfInterfaceStartInPIC + j + i * PIC2DConst::device_ny_PIC;
        int indexMHD = indexOfInterfaceStartInMHD + j + i * IdealMHD2DConst::device_ny_MHD;
        float rhoMHD, uMHD, vMHD, wMHD, bXMHD, bYMHD, bZMHD, eMHD, pMHD;
        float jXMHD, jYMHD, jZMHD, niMHD, neMHD, tiMHD, teMHD;
        float rhoPIC, uPIC, vPIC, wPIC;
        float jXPIC, jYPIC, jZPIC, niPIC, nePIC, vThiPIC, vThePIC;
        int ny = IdealMHD2DConst::device_ny_MHD;
        float dx = IdealMHD2DConst::device_dx_MHD, dy = IdealMHD2DConst::device_dy_MHD;

        //PICのグリッドにMHDを合わせる
        rhoMHD = U[indexMHD].rho;
        uMHD = U[indexMHD].rhoU / rhoMHD;
        vMHD = U[indexMHD].rhoV / rhoMHD;
        wMHD = U[indexMHD].rhoW / rhoMHD;
        bXMHD = 0.5f * (U[indexMHD].bX + U[indexMHD - ny].bX);
        bYMHD = 0.5f * (U[indexMHD].bY + U[indexMHD - 1].bY);
        bZMHD = U[indexMHD].bZ;
        eMHD = U[indexMHD].e;
        pMHD = (IdealMHD2DConst::device_gamma_MHD - 1.0f)
             * (eMHD - 0.5f * rhoMHD * (uMHD * uMHD + vMHD * vMHD + wMHD * wMHD)
             - 0.5f * (bXMHD * bXMHD + bYMHD * bYMHD + bZMHD * bZMHD));
        jXMHD = (U[indexMHD + 1].bZ - U[indexMHD - 1].bZ) / (2.0f * dy);
        jYMHD = -(U[indexMHD + ny].bZ - U[indexMHD - ny].bZ) / (2.0f * dx);
        jZMHD = 0.25f * ((U[indexMHD + ny].bY - U[indexMHD].bY) / dx - (U[indexMHD + 1].bX - U[indexMHD].bX) / dy 
                       + (U[indexMHD].bY - U[indexMHD - ny].bY) / dx - (U[indexMHD + 1 - ny].bX - U[indexMHD - ny].bX) / dy
                       + (U[indexMHD - 1 + ny].bY - U[indexMHD - 1].bY) / dx - (U[indexMHD].bX - U[indexMHD - 1].bX) / dy
                       + (U[indexMHD - 1].bY - U[indexMHD - 1 - ny].bY) / dx - (U[indexMHD - ny].bX - U[indexMHD - 1 - ny].bX) / dy);

        niMHD = rhoMHD / (PIC2DConst::device_mIon_PIC + PIC2DConst::device_mElectron_PIC);
        neMHD = niMHD;
        tiMHD = pMHD / 2.0f / niMHD;
        teMHD = pMHD / 2.0f / neMHD;

        rhoPIC = PIC2DConst::device_mIon_PIC * zerothMomentIon[indexPIC].n + PIC2DConst::device_mElectron_PIC * zerothMomentElectron[indexPIC].n;
        uPIC = (PIC2DConst::device_mIon_PIC * firstMomentIon[indexPIC].x + PIC2DConst::device_mElectron_PIC * firstMomentElectron[indexPIC].x) / rhoPIC;
        vPIC = (PIC2DConst::device_mIon_PIC * firstMomentIon[indexPIC].y + PIC2DConst::device_mElectron_PIC * firstMomentElectron[indexPIC].y) / rhoPIC;
        wPIC = (PIC2DConst::device_mIon_PIC * firstMomentIon[indexPIC].z + PIC2DConst::device_mElectron_PIC * firstMomentElectron[indexPIC].z) / rhoPIC;
        jXPIC = PIC2DConst::device_qIon_PIC * firstMomentIon[indexPIC].x + PIC2DConst::device_qElectron_PIC * firstMomentElectron[indexPIC].x;
        jYPIC = PIC2DConst::device_qIon_PIC * firstMomentIon[indexPIC].y + PIC2DConst::device_qElectron_PIC * firstMomentElectron[indexPIC].y;
        jZPIC = PIC2DConst::device_qIon_PIC * firstMomentIon[indexPIC].z + PIC2DConst::device_qElectron_PIC * firstMomentElectron[indexPIC].z;

        rhoPIC = interlockingFunctionY[j] * rhoMHD + (1.0f - interlockingFunctionY[j]) * rhoPIC;
        uPIC = interlockingFunctionY[j] * uMHD + (1.0f - interlockingFunctionY[j]) * uPIC;
        vPIC = interlockingFunctionY[j] * vMHD + (1.0f - interlockingFunctionY[j]) * vPIC;
        wPIC = interlockingFunctionY[j] * wMHD + (1.0f - interlockingFunctionY[j]) * wPIC;
        jXPIC = interlockingFunctionY[j] * jXMHD + (1.0f - interlockingFunctionY[j]) * jXPIC;
        jYPIC = interlockingFunctionY[j] * jYMHD + (1.0f - interlockingFunctionY[j]) * jYPIC;
        jZPIC = interlockingFunctionY[j] * jZMHD + (1.0f - interlockingFunctionY[j]) * jZPIC;

        niPIC = rhoPIC / (PIC2DConst::device_mIon_PIC + PIC2DConst::device_mElectron_PIC);
        nePIC = niPIC;
        vThiPIC = sqrt(2.0f * tiMHD / PIC2DConst::device_mIon_PIC);
        vThePIC = sqrt(2.0f * teMHD / PIC2DConst::device_mElectron_PIC);


        reloadParticlesDataIon[j + i * interfaceLength].number = int(niPIC);
        reloadParticlesDataElectron[j + i * interfaceLength].number = int(nePIC);
        reloadParticlesDataIon[j + i * interfaceLength].u = uPIC;
        reloadParticlesDataIon[j + i * interfaceLength].v = vPIC;
        reloadParticlesDataIon[j + i * interfaceLength].w = wPIC;
        reloadParticlesDataElectron[j + i * interfaceLength].u = uPIC - jXPIC / int(nePIC) / abs(PIC2DConst::device_qElectron_PIC);
        reloadParticlesDataElectron[j + i * interfaceLength].v = vPIC - jYPIC / int(nePIC) / abs(PIC2DConst::device_qElectron_PIC);
        reloadParticlesDataElectron[j + i * interfaceLength].w = wPIC - jZPIC / int(nePIC) / abs(PIC2DConst::device_qElectron_PIC);
        reloadParticlesDataIon[j + i * interfaceLength].vth = vThiPIC;
        reloadParticlesDataElectron[j + i * interfaceLength].vth = vThePIC;

        reloadParticlesIndexIon[j + i * interfaceLength] = int(niPIC);
        reloadParticlesIndexElectron[j + i * interfaceLength] = int(nePIC);
    }
}



__global__ void reloadParticles_kernel(
    const float* interlockingFunctionY, 
    const ReloadParticlesData* reloadParticlesDataSpecies, 
    const unsigned long long* reloadParticlesIndexSpecies, 
    const Particle* reloadParticlesSpecies, 
    unsigned long long reloadParticlesTotalNumSpecies, 
    Particle* particlesSpecies, 
    unsigned long long restartParticlesIndexSpecies, 
    int indexOfInterfaceStartInPIC, 
    int interfaceLength, 
    unsigned long long existNumSpecies, 
    int step
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (0 < i && i < PIC2DConst::device_nx_PIC - 1 && 0 < j && j < interfaceLength - 1) {
        int index = j + i * interfaceLength;
        //int reloadNum = reloadParticlesDataSpecies[index].number;
        float u = reloadParticlesDataSpecies[index].u;
        float v = reloadParticlesDataSpecies[index].v;
        float w = reloadParticlesDataSpecies[index].w;
        float vth = reloadParticlesDataSpecies[index].vth;
        Particle particleSource, particleReload;
        float x, y, z, vx, vy, vz, gamma;

        for (unsigned long long k = reloadParticlesIndexSpecies[index]; k < reloadParticlesIndexSpecies[index + 1]; k++) {
            hiprandState state; 
            hiprand_init(step, k, 0, &state);
            float randomValue = hiprand_uniform(&state);

            if (randomValue > 1.0f - interlockingFunctionY[j]) {
                particleSource = reloadParticlesSpecies[(restartParticlesIndexSpecies + k) % reloadParticlesTotalNumSpecies];

                x = particleSource.x; x += i * PIC2DConst::device_dx_PIC;
                y = particleSource.y; y += (indexOfInterfaceStartInPIC + j) * PIC2DConst::device_dy_PIC;
                z = 0.0f;
                vx = particleSource.vx; vx = u + vx * vth;
                vy = particleSource.vx; vy = v + vy * vth;
                vz = particleSource.vx; vz = w + vz * vth;
                gamma = sqrt(1.0f + (vx * vx + vy * vy + vz * vz) / (PIC2DConst::device_c_PIC * PIC2DConst::device_c_PIC));
                
                particleReload.x = x; particleReload.y = y; particleReload.z = z;
                particleReload.vx = vx; particleReload.vy = vy, particleReload.vz = vz; 
                particleReload.gamma = gamma;

                particlesSpecies[existNumSpecies + k] = particleReload;
                particlesSpecies[existNumSpecies + k].isExist = true;
            }

            
        }

    }
}

void Interface2D::sendMHDtoPIC_particle(
    const thrust::device_vector<ConservationParameter>& U, 
    thrust::device_vector<Particle>& particlesIon, 
    thrust::device_vector<Particle>& particlesElectron, 
    int step
)
{
    setMoments(particlesIon, particlesElectron);

    thrust::fill(reloadParticlesIndexIon.begin(), reloadParticlesIndexIon.end(), 0);
    thrust::fill(reloadParticlesIndexElectron.begin(), reloadParticlesIndexElectron.end(), 0);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((PIC2DConst::nx_PIC + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (interfaceLength + threadsPerBlock.y - 1) / threadsPerBlock.y);

    sendMHDtoPIC_particle_yDirection_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(interlockingFunctionY.data()), 
        thrust::raw_pointer_cast(zerothMomentIon.data()), 
        thrust::raw_pointer_cast(zerothMomentElectron.data()), 
        thrust::raw_pointer_cast(firstMomentIon.data()), 
        thrust::raw_pointer_cast(firstMomentElectron.data()), 
        thrust::raw_pointer_cast(U.data()),  
        thrust::raw_pointer_cast(reloadParticlesDataIon.data()), 
        thrust::raw_pointer_cast(reloadParticlesDataElectron.data()), 
        thrust::raw_pointer_cast(reloadParticlesIndexIon.data()), 
        thrust::raw_pointer_cast(reloadParticlesIndexElectron.data()), 
        indexOfInterfaceStartInMHD, 
        indexOfInterfaceStartInPIC, 
        interfaceLength
    );

    hipDeviceSynchronize();
    
    deleteParticles(particlesIon, particlesElectron, step);
    

    host_reloadParticlesIndexIon = reloadParticlesIndexIon;
    host_reloadParticlesIndexElectron = reloadParticlesIndexElectron;
    
    for (int i = 0; i < PIC2DConst::nx_PIC; i++) {
        for (int j = 0; j < interfaceLength; j++) {

            if (j == 0 && i == 0) continue;

            host_reloadParticlesIndexIon[j + i * interfaceLength] += host_reloadParticlesIndexIon[j + i * interfaceLength - 1];
            host_reloadParticlesIndexElectron[j + i * interfaceLength] += host_reloadParticlesIndexElectron[j + i * interfaceLength - 1];
        }
    }

    reloadParticlesIndexIon = host_reloadParticlesIndexIon;
    reloadParticlesIndexElectron = host_reloadParticlesIndexElectron;
    

    std::random_device seedIon, seedElectron;
    std::mt19937 genIon(seedIon()), genElectron(seedElectron());
    std::uniform_int_distribution<unsigned long long> distIon(0, Interface2DConst::reloadParticlesTotalNumIon);
    std::uniform_int_distribution<unsigned long long> distElectron(0, Interface2DConst::reloadParticlesTotalNumElectron);
    restartParticlesIndexIon = distIon(genIon);
    restartParticlesIndexElectron = distElectron(genElectron);

    reloadParticles_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(interlockingFunctionY.data()), 
        thrust::raw_pointer_cast(reloadParticlesDataIon.data()), 
        thrust::raw_pointer_cast(reloadParticlesIndexIon.data()), 
        thrust::raw_pointer_cast(reloadParticlesSourceIon.data()), 
        Interface2DConst::reloadParticlesTotalNumIon,  
        thrust::raw_pointer_cast(particlesIon.data()), 
        restartParticlesIndexIon, 
        indexOfInterfaceStartInPIC, 
        interfaceLength, 
        PIC2DConst::existNumIon_PIC, 
        step
    );

    hipDeviceSynchronize();

    reloadParticles_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(interlockingFunctionY.data()), 
        thrust::raw_pointer_cast(reloadParticlesDataElectron.data()), 
        thrust::raw_pointer_cast(reloadParticlesIndexElectron.data()), 
        thrust::raw_pointer_cast(reloadParticlesSourceElectron.data()), 
        Interface2DConst::reloadParticlesTotalNumElectron, 
        thrust::raw_pointer_cast(particlesElectron.data()), 
        restartParticlesIndexElectron, 
        indexOfInterfaceStartInPIC, 
        interfaceLength, 
        PIC2DConst::existNumElectron_PIC, 
        step
    );

    hipDeviceSynchronize();


    PIC2DConst::existNumIon_PIC = thrust::transform_reduce(
        particlesIon.begin(),
        particlesIon.end(),
        IsExistTransform(), 
        0,               
        thrust::plus<unsigned long long>()
    );

    hipDeviceSynchronize();

    PIC2DConst::existNumElectron_PIC = thrust::transform_reduce(
        particlesElectron.begin(),
        particlesElectron.end(),
        IsExistTransform(), 
        0,               
        thrust::plus<unsigned long long>()
    );

    hipDeviceSynchronize();

    thrust::partition(
        particlesIon.begin(), particlesIon.end(), 
        [] __device__ (const Particle& p) { return p.isExist; }
    );
    hipDeviceSynchronize();

    thrust::partition(
        particlesElectron.begin(), particlesElectron.end(), 
        [] __device__ (const Particle& p) { return p.isExist; }
    );
    hipDeviceSynchronize();
}


void Interface2D::setMoments(
    const thrust::device_vector<Particle>& particlesIon, 
    const thrust::device_vector<Particle>& particlesElectron
)
{
    momentCalculater.calculateZerothMomentOfOneSpecies(
        zerothMomentIon, particlesIon, PIC2DConst::existNumIon_PIC
    );
    momentCalculater.calculateZerothMomentOfOneSpecies(
        zerothMomentElectron, particlesElectron, PIC2DConst::existNumElectron_PIC
    );

    momentCalculater.calculateFirstMomentOfOneSpecies(
        firstMomentIon, particlesIon, PIC2DConst::existNumIon_PIC
    );
    momentCalculater.calculateFirstMomentOfOneSpecies(
        firstMomentElectron, particlesElectron, PIC2DConst::existNumElectron_PIC
    );
}


__global__ void deleteParticles_kernel(
    const float* interlockingFunctionY, 
    Particle* particlesSpecies, 
    const int indexOfInterfaceStartInPIC, 
    int interfaceLength, 
    const unsigned long long existNumSpecies, 
    int step
)
{
    unsigned long long i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < existNumSpecies) {
        float y = particlesSpecies[i].y;
        float interfaceMin = indexOfInterfaceStartInPIC * PIC2DConst::device_dy_PIC;
        float interfaceMax = (indexOfInterfaceStartInPIC + interfaceLength) * PIC2DConst::device_dy_PIC;
        if (y >= interfaceMin + PIC2DConst::device_dy_PIC && y <= interfaceMax - PIC2DConst::device_dy_PIC) {
            int j = floor(y) - indexOfInterfaceStartInPIC;
            hiprandState state; 
            hiprand_init(step, i, 0, &state);
            float randomValue = hiprand_uniform(&state);
            if (randomValue > 1.0f - interlockingFunctionY[j]) {
                particlesSpecies[i].isExist = false;
            }
        }
    }
}

void Interface2D::deleteParticles(
    thrust::device_vector<Particle>& particlesIon, 
    thrust::device_vector<Particle>& particlesElectron, 
    int step
)
{

    dim3 threadsPerBlockForIon(256);
    dim3 blocksPerGridForIon((PIC2DConst::existNumIon_PIC + threadsPerBlockForIon.x - 1) / threadsPerBlockForIon.x);

    deleteParticles_kernel<<<blocksPerGridForIon, threadsPerBlockForIon>>>(
        thrust::raw_pointer_cast(interlockingFunctionY.data()), 
        thrust::raw_pointer_cast(particlesIon.data()),
        indexOfInterfaceStartInPIC, 
        interfaceLength, 
        PIC2DConst::existNumIon_PIC, 
        step
    );

    hipDeviceSynchronize();

    dim3 threadsPerBlockForElectron(256);
    dim3 blocksPerGridForElectron((PIC2DConst::existNumElectron_PIC + threadsPerBlockForElectron.x - 1) / threadsPerBlockForElectron.x);

    deleteParticles_kernel<<<blocksPerGridForElectron, threadsPerBlockForElectron>>>(
        thrust::raw_pointer_cast(interlockingFunctionY.data()), 
        thrust::raw_pointer_cast(particlesElectron.data()),
        indexOfInterfaceStartInPIC, 
        interfaceLength, 
        PIC2DConst::existNumElectron_PIC, 
        step 
    );

    hipDeviceSynchronize();

    
    PIC2DConst::existNumIon_PIC = thrust::transform_reduce(
        particlesIon.begin(),
        particlesIon.end(),
        IsExistTransform(), 
        0,               
        thrust::plus<unsigned long long>()
    );

    hipDeviceSynchronize();

    PIC2DConst::existNumElectron_PIC = thrust::transform_reduce(
        particlesElectron.begin(),
        particlesElectron.end(),
        IsExistTransform(), 
        0,               
        thrust::plus<unsigned long long>()
    );

    hipDeviceSynchronize();


    thrust::partition(
        particlesIon.begin(), particlesIon.end(), 
        [] __device__ (const Particle& p) { return p.isExist; }
    );
    hipDeviceSynchronize();

    thrust::partition(
        particlesElectron.begin(), particlesElectron.end(), 
        [] __device__ (const Particle& p) { return p.isExist; }
    );
    hipDeviceSynchronize();
}



///////////////////////////////////////////////////////


__global__ void setUHalf_kernel(
    const ConservationParameter* UPast, 
    const ConservationParameter* UNext, 
    ConservationParameter* UHalf
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < IdealMHD2DConst::device_nx_MHD && j < IdealMHD2DConst::device_ny_MHD) {
        int index = j + i * IdealMHD2DConst::device_ny_MHD;

        UHalf[index].rho = 0.5 * (UPast[index].rho + UNext[index].rho);
        UHalf[index].rhoU = 0.5 * (UPast[index].rhoU + UNext[index].rhoU);
        UHalf[index].rhoV = 0.5 * (UPast[index].rhoV + UNext[index].rhoV);
        UHalf[index].rhoW = 0.5 * (UPast[index].rhoW + UNext[index].rhoW);
        UHalf[index].bX = 0.5 * (UPast[index].bX + UNext[index].bX);
        UHalf[index].bY = 0.5 * (UPast[index].bY + UNext[index].bY);
        UHalf[index].bZ = 0.5 * (UPast[index].bZ + UNext[index].bZ);
        UHalf[index].e = 0.5 * (UPast[index].e + UNext[index].e);
    }
}


__global__ void sendPICtoMHD_kernel(
    const float* interlockingFunctionY, 
    const ZerothMoment* zerothMomentIon, 
    const ZerothMoment* ZerothMomentElectron, 
    const FirstMoment* firstMomentIon, 
    const FirstMoment* firstMomentElectron, 
    const MagneticField* B, 
    ConservationParameter* U, 
    int indexOfInterfaceStartInMHD, 
    int indexOfInterfaceStartInPIC, 
    int interfaceLength
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (0 < i && i < PIC2DConst::device_nx_PIC && 0 < j && j < interfaceLength - 1) {
        int indexPIC = indexOfInterfaceStartInPIC + j + i * PIC2DConst::device_ny_PIC;
        int indexMHD = indexOfInterfaceStartInMHD + j + i * IdealMHD2DConst::device_ny_MHD;
        float rhoMHD, uMHD, vMHD, wMHD, bXMHD, bYMHD, bZMHD, eMHD, pMHD;
        float rhoPIC, uPIC, vPIC, wPIC, bXPIC, bYPIC, bZPIC;
        float niMHD, neMHD, tiMHD, teMHD;

        //MHDのグリッドにPICを合わせる(=MHDグリッドは整数格子点上にあるので、PICグリッドを整数格子点上に再配置する)
        rhoMHD = U[indexMHD].rho;
        uMHD = U[indexMHD].rhoU / rhoMHD;
        vMHD = U[indexMHD].rhoV / rhoMHD;
        wMHD = U[indexMHD].rhoW / rhoMHD;
        bXMHD = U[indexMHD].bX;
        bYMHD = U[indexMHD].bY;
        bZMHD = U[indexMHD].bZ;
        eMHD = U[indexMHD].e;
        pMHD = (IdealMHD2DConst::device_gamma_MHD - 1.0f)
             * (eMHD - 0.5f * rhoMHD * (uMHD * uMHD + vMHD * vMHD + wMHD * wMHD)
             - 0.5f * (bXMHD * bXMHD + bYMHD * bYMHD + bZMHD * bZMHD));
        //tiMHD, teMHDはMHDの情報のままにするために、この計算が必要。
        niMHD = rhoMHD / (PIC2DConst::device_mIon_PIC + PIC2DConst::device_mElectron_PIC);
        neMHD = niMHD;
        tiMHD = pMHD / 2.0f / niMHD;
        teMHD = pMHD / 2.0f / neMHD;
        
        rhoPIC = PIC2DConst::device_mIon_PIC * zerothMomentIon[indexPIC].n + PIC2DConst::device_mElectron_PIC * ZerothMomentElectron[indexPIC].n;
        uPIC = (PIC2DConst::device_mIon_PIC * firstMomentIon[indexPIC].x + PIC2DConst::device_mElectron_PIC * firstMomentElectron[indexPIC].x) / rhoPIC;
        vPIC = (PIC2DConst::device_mIon_PIC * firstMomentIon[indexPIC].y + PIC2DConst::device_mElectron_PIC * firstMomentElectron[indexPIC].y) / rhoPIC;
        wPIC = (PIC2DConst::device_mIon_PIC * firstMomentIon[indexPIC].z + PIC2DConst::device_mElectron_PIC * firstMomentElectron[indexPIC].z) / rhoPIC;
        bXPIC = 0.5f * (B[indexPIC].bX + B[indexPIC - 1].bX);
        bYPIC = 0.5f * (B[indexPIC].bY + B[indexPIC - PIC2DConst::device_nx_PIC].bY);
        bZPIC = 0.25f * (B[indexPIC].bZ + B[indexPIC - PIC2DConst::device_nx_PIC].bZ + B[indexPIC - 1].bZ + B[indexPIC - PIC2DConst::device_nx_PIC - 1].bZ);

        rhoMHD = interlockingFunctionY[j] * rhoMHD + (1.0f - interlockingFunctionY[j]) * rhoPIC;
        uMHD = interlockingFunctionY[j]   * uMHD   + (1.0f - interlockingFunctionY[j]) * uPIC;
        vMHD = interlockingFunctionY[j]   * vMHD   + (1.0f - interlockingFunctionY[j]) * vPIC;
        wMHD = interlockingFunctionY[j]   * wMHD   + (1.0f - interlockingFunctionY[j]) * wPIC;
        bXMHD = interlockingFunctionY[j]  * bXMHD  + (1.0f - interlockingFunctionY[j]) * bXPIC;
        bYMHD = interlockingFunctionY[j]  * bYMHD  + (1.0f - interlockingFunctionY[j]) * bYPIC;
        bZMHD = interlockingFunctionY[j]  * bZMHD  + (1.0f - interlockingFunctionY[j]) * bZPIC;

        niMHD = rhoMHD / (PIC2DConst::device_mIon_PIC + PIC2DConst::device_mElectron_PIC);
        neMHD = niMHD;
        pMHD = niMHD * tiMHD + neMHD * teMHD;


        U[indexMHD].rho = rhoMHD;
        U[indexMHD].rhoU = rhoMHD * uMHD;
        U[indexMHD].rhoV = rhoMHD * vMHD;
        U[indexMHD].rhoW = rhoMHD * wMHD;
        U[indexMHD].bX = bXMHD;
        U[indexMHD].bY = bYMHD;
        U[indexMHD].bZ = bZMHD;
        eMHD = pMHD / (IdealMHD2DConst::device_gamma_MHD - 1.0f)
             + 0.5f * rhoMHD * (uMHD * uMHD + vMHD * vMHD + wMHD * wMHD)
             + 0.5f * (bXMHD * bXMHD + bYMHD * bYMHD + bZMHD * bZMHD);
        U[indexMHD].e = eMHD;
    }
}


//MHDのグリッドを整数格子点上に再配置してから使うこと
void Interface2D::sendPICtoMHD(
    const thrust::device_vector<ConservationParameter>& UPast, 
    const thrust::device_vector<ConservationParameter>& UNext
)
{
    dim3 threadsPerBlockForSetUHalf(16, 16);
    dim3 blocksPerGridForSetUHalf((IdealMHD2DConst::nx_MHD + threadsPerBlockForSetUHalf.x - 1) / threadsPerBlockForSetUHalf.x,
                                  (IdealMHD2DConst::ny_MHD + threadsPerBlockForSetUHalf.y - 1) / threadsPerBlockForSetUHalf.y);

    setUHalf_kernel<<<blocksPerGridForSetUHalf, threadsPerBlockForSetUHalf>>>(
        thrust::raw_pointer_cast(UPast.data()), 
        thrust::raw_pointer_cast(UNext.data()), 
        thrust::raw_pointer_cast(UHalf.data())
    );


    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((PIC2DConst::nx_PIC + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (interfaceLength + threadsPerBlock.y - 1) / threadsPerBlock.y);

    sendPICtoMHD_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(interlockingFunctionY.data()), 
        thrust::raw_pointer_cast(zerothMomentIon_timeAve.data()), 
        thrust::raw_pointer_cast(zerothMomentElectron_timeAve.data()), 
        thrust::raw_pointer_cast(firstMomentIon_timeAve.data()), 
        thrust::raw_pointer_cast(firstMomentElectron_timeAve.data()), 
        thrust::raw_pointer_cast(B_timeAve.data()), 
        thrust::raw_pointer_cast(UHalf.data()), 
        indexOfInterfaceStartInMHD, 
        indexOfInterfaceStartInPIC, 
        interfaceLength
    );

    hipDeviceSynchronize();

}


thrust::device_vector<ConservationParameter>& Interface2D::getUHalfRef()
{
    return UHalf;
}

//////////////////////////////


__global__ void calculateSubU_kernel(
    const ConservationParameter* UPast, 
    const ConservationParameter* UNext, 
    ConservationParameter* USub, 
    float mixingRatio, 
    int indexOfInterfaceStartInMHD
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < IdealMHD2DConst::device_nx_MHD && j < IdealMHD2DConst::device_ny_MHD) {
        int index = j + i * IdealMHD2DConst::device_ny_MHD;

        USub[index].rho = mixingRatio * UPast[index].rho + (1.0 - mixingRatio) * UNext[index].rho;
        USub[index].rhoU = mixingRatio * UPast[index].rhoU + (1.0 - mixingRatio) * UNext[index].rhoU;
        USub[index].rhoV = mixingRatio * UPast[index].rhoV + (1.0 - mixingRatio) * UNext[index].rhoV;
        USub[index].rhoW = mixingRatio * UPast[index].rhoW + (1.0 - mixingRatio) * UNext[index].rhoW;
        USub[index].bX = mixingRatio * UPast[index].bX + (1.0 - mixingRatio) * UNext[index].bX;
        USub[index].bY = mixingRatio * UPast[index].bY + (1.0 - mixingRatio) * UNext[index].bY;
        USub[index].bZ = mixingRatio * UPast[index].bZ + (1.0 - mixingRatio) * UNext[index].bZ;
        USub[index].e = mixingRatio * UPast[index].e + (1.0 - mixingRatio) * UNext[index].e;
    }
}

thrust::device_vector<ConservationParameter>& Interface2D::calculateAndGetSubU(
    const thrust::device_vector<ConservationParameter>& UPast, 
    const thrust::device_vector<ConservationParameter>& UNext, 
    float mixingRatio
)
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((IdealMHD2DConst::nx_MHD + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (IdealMHD2DConst::ny_MHD + threadsPerBlock.y - 1) / threadsPerBlock.y);

    calculateSubU_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(UPast.data()), 
        thrust::raw_pointer_cast(UNext.data()), 
        thrust::raw_pointer_cast(USub.data()), 
        mixingRatio, 
        indexOfInterfaceStartInMHD
    );

    hipDeviceSynchronize();

    return USub;
}


void Interface2D::resetTimeAveParameters()
{
    thrust::fill(
        B_timeAve.begin(), 
        B_timeAve.end(), 
        MagneticField()
    );

    thrust::fill(
        zerothMomentIon_timeAve.begin(), 
        zerothMomentIon_timeAve.end(), 
        ZerothMoment()
    );
    thrust::fill(
        zerothMomentElectron_timeAve.begin(), 
        zerothMomentElectron_timeAve.end(), 
        ZerothMoment()
    );

    thrust::fill(
        firstMomentIon_timeAve.begin(), 
        firstMomentIon_timeAve.end(), 
        FirstMoment()
    );
    thrust::fill(
        firstMomentElectron_timeAve.begin(), 
        firstMomentElectron_timeAve.end(), 
        FirstMoment()
    );
}


void Interface2D::sumUpTimeAveParameters(
    const thrust::device_vector<MagneticField>& B, 
    const thrust::device_vector<Particle>& particlesIon, 
    const thrust::device_vector<Particle>& particlesElectron
)
{
    thrust::transform(
        B_timeAve.begin(), B_timeAve.end(), B.begin(), 
        B_timeAve.begin(), thrust::plus<MagneticField>()
    );
    
    setMoments(particlesIon, particlesElectron);
    thrust::transform(
        zerothMomentIon_timeAve.begin(), zerothMomentIon_timeAve.end(), zerothMomentIon.begin(), 
        zerothMomentIon_timeAve.begin(), thrust::plus<ZerothMoment>()
    );
    thrust::transform(
        zerothMomentElectron_timeAve.begin(), zerothMomentElectron_timeAve.end(), zerothMomentElectron.begin(), 
        zerothMomentElectron_timeAve.begin(), thrust::plus<ZerothMoment>()
    );
    thrust::transform(
        firstMomentIon_timeAve.begin(), firstMomentIon_timeAve.end(), firstMomentIon.begin(), 
        firstMomentIon_timeAve.begin(), thrust::plus<FirstMoment>()
    );
    thrust::transform(
        firstMomentElectron_timeAve.begin(), firstMomentElectron_timeAve.end(), firstMomentElectron.begin(), 
        firstMomentElectron_timeAve.begin(), thrust::plus<FirstMoment>()
    );
}


__global__ void calculateTimeAveParameters_kernel(
    MagneticField* B_timeAve, 
    ZerothMoment* zerothMomentIon_timeAve, 
    ZerothMoment* zerothMomentElectron_timeAve, 
    FirstMoment* firstMomentIon_timeAve, 
    FirstMoment* firstMomentElectron_timeAve, 
    int substeps 
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < PIC2DConst::device_nx_PIC && j < PIC2DConst::device_ny_PIC) {
        int index = j + i * PIC2DConst::device_ny_PIC;

        B_timeAve[index].bX /= static_cast<float>(substeps);
        B_timeAve[index].bY /= static_cast<float>(substeps);
        B_timeAve[index].bZ /= static_cast<float>(substeps);
        zerothMomentIon_timeAve[index].n /= static_cast<float>(substeps);
        zerothMomentElectron_timeAve[index].n /= static_cast<float>(substeps);
        firstMomentIon_timeAve[index].x /= static_cast<float>(substeps);
        firstMomentIon_timeAve[index].y /= static_cast<float>(substeps);
        firstMomentIon_timeAve[index].z /= static_cast<float>(substeps);
        firstMomentElectron_timeAve[index].x /= static_cast<float>(substeps);
        firstMomentElectron_timeAve[index].y /= static_cast<float>(substeps);
        firstMomentElectron_timeAve[index].z /= static_cast<float>(substeps);
    }
}

void Interface2D::calculateTimeAveParameters(int substeps)
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((PIC2DConst::nx_PIC + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (PIC2DConst::ny_PIC + threadsPerBlock.y - 1) / threadsPerBlock.y);

    calculateTimeAveParameters_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(B_timeAve.data()), 
        thrust::raw_pointer_cast(zerothMomentIon_timeAve.data()), 
        thrust::raw_pointer_cast(zerothMomentElectron_timeAve.data()), 
        thrust::raw_pointer_cast(firstMomentIon_timeAve.data()), 
        thrust::raw_pointer_cast(firstMomentElectron_timeAve.data()), 
        substeps
    );

    hipDeviceSynchronize();
}

