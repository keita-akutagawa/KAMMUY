#include "const.hpp"


using namespace Interface2DConst;


void initializeDeviceConstants_Interface()
{
    hipMemcpyToSymbol(HIP_SYMBOL(device_EPS), &EPS, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_PI), &PI, sizeof(double));

    hipMemcpyToSymbol(HIP_SYMBOL(device_windowSizeForConvolution), &windowSizeForConvolution, sizeof(int));

    hipMemcpyToSymbol(HIP_SYMBOL(device_reloadParticlesTotalNumIon), &reloadParticlesTotalNumIon, sizeof(unsigned long long));
    hipMemcpyToSymbol(HIP_SYMBOL(device_reloadParticlesTotalNumElectron), &reloadParticlesTotalNumElectron, sizeof(unsigned long long));
}


