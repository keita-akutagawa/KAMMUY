#include "const.hpp"


using namespace Interface2DConst;


void initializeDeviceConstants_Interface()
{
    hipMemcpyToSymbol(HIP_SYMBOL(device_PI), &PI, sizeof(double));

    hipMemcpyToSymbol(HIP_SYMBOL(device_windowSizeForRemoveNoiseByConvolution), &windowSizeForRemoveNoiseByConvolution, sizeof(int));

    hipMemcpyToSymbol(HIP_SYMBOL(device_reloadParticlesTotalNumIon), &reloadParticlesTotalNumIon, sizeof(unsigned long long));
    hipMemcpyToSymbol(HIP_SYMBOL(device_reloadParticlesTotalNumElectron), &reloadParticlesTotalNumElectron, sizeof(unsigned long long));
}


