#include "const.hpp"


using namespace Interface2DConst;


void initializeDeviceConstants()
{
    hipMemcpyToSymbol(HIP_SYMBOL(device_PI), &PI, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(device_windowSizeForRemoveNoiseByConvolution), &windowSizeForRemoveNoiseByConvolution, sizeof(int));

    hipMemcpyToSymbol(HIP_SYMBOL(device_reloadParticlesTotalNumIon), &reloadParticlesTotalNumIon, sizeof(unsigned long long));
    hipMemcpyToSymbol(HIP_SYMBOL(device_reloadParticlesTotalNumElectron), &reloadParticlesTotalNumElectron, sizeof(unsigned long long));
}


