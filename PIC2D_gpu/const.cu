#include "const.hpp"


using namespace PIC2DConst;

void initializeDeviceConstants_PIC()
{
    hipMemcpyToSymbol(HIP_SYMBOL(device_c_PIC), &c_PIC, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_epsilon0_PIC), &epsilon0_PIC, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_mu0_PIC), &mu0_PIC, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_dOfLangdonMarderTypeCorrection_PIC), &dOfLangdonMarderTypeCorrection_PIC, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_EPS_PIC), &EPS_PIC, sizeof(double));

    hipMemcpyToSymbol(HIP_SYMBOL(device_nx_PIC), &nx_PIC, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(device_dx_PIC), &dx_PIC, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_xmin_PIC), &xmin_PIC, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_xmax_PIC), &xmax_PIC, sizeof(double));

    hipMemcpyToSymbol(HIP_SYMBOL(device_ny_PIC), &ny_PIC, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(device_dy_PIC), &dy_PIC, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_ymin_PIC), &ymin_PIC, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_ymax_PIC), &ymax_PIC, sizeof(double));

    hipMemcpyToSymbol(HIP_SYMBOL(device_dt_PIC), &dt_PIC, sizeof(double));

    hipMemcpyToSymbol(HIP_SYMBOL(device_numberDensityIon_PIC), &numberDensityIon_PIC, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(device_numberDensityElectron_PIC), &numberDensityElectron_PIC, sizeof(int));

    hipMemcpyToSymbol(HIP_SYMBOL(device_totalNumIon_PIC), &totalNumIon_PIC, sizeof(unsigned long long));
    hipMemcpyToSymbol(HIP_SYMBOL(device_totalNumElectron_PIC), &totalNumElectron_PIC, sizeof(unsigned long long));
    hipMemcpyToSymbol(HIP_SYMBOL(device_totalNumParticles_PIC), &totalNumParticles_PIC, sizeof(unsigned long long));
    hipMemcpyToSymbol(HIP_SYMBOL(device_existNumIon_PIC), &existNumIon_PIC, sizeof(unsigned long long));
    hipMemcpyToSymbol(HIP_SYMBOL(device_existNumElectron_PIC), &existNumElectron_PIC, sizeof(unsigned long long));

    hipMemcpyToSymbol(HIP_SYMBOL(device_B0_PIC), &B0_PIC, sizeof(double));

    hipMemcpyToSymbol(HIP_SYMBOL(device_mRatio_PIC), &mRatio_PIC, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_mIon_PIC), &mIon_PIC, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_mElectron_PIC), &mElectron_PIC, sizeof(double));

    hipMemcpyToSymbol(HIP_SYMBOL(device_tRatio_PIC), &tRatio_PIC, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_tIon_PIC), &tIon_PIC, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_tElectron_PIC), &tElectron_PIC, sizeof(double));

    hipMemcpyToSymbol(HIP_SYMBOL(device_qRatio_PIC), &qRatio_PIC, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_qIon_PIC), &qIon_PIC, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_qElectron_PIC), &qElectron_PIC, sizeof(double));

    hipMemcpyToSymbol(HIP_SYMBOL(device_omegaPe_PIC), &omegaPe_PIC, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_omegaPi_PIC), &omegaPi_PIC, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_omegaCe_PIC), &omegaCe_PIC, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_omegaCi_PIC), &omegaCi_PIC, sizeof(double));

    hipMemcpyToSymbol(HIP_SYMBOL(device_debyeLength_PIC), &debyeLength_PIC, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_ionInertialLength_PIC), &ionInertialLength_PIC, sizeof(double));

    hipMemcpyToSymbol(HIP_SYMBOL(device_vThIon_PIC), &vThIon_PIC, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_vThElectron_PIC), &vThElectron_PIC, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_bulkVxIon_PIC), &bulkVxIon_PIC, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_bulkVyIon_PIC), &bulkVyIon_PIC, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_bulkVzIon_PIC), &bulkVzIon_PIC, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_bulkVxElectron_PIC), &bulkVxElectron_PIC, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_bulkVyElectron_PIC), &bulkVyElectron_PIC, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_bulkVzElectron_PIC), &bulkVzElectron_PIC, sizeof(double));

    hipMemcpyToSymbol(HIP_SYMBOL(device_totalStep_PIC), &totalStep_PIC, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(device_totalTime_PIC), &totalTime_PIC, sizeof(double));
}

