#include "hip/hip_runtime.h"
#include "moment_calculator.hpp"


MomentCalculator::MomentCalculator(PIC2DMPI::MPIInfo& mPIInfo)
    : mPIInfo(mPIInfo)
{
}


void MomentCalculator::resetZerothMomentOfOneSpecies(
    thrust::device_vector<ZerothMoment>& zerothMomentOfOneSpecies
)
{
    thrust::fill(
        zerothMomentOfOneSpecies.begin(), 
        zerothMomentOfOneSpecies.end(), 
        ZerothMoment()
    );
    hipDeviceSynchronize();
}

void MomentCalculator::resetFirstMomentOfOneSpecies(
    thrust::device_vector<FirstMoment>& firstMomentOfOneSpecies
)
{
    thrust::fill(
        firstMomentOfOneSpecies.begin(), 
        firstMomentOfOneSpecies.end(), 
        FirstMoment()
    );
    hipDeviceSynchronize();
}

void MomentCalculator::resetSecondMomentOfOneSpecies(
    thrust::device_vector<SecondMoment>& secondMomentOfOneSpecies
)
{
    thrust::fill(
        secondMomentOfOneSpecies.begin(), 
        secondMomentOfOneSpecies.end(), 
        SecondMoment()
    );
    hipDeviceSynchronize();
}

//////////

__global__ void calculateZerothMomentOfOneSpecies_kernel(
    ZerothMoment* zerothMomentOfOneSpecies, 
    const Particle* particlesSpecies, 
    const unsigned long long existNumSpecies, 
    const int localNx, const int buffer, 
    const int localSizeX, 
    const double xminForProcs, const double xmaxForProcs
)
{
    unsigned long long i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < existNumSpecies) {

        double cx1, cx2; 
        int xIndex1, xIndex2;
        double xOverDx;
        double cy1, cy2; 
        int yIndex1, yIndex2;
        double yOverDy;

        xOverDx = (particlesSpecies[i].x - xminForProcs + buffer * PIC2DConst::device_dx) / PIC2DConst::device_dx;
        yOverDy = (particlesSpecies[i].y - PIC2DConst::device_ymin) / PIC2DConst::device_dy;

        xIndex1 = floor(xOverDx);
        xIndex2 = xIndex1 + 1;
        xIndex2 = (xIndex2 == localSizeX) ? 0 : xIndex2;
        yIndex1 = floor(yOverDy);
        yIndex2 = yIndex1 + 1;
        yIndex2 = (yIndex2 == PIC2DConst::device_ny) ? 0 : yIndex2;

        if (xIndex1 < 0 || xIndex1 >= localSizeX) printf("x = %f, index = %d, ERROR\n", particlesSpecies[i].x, xIndex1); 
        if (yIndex1 < 0 || yIndex1 >= PIC2DConst::device_ny) printf("y = %f, index = %d, ERROR\n", particlesSpecies[i].y, yIndex1);

        cx1 = xOverDx - xIndex1;
        cx2 = 1.0 - cx1;
        cy1 = yOverDy - yIndex1;
        cy2 = 1.0 - cy1;

        unsigned long long index11 = yIndex1 + PIC2DConst::device_ny * xIndex1; 
        unsigned long long index12 = yIndex2 + PIC2DConst::device_ny * xIndex1; 
        unsigned long long index21 = yIndex1 + PIC2DConst::device_ny * xIndex2; 
        unsigned long long index22 = yIndex2 + PIC2DConst::device_ny * xIndex2; 
        atomicAdd(&(zerothMomentOfOneSpecies[index11].n), cx2 * cy2);
        atomicAdd(&(zerothMomentOfOneSpecies[index12].n), cx2 * cy1);
        atomicAdd(&(zerothMomentOfOneSpecies[index21].n), cx1 * cy2);
        atomicAdd(&(zerothMomentOfOneSpecies[index22].n), cx1 * cy1);
    }
};


void MomentCalculator::calculateZerothMomentOfOneSpecies(
    thrust::device_vector<ZerothMoment>& zerothMomentOfOneSpecies, 
    const thrust::device_vector<Particle>& particlesSpecies, 
    unsigned long long existNumSpecies
)
{
    resetZerothMomentOfOneSpecies(zerothMomentOfOneSpecies);

    dim3 threadsPerBlock(256);
    dim3 blocksPerGrid((existNumSpecies + threadsPerBlock.x - 1) / threadsPerBlock.x);

    calculateZerothMomentOfOneSpecies_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(zerothMomentOfOneSpecies.data()), 
        thrust::raw_pointer_cast(particlesSpecies.data()), 
        existNumSpecies, 
        mPIInfo.localNx, mPIInfo.buffer, 
        mPIInfo.localSizeX, 
        mPIInfo.xminForProcs, mPIInfo.xmaxForProcs
    );
    hipDeviceSynchronize();
}



__global__ void calculateFirstMomentOfOneSpecies_kernel(
    FirstMoment* firstMomentOfOneSpecies, 
    const Particle* particlesSpecies, 
    const unsigned long long existNumSpecies, 
    const int localNx, const int buffer, 
    const int localSizeX, 
    const double xminForProcs, const double xmaxForProcs
)
{
    unsigned long long i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < existNumSpecies) {
    
        double cx1, cx2; 
        int xIndex1, xIndex2;
        double xOverDx;
        double cy1, cy2; 
        int yIndex1, yIndex2;
        double yOverDy;
        double vx, vy, vz;

        xOverDx = (particlesSpecies[i].x - xminForProcs + buffer * PIC2DConst::device_dx) / PIC2DConst::device_dx;
        yOverDy = (particlesSpecies[i].y - PIC2DConst::device_ymin) / PIC2DConst::device_dy;

        xIndex1 = floor(xOverDx);
        xIndex2 = xIndex1 + 1;
        xIndex2 = (xIndex2 == localSizeX) ? 0 : xIndex2;
        yIndex1 = floor(yOverDy);
        yIndex2 = yIndex1 + 1;
        yIndex2 = (yIndex2 == PIC2DConst::device_ny) ? 0 : yIndex2;
        
        if (xIndex1 < 0 || xIndex1 >= localSizeX) printf("x = %f, index = %d, ERROR\n", particlesSpecies[i].x, xIndex1); 
        if (yIndex1 < 0 || yIndex1 >= PIC2DConst::device_ny) printf("y = %f, index = %d, ERROR\n", particlesSpecies[i].y, yIndex1);

        cx1 = xOverDx - xIndex1;
        cx2 = 1.0 - cx1;
        cy1 = yOverDy - yIndex1;
        cy2 = 1.0 - cy1;

        vx = particlesSpecies[i].vx / particlesSpecies[i].gamma;
        vy = particlesSpecies[i].vy / particlesSpecies[i].gamma;
        vz = particlesSpecies[i].vz / particlesSpecies[i].gamma;

        unsigned long long index11 = yIndex1 + PIC2DConst::device_ny * xIndex1; 
        unsigned long long index12 = yIndex2 + PIC2DConst::device_ny * xIndex1; 
        unsigned long long index21 = yIndex1 + PIC2DConst::device_ny * xIndex2; 
        unsigned long long index22 = yIndex2 + PIC2DConst::device_ny * xIndex2; 

        atomicAdd(&(firstMomentOfOneSpecies[index11].x), vx * cx2 * cy2);
        atomicAdd(&(firstMomentOfOneSpecies[index12].x), vx * cx2 * cy1);
        atomicAdd(&(firstMomentOfOneSpecies[index21].x), vx * cx1 * cy2);
        atomicAdd(&(firstMomentOfOneSpecies[index22].x), vx * cx1 * cy1);

        atomicAdd(&(firstMomentOfOneSpecies[index11].y), vy * cx2 * cy2);
        atomicAdd(&(firstMomentOfOneSpecies[index12].y), vy * cx2 * cy1);
        atomicAdd(&(firstMomentOfOneSpecies[index21].y), vy * cx1 * cy2);
        atomicAdd(&(firstMomentOfOneSpecies[index22].y), vy * cx1 * cy1);

        atomicAdd(&(firstMomentOfOneSpecies[index11].z), vz * cx2 * cy2);
        atomicAdd(&(firstMomentOfOneSpecies[index12].z), vz * cx2 * cy1);
        atomicAdd(&(firstMomentOfOneSpecies[index21].z), vz * cx1 * cy2);
        atomicAdd(&(firstMomentOfOneSpecies[index22].z), vz * cx1 * cy1);
    }
};


void MomentCalculator::calculateFirstMomentOfOneSpecies(
    thrust::device_vector<FirstMoment>& firstMomentOfOneSpecies, 
    const thrust::device_vector<Particle>& particlesSpecies, 
    unsigned long long existNumSpecies
)
{
    resetFirstMomentOfOneSpecies(firstMomentOfOneSpecies);

    dim3 threadsPerBlock(256);
    dim3 blocksPerGrid((existNumSpecies + threadsPerBlock.x - 1) / threadsPerBlock.x);

    calculateFirstMomentOfOneSpecies_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(firstMomentOfOneSpecies.data()), 
        thrust::raw_pointer_cast(particlesSpecies.data()), 
        existNumSpecies, 
        mPIInfo.localNx, mPIInfo.buffer, 
        mPIInfo.localSizeX, 
        mPIInfo.xminForProcs, mPIInfo.xmaxForProcs
    );
    hipDeviceSynchronize();
}


__global__ void calculateSecondMomentOfOneSpecies_kernel(
    SecondMoment* secondMomentOfOneSpecies, 
    const Particle* particlesSpecies, 
    const unsigned long long existNumSpecies, 
    const int localNx, const int buffer, 
    const int localSizeX,
    const double xminForProcs, const double xmaxForProcs
)
{
    unsigned long long i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < existNumSpecies) {
    
        double cx1, cx2; 
        int xIndex1, xIndex2;
        double xOverDx;
        double cy1, cy2; 
        int yIndex1, yIndex2;
        double yOverDy;
        double vx, vy, vz;

        xOverDx = (particlesSpecies[i].x - xminForProcs + buffer * PIC2DConst::device_dx) / PIC2DConst::device_dx;
        yOverDy = (particlesSpecies[i].y - PIC2DConst::device_ymin) / PIC2DConst::device_dy;

        xIndex1 = floor(xOverDx);
        xIndex2 = xIndex1 + 1;
        xIndex2 = (xIndex2 == localSizeX) ? 0 : xIndex2;
        yIndex1 = floor(yOverDy);
        yIndex2 = yIndex1 + 1;
        yIndex2 = (yIndex2 == PIC2DConst::device_ny) ? 0 : yIndex2;
        
        if (xIndex1 < 0 || xIndex1 >= localSizeX) printf("x = %f, index = %d, ERROR\n", particlesSpecies[i].x, xIndex1); 
        if (yIndex1 < 0 || yIndex1 >= PIC2DConst::device_ny) printf("y = %f, index = %d, ERROR\n", particlesSpecies[i].y, yIndex1);

        cx1 = xOverDx - xIndex1;
        cx2 = 1.0 - cx1;
        cy1 = yOverDy - yIndex1;
        cy2 = 1.0 - cy1;

        vx = particlesSpecies[i].vx / particlesSpecies[i].gamma;
        vy = particlesSpecies[i].vy / particlesSpecies[i].gamma;
        vz = particlesSpecies[i].vz / particlesSpecies[i].gamma;

        unsigned long long index11 = yIndex1 + PIC2DConst::device_ny * xIndex1; 
        unsigned long long index12 = yIndex2 + PIC2DConst::device_ny * xIndex1; 
        unsigned long long index21 = yIndex1 + PIC2DConst::device_ny * xIndex2; 
        unsigned long long index22 = yIndex2 + PIC2DConst::device_ny * xIndex2; 

        atomicAdd(&(secondMomentOfOneSpecies[index11].xx), vx * vx * cx2 * cy2);
        atomicAdd(&(secondMomentOfOneSpecies[index12].xx), vx * vx * cx2 * cy1);
        atomicAdd(&(secondMomentOfOneSpecies[index21].xx), vx * vx * cx1 * cy2);
        atomicAdd(&(secondMomentOfOneSpecies[index22].xx), vx * vx * cx1 * cy1);

        atomicAdd(&(secondMomentOfOneSpecies[index11].yy), vy * vy * cx2 * cy2);
        atomicAdd(&(secondMomentOfOneSpecies[index12].yy), vy * vy * cx2 * cy1);
        atomicAdd(&(secondMomentOfOneSpecies[index21].yy), vy * vy * cx1 * cy2);
        atomicAdd(&(secondMomentOfOneSpecies[index22].yy), vy * vy * cx1 * cy1);

        atomicAdd(&(secondMomentOfOneSpecies[index11].zz), vz * vz * cx2 * cy2);
        atomicAdd(&(secondMomentOfOneSpecies[index12].zz), vz * vz * cx2 * cy1);
        atomicAdd(&(secondMomentOfOneSpecies[index21].zz), vz * vz * cx1 * cy2);
        atomicAdd(&(secondMomentOfOneSpecies[index22].zz), vz * vz * cx1 * cy1);

        atomicAdd(&(secondMomentOfOneSpecies[index11].xy), vx * vy * cx2 * cy2);
        atomicAdd(&(secondMomentOfOneSpecies[index12].xy), vx * vy * cx2 * cy1);
        atomicAdd(&(secondMomentOfOneSpecies[index21].xy), vx * vy * cx1 * cy2);
        atomicAdd(&(secondMomentOfOneSpecies[index22].xy), vx * vy * cx1 * cy1);

        atomicAdd(&(secondMomentOfOneSpecies[index11].xz), vx * vz * cx2 * cy2);
        atomicAdd(&(secondMomentOfOneSpecies[index12].xz), vx * vz * cx2 * cy1);
        atomicAdd(&(secondMomentOfOneSpecies[index21].xz), vx * vz * cx1 * cy2);
        atomicAdd(&(secondMomentOfOneSpecies[index22].xz), vx * vz * cx1 * cy1);

        atomicAdd(&(secondMomentOfOneSpecies[index11].yz), vy * vz * cx2 * cy2);
        atomicAdd(&(secondMomentOfOneSpecies[index12].yz), vy * vz * cx2 * cy1);
        atomicAdd(&(secondMomentOfOneSpecies[index21].yz), vy * vz * cx1 * cy2);
        atomicAdd(&(secondMomentOfOneSpecies[index22].yz), vy * vz * cx1 * cy1);
    }
};


void MomentCalculator::calculateSecondMomentOfOneSpecies(
    thrust::device_vector<SecondMoment>& secondMomentOfOneSpecies, 
    const thrust::device_vector<Particle>& particlesSpecies, 
    unsigned long long existNumSpecies
)
{
    resetSecondMomentOfOneSpecies(secondMomentOfOneSpecies);

    dim3 threadsPerBlock(256);
    dim3 blocksPerGrid((existNumSpecies + threadsPerBlock.x - 1) / threadsPerBlock.x);

    calculateSecondMomentOfOneSpecies_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(secondMomentOfOneSpecies.data()), 
        thrust::raw_pointer_cast(particlesSpecies.data()), 
        existNumSpecies, 
        mPIInfo.localNx, mPIInfo.buffer, 
        mPIInfo.localSizeX, 
        mPIInfo.xminForProcs, mPIInfo.xmaxForProcs
    );
    hipDeviceSynchronize();
}



