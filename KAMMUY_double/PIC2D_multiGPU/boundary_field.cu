#include "hip/hip_runtime.h"
#include "boundary.hpp"



//////////

void BoundaryPIC::periodicBoundaryB_x(
    thrust::device_vector<MagneticField>& B
)
{
    MPI_Barrier(MPI_COMM_WORLD); 

    PIC2DMPI::sendrecv_magneticField_x(
        B, 
        sendMagneticFieldLeft, sendMagneticFieldRight, 
        recvMagneticFieldLeft, recvMagneticFieldRight, 
        mPIInfo
    ); 
}


void freeBoundaryB_x(
    thrust::device_vector<MagneticField>& B
)
{
    std::cout << "Not writtern yet. Finish your calculation now!" << std::endl;
}


__global__ void freeBoundaryB_y_kernel(
    MagneticField* B, 
    int localSizeX, 
    int buffer
)
{
    unsigned long long i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < localSizeX) {
        B[0 + PIC2DConst::device_ny * i] = B[1 + PIC2DConst::device_ny * i];
        B[PIC2DConst::device_ny - 1 + PIC2DConst::device_ny * i] = B[PIC2DConst::device_ny - 2 + PIC2DConst::device_ny * i];
    }
}


void BoundaryPIC::freeBoundaryB_y(
    thrust::device_vector<MagneticField>& B
)
{
    MPI_Barrier(MPI_COMM_WORLD); 

    int threadsPerBlock = 256;
    int blocksPerGrid = (mPIInfo.localSizeX + threadsPerBlock - 1) / threadsPerBlock;

    freeBoundaryB_y_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(B.data()), 
        mPIInfo.localSizeX, 
        mPIInfo.buffer
    );
    hipDeviceSynchronize();
}



//////////////////////////////////////////////////

void BoundaryPIC::periodicBoundaryE_x(
    thrust::device_vector<ElectricField>& E
)
{
    MPI_Barrier(MPI_COMM_WORLD); 

    PIC2DMPI::sendrecv_electricField_x(
        E, 
        sendElectricFieldLeft, sendElectricFieldRight, 
        recvElectricFieldLeft, recvElectricFieldRight, 
        mPIInfo
    ); 
}


void freeBoundaryE_x(
    thrust::device_vector<ElectricField>& E
)
{
    std::cout << "Not writtern yet. Finish your calculation now!" << std::endl;
}


__global__ void freeBoundaryE_y_kernel(
    ElectricField* E, 
    int localSizeX, 
    int buffer
)
{
    unsigned long long i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < localSizeX) {
        E[0 + PIC2DConst::device_ny * i] = E[1 + PIC2DConst::device_ny * i];
        E[PIC2DConst::device_ny - 1 + PIC2DConst::device_ny * i] = E[PIC2DConst::device_ny - 2 + PIC2DConst::device_ny * i];
    }
}

void BoundaryPIC::freeBoundaryE_y(
    thrust::device_vector<ElectricField>& E
)
{
    MPI_Barrier(MPI_COMM_WORLD); 

    int threadsPerBlock = 256;
    int blocksPerGrid = (mPIInfo.localSizeX + threadsPerBlock - 1) / threadsPerBlock;

    freeBoundaryE_y_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(E.data()), 
        mPIInfo.localSizeX, 
        mPIInfo.buffer
    );
    hipDeviceSynchronize();
}

//////////////////////////////////////////////////

void BoundaryPIC::periodicBoundaryCurrent_x(
    thrust::device_vector<CurrentField>& current
)
{
    MPI_Barrier(MPI_COMM_WORLD); 

    PIC2DMPI::sendrecv_currentField_x(
        current, 
        sendCurrentFieldLeft, sendCurrentFieldRight, 
        recvCurrentFieldLeft, recvCurrentFieldRight, 
        mPIInfo
    ); 
}


void freeBoundaryCurrent_x(
    thrust::device_vector<CurrentField>& current
)
{
    std::cout << "Not writtern yet. Finish your calculation now!" << std::endl;
}


__global__ void freeBoundaryCurrent_y_kernel(
    CurrentField* current,  
    int localSizeX, 
    int buffer
)
{
    unsigned long long i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < localSizeX) {
        current[0 + PIC2DConst::device_ny * i] = current[1 + PIC2DConst::device_ny * i];
        current[PIC2DConst::device_ny - 1 + PIC2DConst::device_ny * i] = current[PIC2DConst::device_ny - 2 + PIC2DConst::device_ny * i];
    }
}


void BoundaryPIC::freeBoundaryCurrent_y(
    thrust::device_vector<CurrentField>& current
)
{
    MPI_Barrier(MPI_COMM_WORLD); 
    
    int threadsPerBlock = 256;
    int blocksPerGrid = (mPIInfo.localSizeX + threadsPerBlock - 1) / threadsPerBlock;

    freeBoundaryCurrent_y_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(current.data()), 
        mPIInfo.localSizeX,  
        mPIInfo.buffer
    );
    hipDeviceSynchronize();
}



