#include "hip/hip_runtime.h"
#include "main_mrx_reload_const.hpp"


void IdealMHD2D::initializeU()
{
}


void PIC2D::initialize()
{
}


int main(int argc, char** argv)
{
    MPI_Init(&argc, &argv);

    int rank = 0, procs = 0;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &procs);

    PIC2DMPI::MPIInfo mPIInfoPIC;
    int mpiBufNumParticles = 10000000; 
    PIC2DMPI::setupInfo(mPIInfoPIC, bufferPIC, mpiBufNumParticles);
    IdealMHD2DMPI::MPIInfo mPIInfoMHD;
    IdealMHD2DMPI::setupInfo(mPIInfoMHD, bufferMHD);
    Interface2DMPI::MPIInfo mPIInfoInterface; 
    Interface2DMPI::setupInfo(mPIInfoInterface); 

    if (mPIInfoPIC.rank == 0) {
        std::cout   << mPIInfoPIC.gridX << std::endl;
        mpifile_PIC << mPIInfoPIC.gridX << std::endl;
    }
    if (mPIInfoMHD.rank == 0) {
        std::cout   << mPIInfoMHD.gridX << std::endl;
        mpifile_MHD << mPIInfoMHD.gridX << std::endl;
    }
    if (mPIInfoInterface.rank == 0) {
        std::cout   << mPIInfoInterface.gridX << std::endl;
        mpifile_Interface << mPIInfoInterface.gridX << std::endl;
    }

    hipSetDevice(mPIInfoPIC.rank);

    PIC2DConst::initializeDeviceConstants();
    IdealMHD2DConst::initializeDeviceConstants();
    Interface2DConst::initializeDeviceConstants();

    mPIInfoPIC.existNumIonPerProcs      = static_cast<unsigned long long>(PIC2DConst::totalNumIon / mPIInfoPIC.procs);
    mPIInfoPIC.existNumElectronPerProcs = static_cast<unsigned long long>(PIC2DConst::totalNumElectron / mPIInfoPIC.procs);
    mPIInfoPIC.totalNumIonPerProcs      = static_cast<unsigned long long>(mPIInfoPIC.existNumIonPerProcs * 2.0);
    mPIInfoPIC.totalNumElectronPerProcs = static_cast<unsigned long long>(mPIInfoPIC.existNumElectronPerProcs * 2.0);

    mPIInfoPIC.xminForProcs = PIC2DConst::xmin
                            + (PIC2DConst::xmax - PIC2DConst::xmin) / mPIInfoPIC.gridX
                            * mPIInfoPIC.localGridX;
    mPIInfoPIC.xmaxForProcs = PIC2DConst::xmin
                            + (PIC2DConst::xmax - PIC2DConst::xmin) / mPIInfoPIC.gridX
                            * (mPIInfoPIC.localGridX + 1);


    thrust::host_vector<double> host_interlockingFunctionY(mPIInfoPIC.localSizeX * PIC2DConst::ny, 0.0);
    int bufferForInterlocking = 0;  
    for (int i = 0; i < mPIInfoPIC.localSizeX; i++) {
        for (int j = 0; j < PIC2DConst::ny / 2; j++) {
            if (j < bufferForInterlocking) {
                host_interlockingFunctionY[j + i * PIC2DConst::ny] = 1.0;
            } else if (bufferForInterlocking <= j && j < Interface2DConst::deltaForInterlockingFunction + bufferForInterlocking) {
                host_interlockingFunctionY[j + i * PIC2DConst::ny] = 0.5 * (1.0 + cos(Interface2DConst::PI * (j - bufferForInterlocking) / Interface2DConst::deltaForInterlockingFunction));
            } else {
                host_interlockingFunctionY[j + i * PIC2DConst::ny] = 0.0;
            }
        }
    }
    for (int i = 0; i < mPIInfoPIC.localSizeX; i++) {
        for (int j = PIC2DConst::ny / 2; j < PIC2DConst::ny; j++) {
            host_interlockingFunctionY[j + i * PIC2DConst::ny] = host_interlockingFunctionY[PIC2DConst::ny - 1 - j + i * PIC2DConst::ny];
        }
    }

    IdealMHD2D idealMHD2D(mPIInfoMHD);
    PIC2D pIC2D(mPIInfoPIC); 
    InterfaceNoiseRemover2D interfaceNoiseRemover2D( 
        mPIInfoMHD, mPIInfoPIC
    );
    Interface2D interface2D(
        mPIInfoMHD, mPIInfoPIC, mPIInfoInterface, 
        Interface2DConst::indexOfInterfaceStartInMHD, 
        host_interlockingFunctionY, 
        interfaceNoiseRemover2D
    );
    BoundaryMHD& boundaryMHD = idealMHD2D.getBoundaryMHDRef(); 
    BoundaryPIC& boundaryPIC = pIC2D.getBoundaryPICRef(); 
    Projection& projection = idealMHD2D.getProjectionRef();
    

    if (mPIInfoPIC.rank == 0) {
        size_t free_mem = 0;
        size_t total_mem = 0;
        hipError_t status = hipMemGetInfo(&free_mem, &total_mem);

        std::cout << "Free memory: " << free_mem / (1024 * 1024) << " MB" << std::endl;
        std::cout << "Total memory: " << total_mem / (1024 * 1024) << " MB" << std::endl;

        std::cout << "exist number of partices is " 
                  << mPIInfoPIC.procs * (mPIInfoPIC.existNumIonPerProcs + mPIInfoPIC.existNumElectronPerProcs) 
                  << std::endl;
        std::cout << "exist number of partices + buffer particles is " 
                  << mPIInfoPIC.procs * (mPIInfoPIC.totalNumIonPerProcs + mPIInfoPIC.totalNumElectronPerProcs) 
                  << std::endl;
        
        std::cout << "PIC grid size is " 
                  << mPIInfoPIC.localSizeX << " X " << PIC2DConst::ny 
                  << std::endl;
        std::cout << "MHD grid size is " 
                  << mPIInfoMHD.localSizeX << " X " << IdealMHD2DConst::ny
                  << std::endl;
    }

    idealMHD2D.initializeU(); 
    pIC2D.initialize();


    std::cout << "Reload started" << std::endl;
    //ここから大幅に異なる
    thrust::host_vector<Particle>& host_particlesIon = pIC2D.getHostParticlesIonRef();
    thrust::host_vector<Particle>& host_particlesElectron = pIC2D.getHostParticlesElectronRef();
    thrust::host_vector<MagneticField>& host_B = pIC2D.getHostBRef();
    thrust::host_vector<ElectricField>& host_E = pIC2D.getHostERef();
    thrust::host_vector<ConservationParameter>& host_U = idealMHD2D.getHostURef();

    Reloader reloader(mPIInfoPIC, mPIInfoMHD);
    reloader.reloadPICData(
        host_particlesIon, host_particlesElectron, host_B, host_E, 
        directoryName, filenameWithoutStep, 
        reloadStep
    );
    reloader.reloadMHDData(
        host_U, 
        directoryName, filenameWithoutStep, 
        reloadStep
    );

    thrust::device_vector<Particle>& particlesIon = pIC2D.getParticlesIonRef();
    thrust::device_vector<Particle>& particlesElectron = pIC2D.getParticlesElectronRef();
    thrust::device_vector<MagneticField>& B = pIC2D.getBRef();
    thrust::device_vector<ElectricField>& E = pIC2D.getERef();
    thrust::device_vector<ConservationParameter>& U = idealMHD2D.getURef();

    particlesIon = host_particlesIon; 
    particlesElectron = host_particlesElectron; 
    B = host_B; 
    E = host_E; 
    U = host_U; 

    const int totalSubstep = int(round(sqrt(PIC2DConst::mRatio)) * Interface2DConst::gridSizeRatio);
    for (int step = reloadStep + 1; step < IdealMHD2DConst::totalStep + 1; step++) {
        MPI_Barrier(MPI_COMM_WORLD);

        if (mPIInfoPIC.rank == 0) {
            if (step % recordStep == 0) {
                std::cout << std::to_string(step) << " step done : total time is "
                        << std::setprecision(4) << step * totalSubstep * PIC2DConst::dt * PIC2DConst::omegaPe
                        << " [omega_pe * t]"
                        << std::endl;
            }
        }

        if (step % recordStep == 0) {
            logfile << std::setprecision(6) << IdealMHD2DConst::totalTime << std::endl;
            pIC2D.saveParticle(
                directoryName, filenameWithoutStep, step
            );
            pIC2D.saveFields(
                directoryName, filenameWithoutStep, step
            );
            pIC2D.saveZerothMoments(
                directoryName, filenameWithoutStep, step
            );
            pIC2D.saveFirstMoments(
                directoryName, filenameWithoutStep, step
            );
            pIC2D.saveSecondMoments(
                directoryName, filenameWithoutStep, step
            );
            idealMHD2D.save(
                directoryName, filenameWithoutStep + "_U", step
            );
        }

        double dtCommon = min(0.7 / PIC2DConst::c, 0.1 * 1.0 / PIC2DConst::omegaPe);
        PIC2DConst::dt = dtCommon;
        IdealMHD2DConst::dt = totalSubstep * dtCommon;
        IdealMHD2DConst::eta = 0.0 * pow(IdealMHD2DConst::dx, 2) / IdealMHD2DConst::dt; 
        IdealMHD2DConst::viscosity = 0.0 * pow(IdealMHD2DConst::dx, 2) / IdealMHD2DConst::dt; 
        hipMemcpyToSymbol(HIP_SYMBOL(PIC2DConst::device_dt), &PIC2DConst::dt, sizeof(double));
        hipMemcpyToSymbol(HIP_SYMBOL(IdealMHD2DConst::device_dt), &IdealMHD2DConst::dt, sizeof(double));
        hipMemcpyToSymbol(HIP_SYMBOL(IdealMHD2DConst::device_eta), &IdealMHD2DConst::eta, sizeof(double));
        hipMemcpyToSymbol(HIP_SYMBOL(IdealMHD2DConst::device_viscosity), &IdealMHD2DConst::viscosity, sizeof(double));


        // STEP1 : MHD step

        idealMHD2D.setPastU();
        thrust::device_vector<ConservationParameter>& UPast = idealMHD2D.getUPastRef();

        idealMHD2D.oneStepRK2_periodicXSymmetricY_predictor();

        thrust::device_vector<ConservationParameter>& UNext = idealMHD2D.getURef();

        // STEP2 : PIC step & send MHD to PIC

        interface2D.resetTimeAveragedPICParameters();

        //int sumUpCount = 0;  
        //pIC2D.calculateFullMoments();
        //thrust::device_vector<MagneticField>& B = pIC2D.getBRef();
        //thrust::device_vector<ZerothMoment>& zerothMomentIon = pIC2D.getZerothMomentIonRef(); 
        //thrust::device_vector<ZerothMoment>& zerothMomentElectron = pIC2D.getZerothMomentElectronRef(); 
        //thrust::device_vector<FirstMoment>& firstMomentIon = pIC2D.getFirstMomentIonRef(); 
        //thrust::device_vector<FirstMoment>& firstMomentElectron = pIC2D.getFirstMomentElectronRef(); 
        //thrust::device_vector<SecondMoment>& secondMomentIon = pIC2D.getSecondMomentIonRef(); 
        //thrust::device_vector<SecondMoment>& secondMomentElectron = pIC2D.getSecondMomentElectronRef(); 
        //interface2D.sumUpTimeAveragedPICParameters(
        //    B, 
        //    zerothMomentIon, zerothMomentElectron, 
        //    firstMomentIon, firstMomentElectron, 
        //    secondMomentIon, secondMomentElectron
        //);
        //sumUpCount += 1; 
        for (int substep = 1; substep <= totalSubstep; substep++) {

            double mixingRatio = 1.0 - static_cast<double>(substep) / static_cast<double>(totalSubstep);
            thrust::device_vector<ConservationParameter>& USub = interface2D.calculateAndGetSubU(UPast, UNext, mixingRatio);
            
            unsigned long long seedForReload; 
            seedForReload = substep + step * totalSubstep;
            pIC2D.oneStep_periodicXFreeY(
                interface2D, 
                USub, 
                seedForReload
            );

            //interface2D.sumUpTimeAveragedPICParameters(
            //    B, 
            //    zerothMomentIon, zerothMomentElectron, 
            //    firstMomentIon, firstMomentElectron, 
            //    secondMomentIon, secondMomentElectron
            //);
            //sumUpCount += 1; 
        }

        //interface2D.calculateTimeAveragedPICParameters(sumUpCount); 

        thrust::device_vector<MagneticField>& B = pIC2D.getBRef();
        thrust::device_vector<ZerothMoment>& zerothMomentIon = pIC2D.getZerothMomentIonRef(); 
        thrust::device_vector<ZerothMoment>& zerothMomentElectron = pIC2D.getZerothMomentElectronRef(); 
        thrust::device_vector<FirstMoment>& firstMomentIon = pIC2D.getFirstMomentIonRef(); 
        thrust::device_vector<FirstMoment>& firstMomentElectron = pIC2D.getFirstMomentElectronRef(); 
        thrust::device_vector<SecondMoment>& secondMomentIon = pIC2D.getSecondMomentIonRef(); 
        thrust::device_vector<SecondMoment>& secondMomentElectron = pIC2D.getSecondMomentElectronRef(); 
        interface2D.sumUpTimeAveragedPICParameters(
            B, 
            zerothMomentIon, zerothMomentElectron, 
            firstMomentIon, firstMomentElectron, 
            secondMomentIon, secondMomentElectron
        );

        interface2D.setParametersForPICtoMHD();

        // STEP3 : send PIC to MHD

        //interface2D.calculateUHalf(UPast, UNext); 
        //thrust::device_vector<ConservationParameter>& UHalf = interface2D.getUHalfRef();

        thrust::device_vector<ConservationParameter>& U = idealMHD2D.getURef();

        interface2D.sendPICtoMHD(U);
        boundaryMHD.periodicBoundaryX2nd_U(U);
        boundaryMHD.symmetricBoundaryY2nd_U(U);

        //idealMHD2D.oneStepRK2_periodicXSymmetricY_corrector(UHalf);
        
        if (step % 10 == 0) {
            projection.correctB(U); 
            boundaryMHD.periodicBoundaryX2nd_U(U);
            boundaryMHD.symmetricBoundaryY2nd_U(U);
            
            interfaceNoiseRemover2D.convolveU(U);
            boundaryMHD.periodicBoundaryX2nd_U(U);
            boundaryMHD.symmetricBoundaryY2nd_U(U);
        }

        //when crashed 
        if (idealMHD2D.checkCalculationIsCrashed()) {
            logfile << std::setprecision(6) << PIC2DConst::totalTime << std::endl;
            pIC2D.saveFields(
                directoryName, filenameWithoutStep, step
            );
            pIC2D.saveZerothMoments(
                directoryName, filenameWithoutStep, step
            );
            pIC2D.saveFirstMoments(
                directoryName, filenameWithoutStep, step
            );
            idealMHD2D.save(
                directoryName, filenameWithoutStep + "_U", step
            );
            std::cout << "Calculation stopped! : " << step << " steps" << std::endl;
            break;
        }

        if (mPIInfoMHD.rank == 0) {
            IdealMHD2DConst::totalTime += IdealMHD2DConst::dt;
        }   
    }

    if (mPIInfoMHD.rank == 0) {
        std::cout << "program was completed!" << std::endl;
    }

    MPI_Finalize();

    return 0;
}



