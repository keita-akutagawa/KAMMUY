#include "const.hpp"


using namespace IdealMHD2DConst;

void initializeDeviceConstants() {
    hipMemcpyToSymbol(HIP_SYMBOL(device_EPS_MHD), &EPS_MHD, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_PI_MHD), &PI_MHD, sizeof(double));

    hipMemcpyToSymbol(HIP_SYMBOL(device_dx_MHD), &dx_MHD, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_xmin_MHD), &xmin_MHD, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_xmax_MHD), &xmax_MHD, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_nx_MHD), &nx_MHD, sizeof(int));

    hipMemcpyToSymbol(HIP_SYMBOL(device_dy_MHD), &dy_MHD, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_ymin_MHD), &ymin_MHD, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_ymax_MHD), &ymax_MHD, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_ny_MHD), &ny_MHD, sizeof(int));

    hipMemcpyToSymbol(HIP_SYMBOL(device_CFL_MHD), &CFL_MHD, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_gamma_MHD), &gamma_MHD, sizeof(double));

    hipMemcpyToSymbol(HIP_SYMBOL(device_dt_MHD), &dt_MHD, sizeof(double));
}
