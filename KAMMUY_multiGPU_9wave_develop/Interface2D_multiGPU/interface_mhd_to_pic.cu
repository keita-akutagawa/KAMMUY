#include "hip/hip_runtime.h"
#include "interface.hpp"


__global__ void sendMHDtoPIC_magneticField_y_kernel(
    const double* interlockingFunctionY, 
    const ConservationParameter* U, 
    MagneticField* B, 
    int indexOfInterfaceStartInMHD, 
    int indexOfInterfaceStartInPIC, 
    int localSizeXPIC, int localSizeYPIC, 
    int localSizeXMHD, int localSizeYMHD, 
    int localSizeXInterface, int localSizeYInterface
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (0 < i && i < localSizeXInterface - 1 && 0 < j && j < localSizeYInterface - 1) {
        double bXPIC, bYPIC, bZPIC;
        double bXMHD, bYMHD, bZMHD;
        double bXInterface, bYInterface, bZInterface;

        int indexPIC = indexOfInterfaceStartInPIC + j + i * localSizeYPIC;
        int indexMHD = indexOfInterfaceStartInMHD + j + i * localSizeYMHD;

        //PICのグリッドにMHDを合わせる
        bXPIC = B[indexPIC].bX;
        bYPIC = B[indexPIC].bY;
        bZPIC = B[indexPIC].bZ;
        bXMHD = U[indexMHD].bX;
        bYMHD = U[indexMHD].bY;
        bZMHD = U[indexMHD].bZ;

        bXInterface = interlockingFunctionY[j] * bXMHD + (1.0 - interlockingFunctionY[j]) * bXPIC;
        bYInterface = interlockingFunctionY[j]     * bYMHD + (1.0 - interlockingFunctionY[j])     * bYPIC;
        bZInterface = interlockingFunctionY[j] * bZMHD + (1.0 - interlockingFunctionY[j]) * bZPIC;
        
        B[indexPIC].bX = bXInterface;
        B[indexPIC].bY = bYInterface;
        B[indexPIC].bZ = bZInterface;
    }
}

void Interface2D::sendMHDtoPIC_magneticField_y(
    const thrust::device_vector<ConservationParameter>& U, 
    thrust::device_vector<MagneticField>& B
)
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((localSizeXInterface + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (localSizeYInterface + threadsPerBlock.y - 1) / threadsPerBlock.y);

    sendMHDtoPIC_magneticField_y_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(interlockingFunctionY.data()),
        thrust::raw_pointer_cast(U.data()), 
        thrust::raw_pointer_cast(B.data()), 
        indexOfInterfaceStartInMHD, 
        indexOfInterfaceStartInPIC, 
        localSizeXPIC, localSizeYPIC,  
        localSizeXMHD, localSizeYMHD, 
        localSizeXInterface, localSizeYInterface
    );
    hipDeviceSynchronize();
}


__global__ void sendMHDtoPIC_electricField_y_kernel(
    const double* interlockingFunctionY, 
    const ConservationParameter* U, 
    ElectricField* E, 
    int indexOfInterfaceStartInMHD, 
    int indexOfInterfaceStartInPIC, 
    int localSizeXPIC, int localSizeYPIC, 
    int localSizeXMHD, int localSizeYMHD, 
    int localSizeXInterface, int localSizeYInterface
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (0 < i && i < localSizeXInterface - 1 && 0 < j && j < localSizeYInterface - 1) {
        double eXPIC, eYPIC, eZPIC;
        double eXMHD, eYMHD, eZMHD;
        double rho, u, v, w;
        double bXMHD, bYMHD, bZMHD;
        double eXInterface, eYInterface, eZInterface;

        int indexPIC = indexOfInterfaceStartInPIC + j + i * localSizeYPIC;
        int indexMHD = indexOfInterfaceStartInMHD + j + i * localSizeYMHD;

        //PICのグリッドにMHDを合わせる
        eXPIC = E[indexPIC].eX;
        eYPIC = E[indexPIC].eY;
        eZPIC = E[indexPIC].eZ;

        rho = max(U[indexMHD].rho, IdealMHD2DConst::device_rho0 * 0.1);
        u = U[indexMHD].rhoU / (rho + IdealMHD2DConst::device_EPS);
        v = U[indexMHD].rhoV / (rho + IdealMHD2DConst::device_EPS);
        w = U[indexMHD].rhoW / (rho + IdealMHD2DConst::device_EPS); 
        bXMHD = U[indexMHD].bX; 
        bYMHD = U[indexMHD].bY; 
        bZMHD = U[indexMHD].bZ;
        eXMHD = -(v * bZMHD - w * bYMHD);
        eYMHD = -(w * bXMHD - u * bZMHD);
        eZMHD = -(u * bYMHD - v * bXMHD);

        eXInterface = interlockingFunctionY[j] * eXMHD + (1.0 - interlockingFunctionY[j]) * eXPIC;
        eYInterface = interlockingFunctionY[j] * eYMHD + (1.0 - interlockingFunctionY[j]) * eYPIC;
        eZInterface = interlockingFunctionY[j] * eZMHD + (1.0 - interlockingFunctionY[j]) * eZPIC;
         
        E[indexPIC].eX = eXInterface;
        E[indexPIC].eY = eYInterface;
        E[indexPIC].eZ = eZInterface;
    }
}

void Interface2D::sendMHDtoPIC_electricField_y(
    const thrust::device_vector<ConservationParameter>& U, 
    thrust::device_vector<ElectricField>& E
)
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((localSizeXInterface + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (localSizeYInterface + threadsPerBlock.y - 1) / threadsPerBlock.y);

    sendMHDtoPIC_electricField_y_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(interlockingFunctionY.data()), 
        thrust::raw_pointer_cast(U.data()), 
        thrust::raw_pointer_cast(E.data()), 
        indexOfInterfaceStartInMHD, 
        indexOfInterfaceStartInPIC, 
        localSizeXPIC, localSizeYPIC,  
        localSizeXMHD, localSizeYMHD, 
        localSizeXInterface, localSizeYInterface
    );
    hipDeviceSynchronize();
}


__global__ void sendMHDtoPIC_currentField_y_kernel(
    const double* interlockingFunctionY, 
    const ConservationParameter* U, 
    CurrentField* current, 
    int indexOfInterfaceStartInMHD, 
    int indexOfInterfaceStartInPIC, 
    int localSizeXPIC, int localSizeYPIC, 
    int localSizeXMHD, int localSizeYMHD, 
    int localSizeXInterface, int localSizeYInterface
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (0 < i && i < localSizeXInterface - 1 && 0 < j && j < localSizeYInterface - 1) {
        double jXPIC, jYPIC, jZPIC;
        double jXMHD, jYMHD, jZMHD;
        double jXInterface, jYInterface, jZInterface;
        double dx = IdealMHD2DConst::device_dx, dy = IdealMHD2DConst::device_dy;

        int indexPIC = indexOfInterfaceStartInPIC + j + i * localSizeYPIC;
        int indexMHD = indexOfInterfaceStartInMHD + j + i * localSizeYMHD;

        //PICのグリッドにMHDを合わせる
        jXPIC = current[indexPIC].jX;
        jYPIC = current[indexPIC].jY;
        jZPIC = current[indexPIC].jZ;
        jXMHD = (U[indexMHD + 1].bZ - U[indexMHD - 1].bZ) / (2.0 * dy);
        jYMHD = -(U[indexMHD + localSizeYMHD].bZ - U[indexMHD - localSizeYMHD].bZ) / (2.0 * dx);
        jZMHD = (U[indexMHD + localSizeYMHD].bY - U[indexMHD - localSizeYMHD].bY) / (2.0 * dx) - (U[indexMHD + 1].bX - U[indexMHD - 1].bX) / (2.0 * dy) ;

        
        jXInterface = interlockingFunctionY[j] * jXMHD + (1.0 - interlockingFunctionY[j]) * jXPIC;
        jYInterface = interlockingFunctionY[j] * jYMHD + (1.0 - interlockingFunctionY[j]) * jYPIC;
        jZInterface = interlockingFunctionY[j] * jZMHD + (1.0 - interlockingFunctionY[j]) * jZPIC;
        
        current[indexPIC].jX = jXInterface;
        current[indexPIC].jY = jYInterface;
        current[indexPIC].jZ = jZInterface;
    }
}

void Interface2D::sendMHDtoPIC_currentField_y(
    const thrust::device_vector<ConservationParameter>& U, 
    thrust::device_vector<CurrentField>& current
)
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((localSizeXInterface + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (localSizeYInterface + threadsPerBlock.y - 1) / threadsPerBlock.y);

    sendMHDtoPIC_currentField_y_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(interlockingFunctionY.data()), 
        thrust::raw_pointer_cast(U.data()), 
        thrust::raw_pointer_cast(current.data()), 
        indexOfInterfaceStartInMHD, 
        indexOfInterfaceStartInPIC, 
        localSizeXPIC, localSizeYPIC,  
        localSizeXMHD, localSizeYMHD, 
        localSizeXInterface, localSizeYInterface
    );
    hipDeviceSynchronize();
}


__global__ void sendMHDtoPIC_particle_y_kernel(
    const double* interlockingFunctionY, 
    const ZerothMoment* zerothMomentIon, 
    const ZerothMoment* zerothMomentElectron, 
    const FirstMoment* firstMomentIon, 
    const FirstMoment* firstMomentElectron, 
    const ConservationParameter* U, 
    ReloadParticlesData* reloadParticlesDataIon, 
    ReloadParticlesData* reloadParticlesDataElectron, 
    int indexOfInterfaceStartInMHD, 
    int indexOfInterfaceStartInPIC, 
    int localSizeXPIC, int localSizeYPIC, 
    int localSizeXMHD, int localSizeYMHD, 
    int localSizeXInterface, int localSizeYInterface
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (0 < i && i < localSizeXInterface - 1 && 0 < j && j < localSizeYInterface - 1) {
        int indexForReload = j + i * localSizeYInterface;  
        int indexPIC = indexOfInterfaceStartInPIC + j + i * localSizeYPIC;
        int indexMHD = indexOfInterfaceStartInMHD + j + i * localSizeYMHD;
        double rhoMHD, uMHD, vMHD, wMHD, bXMHD, bYMHD, bZMHD, eMHD, pMHD;
        double jXMHD, jYMHD, jZMHD, niMHD, neMHD, tiMHD, teMHD;
        double rhoPIC, uPIC, vPIC, wPIC;
        double jXPIC, jYPIC, jZPIC, niPIC, nePIC, vThiPIC, vThePIC;
        double dx = IdealMHD2DConst::device_dx, dy = IdealMHD2DConst::device_dy;
        double mIon = PIC2DConst::device_mIon, mElectron = PIC2DConst::device_mElectron;
        double qIon = PIC2DConst::device_qIon, qElectron = PIC2DConst::device_qElectron;

        //整数格子点上で計算する。リロードに使う。
        rhoMHD = max(U[indexMHD].rho, IdealMHD2DConst::device_rho0 * 0.1); 
        uMHD   = U[indexMHD].rhoU / (rhoMHD + IdealMHD2DConst::device_EPS);
        vMHD   = U[indexMHD].rhoV / (rhoMHD + IdealMHD2DConst::device_EPS);
        wMHD   = U[indexMHD].rhoW / (rhoMHD + IdealMHD2DConst::device_EPS); 
        bXMHD  = U[indexMHD].bX;
        bYMHD  = U[indexMHD].bY;
        bZMHD  = U[indexMHD].bZ;
        eMHD   = U[indexMHD].e;
        pMHD   = (IdealMHD2DConst::device_gamma - 1.0)
               * (eMHD - 0.5 * rhoMHD * (uMHD * uMHD + vMHD * vMHD + wMHD * wMHD)
               - 0.5 * (bXMHD * bXMHD + bYMHD * bYMHD + bZMHD * bZMHD));
        pMHD   = max(pMHD, IdealMHD2DConst::device_p0 * 0.1);
        jXMHD  = (U[indexMHD + 1].bZ - U[indexMHD - 1].bZ) / (2.0 * dy);
        jYMHD  = -(U[indexMHD + localSizeYMHD].bZ - U[indexMHD - localSizeYMHD].bZ) / (2.0 * dx);
        jZMHD = (U[indexMHD + localSizeYMHD].bY - U[indexMHD - localSizeYMHD].bY) / (2.0 * dx) - (U[indexMHD + 1].bX - U[indexMHD - 1].bX) / (2.0 * dy) ;

        niMHD = rhoMHD / (mIon + mElectron);
        neMHD = niMHD;
        tiMHD = pMHD / 2.0 / niMHD;
        teMHD = pMHD / 2.0 / neMHD;

        rhoPIC =  max(mIon * zerothMomentIon[indexPIC].n + mElectron * zerothMomentElectron[indexPIC].n, IdealMHD2DConst::device_rho0 * 0.1);
        uPIC   = (mIon * firstMomentIon[indexPIC].x  + mElectron * firstMomentElectron[indexPIC].x) / (rhoPIC + PIC2DConst::device_EPS);
        vPIC   = (mIon * firstMomentIon[indexPIC].y  + mElectron * firstMomentElectron[indexPIC].y) / (rhoPIC + PIC2DConst::device_EPS);
        wPIC   = (mIon * firstMomentIon[indexPIC].z  + mElectron * firstMomentElectron[indexPIC].z) / (rhoPIC + PIC2DConst::device_EPS);
        jXPIC  = qIon  * firstMomentIon[indexPIC].x  + qElectron * firstMomentElectron[indexPIC].x;
        jYPIC  = qIon  * firstMomentIon[indexPIC].y  + qElectron * firstMomentElectron[indexPIC].y;
        jZPIC  = qIon  * firstMomentIon[indexPIC].z  + qElectron * firstMomentElectron[indexPIC].z;

        rhoPIC = interlockingFunctionY[j] * rhoMHD + (1.0 - interlockingFunctionY[j]) * rhoPIC;
        uPIC   = interlockingFunctionY[j] * uMHD   + (1.0 - interlockingFunctionY[j]) * uPIC;
        vPIC   = interlockingFunctionY[j] * vMHD   + (1.0 - interlockingFunctionY[j]) * vPIC;
        wPIC   = interlockingFunctionY[j] * wMHD   + (1.0 - interlockingFunctionY[j]) * wPIC;
        jXPIC  = interlockingFunctionY[j] * jXMHD  + (1.0 - interlockingFunctionY[j]) * jXPIC;
        jYPIC  = interlockingFunctionY[j] * jYMHD  + (1.0 - interlockingFunctionY[j]) * jYPIC;
        jZPIC  = interlockingFunctionY[j] * jZMHD  + (1.0 - interlockingFunctionY[j]) * jZPIC;

        niPIC   = rhoPIC / (mIon + mElectron);
        nePIC   = niPIC;
        vThiPIC = sqrt(2.0 * tiMHD / mIon);
        vThePIC = sqrt(2.0 * teMHD / mElectron);


        reloadParticlesDataIon     [indexForReload].numAndIndex = max(static_cast<unsigned long long>(round(niPIC)), static_cast<unsigned long long>(1));
        reloadParticlesDataElectron[indexForReload].numAndIndex = max(static_cast<unsigned long long>(round(nePIC)), static_cast<unsigned long long>(1));
        reloadParticlesDataIon     [indexForReload].u              = uPIC;
        reloadParticlesDataIon     [indexForReload].v              = vPIC;
        reloadParticlesDataIon     [indexForReload].w              = wPIC;
        reloadParticlesDataElectron[indexForReload].u              = uPIC - jXPIC / max(round(nePIC), 1.0) / abs(qElectron);
        reloadParticlesDataElectron[indexForReload].v              = vPIC - jYPIC / max(round(nePIC), 1.0) / abs(qElectron);
        reloadParticlesDataElectron[indexForReload].w              = wPIC - jZPIC / max(round(nePIC), 1.0) / abs(qElectron);
        reloadParticlesDataIon     [indexForReload].vth            = vThiPIC;
        reloadParticlesDataElectron[indexForReload].vth            = vThePIC;

        if (j == 1) {
            reloadParticlesDataIon[indexForReload - 1]      = reloadParticlesDataIon[indexForReload];
            reloadParticlesDataElectron[indexForReload - 1] = reloadParticlesDataElectron[indexForReload];
        }
        if (j == localSizeYInterface - 2) {
            reloadParticlesDataIon[indexForReload + 1]      = reloadParticlesDataIon[indexForReload];
            reloadParticlesDataElectron[indexForReload + 1] = reloadParticlesDataElectron[indexForReload];
        }
    }
}


__global__ void deleteParticles_kernel(
    const double* interlockingFunctionY, 
    Particle* particlesSpecies, 
    const int indexOfInterfaceStartInPIC, 
    const unsigned long long existNumSpecies, 
    int seed, 
    const float xminForProcs, const float xmaxForProcs, 
    const int buffer, 
    int localSizeXInterface, int localSizeYInterface
)
{
    unsigned long long i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < existNumSpecies) {
        float x = particlesSpecies[i].x;
        float y = particlesSpecies[i].y;
        float deleteXMin = xminForProcs - buffer * PIC2DConst::device_dx;
        float deleteXMax = xmaxForProcs + buffer * PIC2DConst::device_dx;
        float deleteYMin = (indexOfInterfaceStartInPIC) * PIC2DConst::device_dy;
        float deleteYMax = (indexOfInterfaceStartInPIC + localSizeYInterface) * PIC2DConst::device_dy;

        if (deleteXMin < x && x < deleteXMax && deleteYMin < y && y < deleteYMax) {
            int j = floorf(y - deleteYMin);
            hiprandState state; 
            hiprand_init(seed, i, 0, &state);
            float randomValue = hiprand_uniform(&state);
            if (randomValue < interlockingFunctionY[j]) {
                particlesSpecies[i].isExist = false;
            }
        }
    }
}


void Interface2D::deleteParticlesSpecies(
    thrust::device_vector<Particle>& particlesSpecies, 
    unsigned long long& existNumSpeciesPerProcs, 
    int seed
)
{

    dim3 threadsPerBlock(256);
    dim3 blocksPerGrid((existNumSpeciesPerProcs + threadsPerBlock.x - 1) / threadsPerBlock.x);
    
    deleteParticles_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(interlockingFunctionY.data()), 
        thrust::raw_pointer_cast(particlesSpecies.data()),
        indexOfInterfaceStartInPIC, 
        existNumSpeciesPerProcs, 
        seed, 
        mPIInfoPIC.xminForProcs, mPIInfoPIC.xmaxForProcs, 
        mPIInfoPIC.buffer, 
        localSizeXInterface, localSizeYInterface
    );
    hipDeviceSynchronize();

    auto partitionEnd = thrust::partition(
        particlesSpecies.begin(), particlesSpecies.end(), 
        [] __device__ (const Particle& p) { return p.isExist; }
    );
    hipDeviceSynchronize();

    existNumSpeciesPerProcs = thrust::distance(particlesSpecies.begin(), partitionEnd);
}


__global__ void reloadParticlesSpecies_kernel(
    const double* interlockingFunctionY, 
    const ReloadParticlesData* reloadParticlesDataSpecies, 
    const Particle* reloadParticlesSourceSpecies, 
    unsigned long long reloadParticlesTotalNumSpecies, 
    Particle* particlesSpecies, 
    int indexOfInterfaceStartInPIC, 
    unsigned long long* particlesNumCounter, 
    int seed, 
    const float xminForProcs, const float xmaxForProcs, 
    const float yminForProcs, const float ymaxForProcs, 
    int buffer, 
    int localSizeXInterface, int localSizeYInterface
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < localSizeXInterface && j < localSizeYInterface) {
        int index = j + i * localSizeYInterface; 
        float u = reloadParticlesDataSpecies[index].u;
        float v = reloadParticlesDataSpecies[index].v;
        float w = reloadParticlesDataSpecies[index].w;
        float vth = reloadParticlesDataSpecies[index].vth;
        Particle particleSource, particleReload;
        float x, y, z, vx, vy, vz, gamma;

        hiprandState stateReloadIndex, stateReload; 
        hiprand_init(seed, i * j, 0, &stateReloadIndex);
        unsigned long long restartParticlesIndexSpecies = static_cast<unsigned long long>(hiprand_uniform(&stateReloadIndex) * reloadParticlesTotalNumSpecies);

        for (unsigned long long k = 0; k < reloadParticlesDataSpecies[index].numAndIndex; k++) {
            hiprand_init(seed + i * j, k, 0, &stateReload);
            float randomValue = hiprand_uniform(&stateReload);

            if (randomValue < interlockingFunctionY[j]) {
                particleSource = reloadParticlesSourceSpecies[(restartParticlesIndexSpecies + k) % reloadParticlesTotalNumSpecies];

                x = particleSource.x; x = (x + i) * PIC2DConst::device_dx + (xminForProcs - buffer * PIC2DConst::device_dx);
                y = particleSource.y; y = (y + indexOfInterfaceStartInPIC + j) * PIC2DConst::device_dy;
                z = particleSource.z;
                
                vx = particleSource.vx; vx = u + vx * vth;
                vy = particleSource.vy; vy = v + vy * vth;
                vz = particleSource.vz; vz = w + vz * vth;
                if (1.0f - (vx * vx + vy * vy + vz * vz) / pow(PIC2DConst::device_c, 2) < 0.0f){
                    float normalizedVelocity = sqrt(vx * vx + vy * vy + vz * vz);
                    vx = vx / normalizedVelocity * 0.9f * PIC2DConst::device_c;
                    vy = vy / normalizedVelocity * 0.9f * PIC2DConst::device_c;
                    vz = vz / normalizedVelocity * 0.9f * PIC2DConst::device_c;
                };
                gamma = 1.0f / sqrt(1.0f - (vx * vx + vy * vy + vz * vz) / pow(PIC2DConst::device_c, 2));

                particleReload.x = x; particleReload.y = y; particleReload.z = z;
                particleReload.vx = vx * gamma; particleReload.vy = vy * gamma, particleReload.vz = vz * gamma; 
                particleReload.gamma = gamma;
                particleReload.isExist = true;

                unsigned long long loadIndex = atomicAdd(&(particlesNumCounter[0]), 1);
                particlesSpecies[loadIndex] = particleReload;
            } 
        }
    }
}


void Interface2D::reloadParticlesSpecies(
    thrust::device_vector<Particle>& particlesSpecies, 
    thrust::device_vector<ReloadParticlesData>& reloadParticlesDataSpecies, 
    thrust::device_vector<Particle>& reloadParticlesSourceSpecies, 
    unsigned long long& existNumSpeciesPerProcs, 
    int seed 
)
{
    thrust::device_vector<unsigned long long> particlesNumCounter(1, 0);
    particlesNumCounter[0] = existNumSpeciesPerProcs;

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((localSizeXInterface + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (localSizeYInterface + threadsPerBlock.y - 1) / threadsPerBlock.y);

    reloadParticlesSpecies_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(interlockingFunctionY.data()), 
        thrust::raw_pointer_cast(reloadParticlesDataSpecies.data()), 
        thrust::raw_pointer_cast(reloadParticlesSourceSpecies.data()), 
        Interface2DConst::reloadParticlesTotalNum, 
        thrust::raw_pointer_cast(particlesSpecies.data()), 
        indexOfInterfaceStartInPIC, 
        thrust::raw_pointer_cast(particlesNumCounter.data()), 
        seed, 
        mPIInfoPIC.xminForProcs, mPIInfoPIC.xmaxForProcs, 
        mPIInfoPIC.yminForProcs, mPIInfoPIC.ymaxForProcs, 
        mPIInfoPIC.buffer, 
        localSizeXInterface, localSizeYInterface
    );
    hipDeviceSynchronize();

    existNumSpeciesPerProcs = particlesNumCounter[0];
}


void Interface2D::sendMHDtoPIC_particle(
    const thrust::device_vector<ConservationParameter>& U, 
    thrust::device_vector<Particle>& particlesIon, 
    thrust::device_vector<Particle>& particlesElectron, 
    int seed
)
{
    setMoments(particlesIon, particlesElectron); 
    boundaryPIC.periodicBoundaryZerothMoment_x(zerothMomentIon); 
    boundaryPIC.freeBoundaryZerothMoment_y(zerothMomentIon); 
    boundaryPIC.periodicBoundaryZerothMoment_x(zerothMomentElectron); 
    boundaryPIC.freeBoundaryZerothMoment_y(zerothMomentElectron); 
    boundaryPIC.periodicBoundaryFirstMoment_x(firstMomentIon); 
    boundaryPIC.freeBoundaryFirstMoment_y(firstMomentIon); 
    boundaryPIC.periodicBoundaryFirstMoment_x(firstMomentElectron); 
    boundaryPIC.freeBoundaryFirstMoment_y(firstMomentElectron); 


    thrust::fill(reloadParticlesDataIon.begin(), reloadParticlesDataIon.end(), ReloadParticlesData());
    thrust::fill(reloadParticlesDataElectron.begin(), reloadParticlesDataElectron.end(), ReloadParticlesData());

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((localSizeXInterface + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (localSizeYInterface + threadsPerBlock.y - 1) / threadsPerBlock.y);

    sendMHDtoPIC_particle_y_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(interlockingFunctionY.data()), 
        thrust::raw_pointer_cast(zerothMomentIon.data()), 
        thrust::raw_pointer_cast(zerothMomentElectron.data()), 
        thrust::raw_pointer_cast(firstMomentIon.data()), 
        thrust::raw_pointer_cast(firstMomentElectron.data()), 
        thrust::raw_pointer_cast(U.data()),  
        thrust::raw_pointer_cast(reloadParticlesDataIon.data()), 
        thrust::raw_pointer_cast(reloadParticlesDataElectron.data()), 
        indexOfInterfaceStartInMHD, 
        indexOfInterfaceStartInPIC, 
        localSizeXPIC, localSizeYPIC,  
        localSizeXMHD, localSizeYMHD, 
        localSizeXInterface, localSizeYInterface
    );
    hipDeviceSynchronize();

    Interface2DMPI::sendrecv_reloadParticlesData_x(reloadParticlesDataIon, mPIInfoInterface);
    Interface2DMPI::sendrecv_reloadParticlesData_x(reloadParticlesDataElectron, mPIInfoInterface);
    
    deleteParticlesSpecies(
        particlesIon, mPIInfoPIC.existNumIonPerProcs, seed + 100
    );
    deleteParticlesSpecies(
        particlesElectron, mPIInfoPIC.existNumElectronPerProcs, seed + 200
    );

    reloadParticlesSpecies(
        particlesIon, reloadParticlesDataIon, reloadParticlesSourceIon, 
        mPIInfoPIC.existNumIonPerProcs, seed + 300
    ); 
    reloadParticlesSpecies(
        particlesElectron, reloadParticlesDataElectron, reloadParticlesSourceElectron, 
        mPIInfoPIC.existNumElectronPerProcs, seed + 400
    ); 
}


