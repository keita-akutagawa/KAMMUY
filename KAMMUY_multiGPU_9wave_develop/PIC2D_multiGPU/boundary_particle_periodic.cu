#include "hip/hip_runtime.h"
#include "boundary.hpp"


void BoundaryPIC::periodicBoundaryParticle_x(
    thrust::device_vector<Particle>& particlesIon, 
    thrust::device_vector<Particle>& particlesElectron
)
{   
    periodicBoundaryParticleOfOneSpecies_x(
        particlesIon,
        mPIInfo.existNumIonPerProcs, 
        mPIInfo.numForSendParticlesIonLeft, 
        mPIInfo.numForSendParticlesIonRight, 
        mPIInfo.numForRecvParticlesIonLeft, 
        mPIInfo.numForRecvParticlesIonRight
    );
    periodicBoundaryParticleOfOneSpecies_x(
        particlesElectron, 
        mPIInfo.existNumElectronPerProcs,  
        mPIInfo.numForSendParticlesElectronLeft, 
        mPIInfo.numForSendParticlesElectronRight, 
        mPIInfo.numForRecvParticlesElectronLeft, 
        mPIInfo.numForRecvParticlesElectronRight
    );
    MPI_Barrier(MPI_COMM_WORLD);
}


__global__ void periodicBoundaryParticle_x_kernel(
    Particle* particlesSpecies, 
    Particle* sendParticlesSpeciesLeft, 
    Particle* sendParticlesSpeciesRight, 
    unsigned int* countForSendParticlesSpeciesLeft, 
    unsigned int* countForSendParticlesSpeciesRight, 
    const unsigned long long existNumSpecies, 
    const float xminForProcs, const float xmaxForProcs, 
    const int buffer
)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < existNumSpecies) {
        if (particlesSpecies[i].isMPISendRight) {
            unsigned int particleIndex = atomicAdd(&(countForSendParticlesSpeciesRight[0]), 1);
            particlesSpecies[i].isMPISendRight = false;
            Particle sendParticle = particlesSpecies[i];
            if (sendParticle.x > PIC2DConst::device_xmax - buffer * PIC2DConst::device_dx + PIC2DConst::device_EPS) {
                sendParticle.x = sendParticle.x - PIC2DConst::device_xmax + PIC2DConst::device_EPS;
            }
            sendParticlesSpeciesRight[particleIndex] = sendParticle;
        }

        if (particlesSpecies[i].isMPISendLeft) {
            unsigned int particleIndex = atomicAdd(&(countForSendParticlesSpeciesLeft[0]), 1);
            particlesSpecies[i].isMPISendLeft = false;
            Particle sendParticle = particlesSpecies[i];
            if (sendParticle.x < PIC2DConst::device_xmin + buffer * PIC2DConst::device_dx - PIC2DConst::device_EPS) {
                sendParticle.x = sendParticle.x + PIC2DConst::device_xmax - PIC2DConst::device_EPS;
            }
            sendParticlesSpeciesLeft[particleIndex] = sendParticle;
        }
    }
}

void BoundaryPIC::periodicBoundaryParticleOfOneSpecies_x(
    thrust::device_vector<Particle>& particlesSpecies, 
    unsigned long long& existNumSpecies, 
    unsigned int& numForSendParticlesSpeciesLeft, 
    unsigned int& numForSendParticlesSpeciesRight, 
    unsigned int& numForRecvParticlesSpeciesLeft, 
    unsigned int& numForRecvParticlesSpeciesRight
)
{
    thrust::device_vector<unsigned int> countForSendParticlesSpeciesLeft(1, 0); 
    thrust::device_vector<unsigned int> countForSendParticlesSpeciesRight(1, 0); 

    dim3 threadsPerBlock(256);
    dim3 blocksPerGrid((existNumSpecies + threadsPerBlock.x - 1) / threadsPerBlock.x);

    periodicBoundaryParticle_x_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(particlesSpecies.data()), 
        thrust::raw_pointer_cast(sendParticlesSpeciesLeft.data()), 
        thrust::raw_pointer_cast(sendParticlesSpeciesRight.data()), 
        thrust::raw_pointer_cast(countForSendParticlesSpeciesLeft.data()), 
        thrust::raw_pointer_cast(countForSendParticlesSpeciesRight.data()), 
        existNumSpecies, 
        mPIInfo.xminForProcs, mPIInfo.xmaxForProcs, 
        mPIInfo.buffer
    );
    hipDeviceSynchronize();

    PIC2DMPI::sendrecv_numParticle_x(
        numForSendParticlesSpeciesLeft, 
        numForSendParticlesSpeciesRight, 
        numForRecvParticlesSpeciesLeft, 
        numForRecvParticlesSpeciesRight, 
        mPIInfo
    );

    PIC2DMPI::sendrecv_particle_x(
        sendParticlesSpeciesLeft, 
        sendParticlesSpeciesRight,  
        recvParticlesSpeciesLeft, 
        recvParticlesSpeciesRight,  
        numForSendParticlesSpeciesLeft, 
        numForSendParticlesSpeciesRight, 
        numForRecvParticlesSpeciesLeft, 
        numForRecvParticlesSpeciesRight, 
        mPIInfo
    );

    thrust::copy(
        recvParticlesSpeciesLeft.begin(), 
        recvParticlesSpeciesLeft.begin() + numForRecvParticlesSpeciesLeft,
        particlesSpecies.begin() + existNumSpecies
    );
    existNumSpecies += numForRecvParticlesSpeciesLeft;
    thrust::copy(
        recvParticlesSpeciesRight.begin(), 
        recvParticlesSpeciesRight.begin() + numForRecvParticlesSpeciesRight,
        particlesSpecies.begin() + existNumSpecies
    );
    existNumSpecies += numForRecvParticlesSpeciesRight;
}


