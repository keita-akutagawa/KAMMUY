#include "const.hpp"


void IdealMHD2DConst::initializeDeviceConstants() {
    hipMemcpyToSymbol(HIP_SYMBOL(device_EPS), &EPS, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_PI), &PI, sizeof(double));

    hipMemcpyToSymbol(HIP_SYMBOL(device_eta), &eta, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_viscosity), &viscosity, sizeof(double));

    hipMemcpyToSymbol(HIP_SYMBOL(device_B0), &B0, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_rho0), &rho0, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_p0), &p0, sizeof(double));

    hipMemcpyToSymbol(HIP_SYMBOL(device_dx), &dx, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_xmin), &xmin, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_xmax), &xmax, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_nx), &nx, sizeof(int));

    hipMemcpyToSymbol(HIP_SYMBOL(device_dy), &dy, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_ymin), &ymin, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_ymax), &ymax, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_ny), &ny, sizeof(int));

    hipMemcpyToSymbol(HIP_SYMBOL(device_CFL), &CFL, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_gamma), &gamma, sizeof(double));

    hipMemcpyToSymbol(HIP_SYMBOL(device_dt), &dt, sizeof(double));
}
