#include "hip/hip_runtime.h"
#include "filter.hpp"
#include <thrust/fill.h>


Filter::Filter(PIC2DMPI::MPIInfo& mPIInfo)
    : mPIInfo(mPIInfo), 
      rho(mPIInfo.localSizeX * PIC2DConst::ny), 
      F(mPIInfo.localSizeX * PIC2DConst::ny)
{
}


__global__ void calculateF_kernel(
    FilterField* F, ElectricField* E, RhoField* rho, 
    int localSizeX
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if ((0 < i) && (i < localSizeX - 1) && (0 < j) && (j < PIC2DConst::device_ny - 1)) {
        int index = j + i * PIC2DConst::device_ny;

        F[index].F = ((E[index].eX - E[index - PIC2DConst::device_ny].eX) / PIC2DConst::device_dx 
                   + (E[index].eY - E[index - 1].eY) / PIC2DConst::device_dy)
                   - rho[index].rho / PIC2DConst::device_epsilon0;
    }
}

__global__ void correctE_kernel(
    ElectricField* E, FilterField* F, float dt, 
    int localSizeX
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if ((0 < i) && (i < localSizeX - 1) && (0 < j) && (j < PIC2DConst::device_ny - 1)) {
        int index = j + i * PIC2DConst::device_ny;

        E[index].eX += PIC2DConst::device_dOfLangdonMarderTypeCorrection
                     * (F[index + PIC2DConst::device_ny].F - F[index].F) / PIC2DConst::device_dx * dt;
        E[index].eY += PIC2DConst::device_dOfLangdonMarderTypeCorrection
                     * (F[index + 1].F - F[index].F) / PIC2DConst::device_dy * dt;
    }
}


void Filter::langdonMarderTypeCorrection(
    thrust::device_vector<ElectricField>& E, 
    const thrust::device_vector<Particle>& particlesIon, 
    const thrust::device_vector<Particle>& particlesElectron, 
    const float dt
)
{
    calculateRho(particlesIon, particlesElectron);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((mPIInfo.localSizeX + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (PIC2DConst::ny + threadsPerBlock.y - 1) / threadsPerBlock.y);

    calculateF_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(F.data()), 
        thrust::raw_pointer_cast(E.data()), 
        thrust::raw_pointer_cast(rho.data()), 
        mPIInfo.localSizeX
    );
    hipDeviceSynchronize();

    correctE_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(E.data()), 
        thrust::raw_pointer_cast(F.data()), 
        dt, 
        mPIInfo.localSizeX
    );
    hipDeviceSynchronize();
}


void Filter::resetRho()
{
    thrust::fill(rho.begin(), rho.end(), RhoField());
}


void Filter::calculateRho(
    const thrust::device_vector<Particle>& particlesIon, 
    const thrust::device_vector<Particle>& particlesElectron
)
{
    resetRho();

    calculateRhoOfOneSpecies(particlesIon, PIC2DConst::qIon, mPIInfo.existNumIonPerProcs);
    calculateRhoOfOneSpecies(particlesElectron, PIC2DConst::qElectron, mPIInfo.existNumElectronPerProcs);
}


__global__ void calculateRhoOfOneSpecies_kernel(
    RhoField* rho, const Particle* particlesSpecies, 
    const float q, const unsigned long long existNumSpecies, 
    const int localNx, const int buffer, 
    const int localSizeX, 
    const float xminForProcs, const float xmaxForProcs
)
{
    unsigned long long i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < existNumSpecies) {

        float cx1, cx2; 
        int xIndex1, xIndex2;
        float xOverDx;
        float cy1, cy2; 
        int yIndex1, yIndex2;
        float yOverDy;

        xOverDx = (particlesSpecies[i].x - xminForProcs + buffer * PIC2DConst::device_dx) / PIC2DConst::device_dx;
        yOverDy = (particlesSpecies[i].y - PIC2DConst::device_ymin) / PIC2DConst::device_dy;

        xIndex1 = floorf(xOverDx);
        xIndex2 = xIndex1 + 1;
        xIndex2 = (xIndex2 == localSizeX) ? 0 : xIndex2;
        yIndex1 = floorf(yOverDy);
        yIndex2 = yIndex1 + 1;
        yIndex2 = (yIndex2 == PIC2DConst::device_ny) ? 0 : yIndex2;

        cx1 = xOverDx - xIndex1;
        cx2 = 1.0f - cx1;
        cy1 = yOverDy - yIndex1;
        cy2 = 1.0f - cy1;

        atomicAdd(&(rho[yIndex1 + PIC2DConst::device_ny * xIndex1].rho), q * cx2 * cy2);
        atomicAdd(&(rho[yIndex2 + PIC2DConst::device_ny * xIndex1].rho), q * cx2 * cy1);
        atomicAdd(&(rho[yIndex1 + PIC2DConst::device_ny * xIndex2].rho), q * cx1 * cy2);
        atomicAdd(&(rho[yIndex2 + PIC2DConst::device_ny * xIndex2].rho), q * cx1 * cy1);
    }
};


void Filter::calculateRhoOfOneSpecies(
    const thrust::device_vector<Particle>& particlesSpecies, 
    float q, unsigned long long existNumSpecies
)
{
    dim3 threadsPerBlock(256);
    dim3 blocksPerGrid((existNumSpecies + threadsPerBlock.x - 1) / threadsPerBlock.x);

    calculateRhoOfOneSpecies_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(rho.data()), 
        thrust::raw_pointer_cast(particlesSpecies.data()), 
        q, existNumSpecies, 
        mPIInfo.localNx, mPIInfo.buffer, 
        mPIInfo.localSizeX, 
        mPIInfo.xminForProcs, mPIInfo.xmaxForProcs
    );
    hipDeviceSynchronize();
}



