#include "hip/hip_runtime.h"
#include "main_stay_const.hpp"

__global__ void initializeU_kernel(
    ConservationParameter* U, 
    IdealMHD2DMPI::MPIInfo* device_mPIInfo
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < IdealMHD2DConst::device_nx && j < IdealMHD2DConst::device_ny) {
        IdealMHD2DMPI::MPIInfo mPIInfo = *device_mPIInfo;

        if (mPIInfo.isInside(i)) {
            int index = mPIInfo.globalToLocal(i, j);

            double rho, u, v, w, bX, bY, bZ, e, p;
            
            rho = IdealMHD2DConst::device_rho0;
            u   = 0.0;
            v   = 0.0;
            w   = 0.0;
            bX  = 0.0;
            bY  = 0.0;
            bZ  = 0.0f;
            p   = IdealMHD2DConst::device_p0;
            e   = p / (IdealMHD2DConst::device_gamma - 1.0)
                + 0.5 * rho * (u * u + v * v + w * w)
                + 0.5 * (bX * bX + bY * bY + bZ * bZ);

            U[index].rho  = rho;
            U[index].rhoU = rho * u;
            U[index].rhoV = rho * v;
            U[index].rhoW = rho * w;
            U[index].bX   = bX;
            U[index].bY   = bY;
            U[index].bZ   = bZ;
            U[index].e    = e;
        }
    }
}

void IdealMHD2D::initializeU()
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((IdealMHD2DConst::nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (IdealMHD2DConst::ny + threadsPerBlock.y - 1) / threadsPerBlock.y);

    initializeU_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(U.data()), 
        device_mPIInfo
    );
    hipDeviceSynchronize();

    MPI_Barrier(MPI_COMM_WORLD);

    boundaryMHD.periodicBoundaryX2nd_U(U);

    MPI_Barrier(MPI_COMM_WORLD);
}


__global__ void initializePICField_kernel(
    ElectricField* E, MagneticField* B, 
    PIC2DMPI::MPIInfo* device_mPIInfo
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < PIC2DConst::device_nx && j < PIC2DConst::device_ny) {
        PIC2DMPI::MPIInfo mPIInfo = *device_mPIInfo;

        if (mPIInfo.isInside(i)) {
            int index = mPIInfo.globalToLocal(i, j);
            float bX, bY, bZ, eX, eY, eZ;

            bX = 0.0f; 
            bY = 0.0f;
            bZ = 0.0f; 
            eX = 0.0f; 
            eY = 0.0f; 
            eZ = 0.0f; 

            E[index].eX = eX;
            E[index].eY = eY;
            E[index].eZ = eZ;
            B[index].bX = bX;
            B[index].bY = bY; 
            B[index].bZ = bZ;
        }
    }
}

void PIC2D::initialize()
{
    for (int i = 0; i < mPIInfo.localNx; i++) {
        for (int j = 0; j < PIC2DConst::ny; j++) {
            float xminLocal, xmaxLocal, yminLocal, ymaxLocal;
            float bulkVx, bulkVy, bulkVz;

            xminLocal = i * PIC2DConst::dx + mPIInfo.xminForProcs;
            xmaxLocal = (i + 1) * PIC2DConst::dx + mPIInfo.xminForProcs;
            yminLocal = j * PIC2DConst::dy + PIC2DConst::ymin;
            ymaxLocal = (j + 1) * PIC2DConst::dy + PIC2DConst::ymin;
            bulkVx = 0.0f;
            bulkVy = 0.0f;
            bulkVz = 0.0f;

            initializeParticle.uniformForPosition_xy_maxwellDistributionForVelocity_eachCell(
                xminLocal, xmaxLocal, yminLocal, ymaxLocal, 
                bulkVx, bulkVy, bulkVz,  
                PIC2DConst::vThIon, PIC2DConst::vThIon, PIC2DConst::vThIon, 
                (j + i * PIC2DConst::ny) * PIC2DConst::numberDensityIon, (j + i * PIC2DConst::ny + 1) * PIC2DConst::numberDensityIon, 
                j + i * PIC2DConst::ny + mPIInfo.rank * mPIInfo.localNx * PIC2DConst::ny, 
                particlesIon
            );
            initializeParticle.uniformForPosition_xy_maxwellDistributionForVelocity_eachCell(
                xminLocal, xmaxLocal, yminLocal, ymaxLocal, 
                bulkVx, bulkVy, bulkVz,  
                PIC2DConst::vThElectron, PIC2DConst::vThElectron, PIC2DConst::vThElectron, 
                (j + i * PIC2DConst::ny) * PIC2DConst::numberDensityElectron, (j + i * PIC2DConst::ny + 1) * PIC2DConst::numberDensityElectron, 
                j + i * PIC2DConst::ny + mPIInfo.localNx * PIC2DConst::ny + mPIInfo.rank * mPIInfo.localNx * PIC2DConst::ny, 
                particlesElectron
            );
        }
    }


    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((PIC2DConst::nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (PIC2DConst::ny + threadsPerBlock.y - 1) / threadsPerBlock.y);

    initializePICField_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(E.data()), thrust::raw_pointer_cast(B.data()), 
        device_mPIInfo
    );
    hipDeviceSynchronize();

    MPI_Barrier(MPI_COMM_WORLD);

    boundaryPIC.periodicBoundaryB_x(B);
    boundaryPIC.periodicBoundaryE_x(E);
    boundaryPIC.periodicBoundaryCurrent_x(current);
    boundaryPIC.periodicBoundaryForInitializeParticle_x(particlesIon, particlesElectron);
    
    MPI_Barrier(MPI_COMM_WORLD);
}



int main(int argc, char** argv)
{
    MPI_Init(&argc, &argv);

    int rank = 0, procs = 0;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &procs);

    PIC2DMPI::MPIInfo mPIInfoPIC;
    int mpiBufNumParticles = 1000000; 
    PIC2DMPI::setupInfo(mPIInfoPIC, buffer, mpiBufNumParticles);
    IdealMHD2DMPI::MPIInfo mPIInfoMHD;
    IdealMHD2DMPI::setupInfo(mPIInfoMHD, buffer);
    Interface2DMPI::MPIInfo mPIInfoInterface; 
    Interface2DMPI::setupInfo(mPIInfoInterface, buffer); 

    if (mPIInfoPIC.rank == 0) {
        std::cout   << mPIInfoPIC.gridX << std::endl;
        mpifile_PIC << mPIInfoPIC.gridX << std::endl;
    }
    if (mPIInfoMHD.rank == 0) {
        std::cout   << mPIInfoMHD.gridX << std::endl;
        mpifile_MHD << mPIInfoMHD.gridX << std::endl;
    }
    if (mPIInfoInterface.rank == 0) {
        std::cout   << mPIInfoInterface.gridX << std::endl;
        mpifile_Interface << mPIInfoInterface.gridX << std::endl;
    }

    hipSetDevice(mPIInfoPIC.rank);

    PIC2DConst::initializeDeviceConstants();
    IdealMHD2DConst::initializeDeviceConstants();
    Interface2DConst::initializeDeviceConstants();

    mPIInfoPIC.existNumIonPerProcs      = PIC2DConst::totalNumIon / mPIInfoPIC.procs;
    mPIInfoPIC.existNumElectronPerProcs = PIC2DConst::totalNumElectron / mPIInfoPIC.procs;
    mPIInfoPIC.totalNumIonPerProcs = mPIInfoPIC.existNumIonPerProcs * 2;
    mPIInfoPIC.totalNumElectronPerProcs = mPIInfoPIC.existNumElectronPerProcs * 2;

    mPIInfoPIC.xminForProcs = PIC2DConst::xmin
                            + (PIC2DConst::xmax - PIC2DConst::xmin) / mPIInfoPIC.gridX
                            * mPIInfoPIC.localGridX;
    mPIInfoPIC.xmaxForProcs = PIC2DConst::xmin
                            + (PIC2DConst::xmax - PIC2DConst::xmin) / mPIInfoPIC.gridX
                            * (mPIInfoPIC.localGridX + 1);
    
    for (int i = 0; i < mPIInfoPIC.localSizeX; i++) {
        for (int j = 0; j < PIC2DConst::ny; j++) {
            host_interlockingFunctionY[j + i * PIC2DConst::ny]
                = max(1.0
                - (1.0 - exp(-pow((j - 0) / Interface2DConst::deltaForInterlockingFunction, 2)))
                * (1.0 - exp(-pow((j - (PIC2DConst::ny - 1)) / Interface2DConst::deltaForInterlockingFunction, 2))), 
                Interface2DConst::EPS); 
        }
    }
    
    IdealMHD2D idealMHD2D(mPIInfoMHD);
    PIC2D pIC2D(mPIInfoPIC); 
    InterfaceNoiseRemover2D interfaceNoiseRemover2D( 
        mPIInfoMHD, mPIInfoPIC
    );
    Interface2D interface2D(
        mPIInfoMHD, mPIInfoPIC, mPIInfoInterface, 
        indexOfInterfaceStartInMHD, 
        host_interlockingFunctionY, 
        interfaceNoiseRemover2D
    );
    BoundaryMHD& boundaryMHD = idealMHD2D.getBoundaryMHDRef(); 
    BoundaryPIC& boundaryPIC = pIC2D.getBoundaryPICRef(); 
    

    if (mPIInfoPIC.rank == 0) {
        size_t free_mem = 0;
        size_t total_mem = 0;
        hipError_t status = hipMemGetInfo(&free_mem, &total_mem);

        std::cout << "Free memory: " << free_mem / (1024 * 1024) << " MB" << std::endl;
        std::cout << "Total memory: " << total_mem / (1024 * 1024) << " MB" << std::endl;

        std::cout << "exist number of partices is " 
                  << mPIInfoPIC.procs * (mPIInfoPIC.existNumIonPerProcs + mPIInfoPIC.existNumElectronPerProcs) 
                  << std::endl;
        std::cout << "exist number of partices + buffer particles is " 
                  << mPIInfoPIC.procs * (mPIInfoPIC.totalNumIonPerProcs + mPIInfoPIC.totalNumElectronPerProcs) 
                  << std::endl;
        
        std::cout << "PIC grid size is " 
                  << mPIInfoPIC.localSizeX << " X " << PIC2DConst::ny 
                  << std::endl;
        std::cout << "MHD grid size is " 
                  << mPIInfoMHD.localSizeX << " X " << IdealMHD2DConst::ny
                  << std::endl;
    }

    idealMHD2D.initializeU(); 
    pIC2D.initialize();

    const int totalSubstep = int(round(sqrt(PIC2DConst::mRatio)));
    for (int step = 0; step < IdealMHD2DConst::totalStep + 1; step++) {
        MPI_Barrier(MPI_COMM_WORLD);

        if (mPIInfoPIC.rank == 0) {
            if (step % recordStep == 0) {
                std::cout << std::to_string(step) << " step done : total time is "
                        << std::setprecision(4) << step * totalSubstep * PIC2DConst::dt * PIC2DConst::omegaPe
                        << " [omega_pe * t]"
                        << std::endl;
            }
        }

        if (step % recordStep == 0) {
            logfile << std::setprecision(6) << IdealMHD2DConst::totalTime << std::endl;
            pIC2D.saveParticle(
                directoryName, filenameWithoutStep, step
            );
            pIC2D.saveFields(
                directoryName, filenameWithoutStep, step
            );
            pIC2D.saveZerothMoments(
                directoryName, filenameWithoutStep, step
            );
            pIC2D.saveFirstMoments(
                directoryName, filenameWithoutStep, step
            );
            idealMHD2D.save(
                directoryName, filenameWithoutStep + "_U", step
            );
        }

        double dtCommon = min(0.7 * PIC2DConst::c, 0.1 * 1.0 / PIC2DConst::omegaPe);
        PIC2DConst::dt = dtCommon;
        IdealMHD2DConst::dt = totalSubstep * dtCommon;
        hipMemcpyToSymbol(HIP_SYMBOL(PIC2DConst::device_dt), &PIC2DConst::dt, sizeof(float));
        hipMemcpyToSymbol(HIP_SYMBOL(IdealMHD2DConst::device_dt), &IdealMHD2DConst::dt, sizeof(double));

        // STEP1 : MHD step

        idealMHD2D.setPastU();
        thrust::device_vector<ConservationParameter>& UPast = idealMHD2D.getUPastRef();

        idealMHD2D.oneStepRK2_periodicXSymmetricY_predictor();

        thrust::device_vector<ConservationParameter>& UNext = idealMHD2D.getURef();

        // STEP2 : PIC step & send MHD to PIC

        interface2D.resetTimeAveragedPICParameters();

        int sumUpCount; 
        sumUpCount = 0; 
        for (int substep = 1; substep <= totalSubstep; substep++) {

            float mixingRatio = 1.0 - substep / totalSubstep;
            thrust::device_vector<ConservationParameter>& USub = interface2D.calculateAndGetSubU(UPast, UNext, mixingRatio);
            
            unsigned long long seedForReload; 
            seedForReload = substep + step * totalSubstep;
            pIC2D.oneStep_periodicXFreeY(
                interface2D, 
                USub, 
                seedForReload
            );

            thrust::device_vector<MagneticField>& B = pIC2D.getBRef();
            thrust::device_vector<ZerothMoment>& zerothMomentIon = pIC2D.getZerothMomentIonRef(); 
            thrust::device_vector<ZerothMoment>& zerothMomentElectron = pIC2D.getZerothMomentElectronRef(); 
            thrust::device_vector<FirstMoment>& firstMomentIon = pIC2D.getFirstMomentIonRef(); 
            thrust::device_vector<FirstMoment>& firstMomentElectron = pIC2D.getFirstMomentElectronRef(); 
            interface2D.sumUpTimeAveragedPICParameters(
                B, 
                zerothMomentIon, zerothMomentElectron, 
                firstMomentIon, firstMomentElectron
            );
            sumUpCount += 1; 
        }

        interface2D.calculateTimeAveragedPICParameters(sumUpCount); 

        interface2D.setParametersForPICtoMHD();

        // STEP3 : send PIC to MHD

        interface2D.calculateUHalf(UPast, UNext); 
        thrust::device_vector<ConservationParameter>& UHalf = interface2D.getUHalfRef();

        //interface2D.sendPICtoMHD(UHalf);
        boundaryMHD.periodicBoundaryX2nd_U(UHalf);
        boundaryMHD.symmetricBoundaryY2nd_U(UHalf);

        for (int count = 0; count < Interface2DConst::convolutionCount; count++) {
            interfaceNoiseRemover2D.convolveU(UHalf);

            boundaryMHD.periodicBoundaryX2nd_U(UHalf);
            boundaryMHD.symmetricBoundaryY2nd_U(UHalf);
        }

        idealMHD2D.oneStepRK2_periodicXSymmetricY_corrector(UHalf);


        //when crashed 
        if (idealMHD2D.checkCalculationIsCrashed()) {
            logfile << std::setprecision(6) << PIC2DConst::totalTime << std::endl;
            pIC2D.saveFields(
                directoryName, filenameWithoutStep, step
            );
            pIC2D.saveZerothMoments(
                directoryName, filenameWithoutStep, step
            );
            pIC2D.saveFirstMoments(
                directoryName, filenameWithoutStep, step
            );
            idealMHD2D.save(
                directoryName, filenameWithoutStep + "_U", step
            );
            std::cout << "Calculation stopped! : " << step << " steps" << std::endl;
            break;
        }

        if (mPIInfoMHD.rank == 0) {
            IdealMHD2DConst::totalTime += IdealMHD2DConst::dt;
        }   
    }

    MPI_Finalize();

    if (mPIInfoMHD.rank == 0) {
        std::cout << "program was completed!" << std::endl;
    }

    return 0;
}



