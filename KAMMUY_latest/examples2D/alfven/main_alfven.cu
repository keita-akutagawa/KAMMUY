#include "hip/hip_runtime.h"
#include "main_alfven_const.hpp"


__global__ void initializeU_kernel(
    ConservationParameter* U, 
    double VA, double waveAmp, double waveNumber, 
    IdealMHD2DMPI::MPIInfo* device_mPIInfo
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < IdealMHD2DConst::device_nx && j < IdealMHD2DConst::device_ny) {
        IdealMHD2DMPI::MPIInfo mPIInfo = *device_mPIInfo;

        if (mPIInfo.isInside(i)) {
            int index = mPIInfo.globalToLocal(i, j);

            double rho, u, v, w, bX, bY, bZ, e, p;
            double y = j * IdealMHD2DConst::device_dy + IdealMHD2DConst::device_ymin;
            
            rho = IdealMHD2DConst::device_rho0;
            u   = waveAmp * VA * cos(waveNumber * y);
            v   = 0.0;
            w   = -waveAmp * VA * sin(waveNumber * y);
            bX  = -waveAmp * IdealMHD2DConst::device_B0 * cos(waveNumber * y);
            bY  = IdealMHD2DConst::device_B0;
            bZ  = waveAmp * IdealMHD2DConst::device_B0 * sin(waveNumber * y);
            p   = IdealMHD2DConst::device_p0;
            e   = p / (IdealMHD2DConst::device_gamma - 1.0)
                + 0.5 * rho * (u * u + v * v + w * w)
                + 0.5 * (bX * bX + bY * bY + bZ * bZ);

            U[index].rho  = rho;
            U[index].rhoU = rho * u;
            U[index].rhoV = rho * v;
            U[index].rhoW = rho * w;
            U[index].bX   = bX;
            U[index].bY   = bY;
            U[index].bZ   = bZ;
            U[index].e    = e;
        }
    }
}

void IdealMHD2D::initializeU()
{
    double VA = IdealMHD2DConst::B0 / sqrt(IdealMHD2DConst::rho0); 

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((IdealMHD2DConst::nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (IdealMHD2DConst::ny + threadsPerBlock.y - 1) / threadsPerBlock.y);

    initializeU_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(U.data()), 
        VA, waveAmp, waveNumber, 
        device_mPIInfo
    );
    hipDeviceSynchronize();

    MPI_Barrier(MPI_COMM_WORLD);

    boundaryMHD.periodicBoundaryX2nd_U(U);

    MPI_Barrier(MPI_COMM_WORLD);
}


__global__ void initializePICField_kernel(
    ElectricField* E, MagneticField* B, 
    double VA, double waveAmp, double waveNumber, 
    PIC2DMPI::MPIInfo* device_mPIInfo
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < PIC2DConst::device_nx && j < PIC2DConst::device_ny) {
        PIC2DMPI::MPIInfo mPIInfo = *device_mPIInfo;

        if (mPIInfo.isInside(i)) {
            int index = mPIInfo.globalToLocal(i, j);
            double u, v, w, bX, bY, bZ, eX, eY, eZ;
            double y = j * PIC2DConst::device_dy + 2450 * IdealMHD2DConst::device_dy + PIC2DConst::device_ymin;

            bX = -waveAmp * PIC2DConst::device_B0 * cos(waveNumber * y);
            bY = PIC2DConst::device_B0; 
            bZ = waveAmp * PIC2DConst::device_B0 * sin(waveNumber * y);
            u  = waveAmp * VA * cos(waveNumber * y);
            v  = 0.0;
            w  = -waveAmp * VA * sin(waveNumber * y);
            eX = -(v * bZ - w * bY);
            eY = -(w * bX - u * bZ);
            eZ = -(u * bY - v * bX);

            E[index].eX = eX;
            E[index].eY = eY;
            E[index].eZ = eZ;
            B[index].bX = bX;
            B[index].bY = bY; 
            B[index].bZ = bZ;
        }
    }
}

void PIC2D::initialize()
{
    float VA = IdealMHD2DConst::B0 / sqrt(IdealMHD2DConst::rho0); 

    for (int i = 0; i < mPIInfo.localNx; i++) {
        for (int j = 0; j < PIC2DConst::ny; j++) {
            float xminLocal, xmaxLocal, yminLocal, ymaxLocal;
            float bulkVx, bulkVy, bulkVz;
            float y = j * PIC2DConst::dy + 2450 * IdealMHD2DConst::dy;

            xminLocal = i * PIC2DConst::dx + mPIInfo.xminForProcs;
            xmaxLocal = (i + 1) * PIC2DConst::dx + mPIInfo.xminForProcs;
            yminLocal = j * PIC2DConst::dy + PIC2DConst::ymin;
            ymaxLocal = (j + 1) * PIC2DConst::dy + PIC2DConst::ymin;
            bulkVx = waveAmp * VA * cos(waveNumber * y);
            bulkVy = 0.0;
            bulkVz = -waveAmp * VA * sin(waveNumber * y);

            initializeParticle.uniformForPosition_xy_maxwellDistributionForVelocity_eachCell(
                xminLocal, xmaxLocal, yminLocal, ymaxLocal, 
                bulkVx, bulkVy, bulkVz,  
                PIC2DConst::vThIon, PIC2DConst::vThIon, PIC2DConst::vThIon, 
                (j + i * PIC2DConst::ny) * PIC2DConst::numberDensityIon, (j + i * PIC2DConst::ny + 1) * PIC2DConst::numberDensityIon, 
                j + i * PIC2DConst::ny + mPIInfo.rank * mPIInfo.localNx * PIC2DConst::ny, 
                particlesIon
            );
            initializeParticle.uniformForPosition_xy_maxwellDistributionForVelocity_eachCell(
                xminLocal, xmaxLocal, yminLocal, ymaxLocal, 
                bulkVx, bulkVy, bulkVz,  
                PIC2DConst::vThElectron, PIC2DConst::vThElectron, PIC2DConst::vThElectron, 
                (j + i * PIC2DConst::ny) * PIC2DConst::numberDensityElectron, (j + i * PIC2DConst::ny + 1) * PIC2DConst::numberDensityElectron, 
                j + i * PIC2DConst::ny + mPIInfo.localNx * PIC2DConst::ny + mPIInfo.rank * mPIInfo.localNx * PIC2DConst::ny, 
                particlesElectron
            );
        }
    }


    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((PIC2DConst::nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (PIC2DConst::ny + threadsPerBlock.y - 1) / threadsPerBlock.y);

    initializePICField_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(E.data()), thrust::raw_pointer_cast(B.data()), 
        VA, waveAmp, waveNumber, 
        device_mPIInfo
    );
    hipDeviceSynchronize();

    MPI_Barrier(MPI_COMM_WORLD);

    boundaryPIC.periodicBoundaryB_x(B);
    boundaryPIC.periodicBoundaryE_x(E);
    boundaryPIC.periodicBoundaryCurrent_x(current);
    boundaryPIC.periodicBoundaryForInitializeParticle_x(particlesIon, particlesElectron);
    
    MPI_Barrier(MPI_COMM_WORLD);
}



int main(int argc, char** argv)
{
    MPI_Init(&argc, &argv);

    int rank = 0, procs = 0;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &procs);

    PIC2DMPI::MPIInfo mPIInfoPIC;
    int mpiBufNumParticles = 1000000; 
    PIC2DMPI::setupInfo(mPIInfoPIC, buffer, mpiBufNumParticles);
    IdealMHD2DMPI::MPIInfo mPIInfoMHD;
    IdealMHD2DMPI::setupInfo(mPIInfoMHD, buffer);
    Interface2DMPI::MPIInfo mPIInfoInterface; 
    Interface2DMPI::setupInfo(mPIInfoInterface, buffer); 

    if (mPIInfoPIC.rank == 0) {
        std::cout   << mPIInfoPIC.gridX << std::endl;
        mpifile_PIC << mPIInfoPIC.gridX << std::endl;
    }
    if (mPIInfoMHD.rank == 0) {
        std::cout   << mPIInfoMHD.gridX << std::endl;
        mpifile_MHD << mPIInfoMHD.gridX << std::endl;
    }
    if (mPIInfoInterface.rank == 0) {
        std::cout   << mPIInfoInterface.gridX << std::endl;
        mpifile_Interface << mPIInfoInterface.gridX << std::endl;
    }

    hipSetDevice(mPIInfoPIC.rank);

    PIC2DConst::initializeDeviceConstants();
    IdealMHD2DConst::initializeDeviceConstants();
    Interface2DConst::initializeDeviceConstants();

    mPIInfoPIC.existNumIonPerProcs      = PIC2DConst::totalNumIon / mPIInfoPIC.procs;
    mPIInfoPIC.existNumElectronPerProcs = PIC2DConst::totalNumElectron / mPIInfoPIC.procs;
    mPIInfoPIC.totalNumIonPerProcs = mPIInfoPIC.existNumIonPerProcs * 2;
    mPIInfoPIC.totalNumElectronPerProcs = mPIInfoPIC.existNumElectronPerProcs * 2;

    mPIInfoPIC.xminForProcs = PIC2DConst::xmin
                            + (PIC2DConst::xmax - PIC2DConst::xmin) / mPIInfoPIC.gridX
                            * mPIInfoPIC.localGridX;
    mPIInfoPIC.xmaxForProcs = PIC2DConst::xmin
                            + (PIC2DConst::xmax - PIC2DConst::xmin) / mPIInfoPIC.gridX
                            * (mPIInfoPIC.localGridX + 1);
    
    for (int i = 0; i < mPIInfoPIC.localSizeX; i++) {
        for (int j = 0; j < PIC2DConst::ny; j++) {
            double delta = 3.0; 

            host_interlockingFunctionY[j + i * PIC2DConst::ny]
                = 1.0
                - (1.0 - exp(-pow((j - 0) / delta, 2)))
                * (1.0 - exp(-pow((j - (PIC2DConst::ny - 1)) / delta, 2))); 
        }
    }
    
    IdealMHD2D idealMHD2D(mPIInfoMHD);
    PIC2D pIC2D(mPIInfoPIC); 
    InterfaceNoiseRemover2D interfaceNoiseRemover2D( 
        mPIInfoMHD, mPIInfoPIC
    );
    Interface2D interface2D(
        mPIInfoMHD, mPIInfoPIC, mPIInfoInterface, 
        indexOfInterfaceStartInMHD, 
        host_interlockingFunctionY, 
        interfaceNoiseRemover2D
    );
    BoundaryMHD& boundaryMHD = idealMHD2D.getBoundaryMHDRef(); 
    BoundaryPIC& boundaryPIC = pIC2D.getBoundaryPICRef(); 
    

    if (mPIInfoPIC.rank == 0) {
        size_t free_mem = 0;
        size_t total_mem = 0;
        hipError_t status = hipMemGetInfo(&free_mem, &total_mem);

        std::cout << "Free memory: " << free_mem / (1024 * 1024) << " MB" << std::endl;
        std::cout << "Total memory: " << total_mem / (1024 * 1024) << " MB" << std::endl;

        std::cout << "exist number of partices is " 
                  << mPIInfoPIC.procs * (mPIInfoPIC.existNumIonPerProcs + mPIInfoPIC.existNumElectronPerProcs) 
                  << std::endl;
        std::cout << "exist number of partices + buffer particles is " 
                  << mPIInfoPIC.procs * (mPIInfoPIC.totalNumIonPerProcs + mPIInfoPIC.totalNumElectronPerProcs) 
                  << std::endl;
    }

    idealMHD2D.initializeU(); 
    pIC2D.initialize();

    const int totalSubstep = int(round(sqrt(PIC2DConst::mRatio)));
    for (int step = 0; step < IdealMHD2DConst::totalStep + 1; step++) {
        MPI_Barrier(MPI_COMM_WORLD);

        if (mPIInfoPIC.rank == 0) {
            if (step % recordStep == 0) {
                std::cout << std::to_string(step) << " step done : total time is "
                        << std::setprecision(4) << step * totalSubstep * PIC2DConst::dt * PIC2DConst::omegaPe
                        << " [omega_pe * t]"
                        << std::endl;
            }
        }

        if (step % recordStep == 0) {
            logfile << std::setprecision(6) << IdealMHD2DConst::totalTime << std::endl;
            pIC2D.saveParticle(
                directoryName, filenameWithoutStep, step
            );
            pIC2D.saveFields(
                directoryName, filenameWithoutStep, step
            );
            pIC2D.saveZerothMoments(
                directoryName, filenameWithoutStep, step
            );
            pIC2D.saveFirstMoments(
                directoryName, filenameWithoutStep, step
            );
            idealMHD2D.save(
                directoryName, filenameWithoutStep + "_U", step
            );
        }

        double dtCommon = min(0.7 * PIC2DConst::c, 0.1 * 1.0 / PIC2DConst::omegaPe);
        PIC2DConst::dt = dtCommon;
        IdealMHD2DConst::dt = totalSubstep * dtCommon;
        hipMemcpyToSymbol(HIP_SYMBOL(PIC2DConst::device_dt), &PIC2DConst::dt, sizeof(float));
        hipMemcpyToSymbol(HIP_SYMBOL(IdealMHD2DConst::device_dt), &IdealMHD2DConst::dt, sizeof(double));

        // STEP1 : MHD step

        idealMHD2D.setPastU();
        thrust::device_vector<ConservationParameter>& UPast = idealMHD2D.getUPastRef();

        idealMHD2D.oneStepRK2_periodicXSymmetricY_predictor();

        thrust::device_vector<ConservationParameter>& UNext = idealMHD2D.getURef();


        // STEP2 : send MHD to PIC

        float mixingRatio = 0.5f;
        thrust::device_vector<ConservationParameter>& USub = interface2D.calculateAndGetSubU(UPast, UNext, mixingRatio);
        
        thrust::device_vector<MagneticField>& B = pIC2D.getBRef();
        interface2D.sendMHDtoPIC_magneticField_y(USub, B);
        boundaryPIC.periodicBoundaryB_x(B);
        boundaryPIC.freeBoundaryB_y(B);
        
        thrust::device_vector<ElectricField>& E = pIC2D.getERef();
        interface2D.sendMHDtoPIC_electricField_y(USub, E);
        boundaryPIC.periodicBoundaryE_x(E);
        boundaryPIC.freeBoundaryE_y(E);    

        thrust::device_vector<Particle>& particlesIon = pIC2D.getParticlesIonRef();
        thrust::device_vector<Particle>& particlesElectron = pIC2D.getParticlesElectronRef();
        interface2D.sendMHDtoPIC_particle(
            USub, particlesIon, particlesElectron, step * totalSubstep
        );
        boundaryPIC.periodicBoundaryParticle_x(
            particlesIon, particlesElectron
        );
        boundaryPIC.freeBoundaryParticle_y(
            particlesIon, particlesElectron
        );

        // STEP3 : PIC step

        interface2D.resetTimeAveParameters();

        int getDataSubstep = totalSubstep / 2 + 1; 
        for (int substep = 1; substep <= totalSubstep; substep++) {
            pIC2D.oneStep_periodicXFreeY();

            if (substep == getDataSubstep) {
                thrust::device_vector<MagneticField>& B = pIC2D.getBRef();
                thrust::device_vector<MagneticField>& tmpB = pIC2D.getTmpBRef();
                thrust::copy(B.begin(), B.end(), tmpB.begin());
                thrust::device_vector<Particle>& particlesIon = pIC2D.getParticlesIonRef();
                thrust::device_vector<Particle>& particlesElectron = pIC2D.getParticlesElectronRef();
                
                interface2D.sumUpTimeAveParameters(tmpB, particlesIon, particlesElectron);
            }
        }

        // STEP4 : send PIC to MHD
        
        thrust::device_vector<ConservationParameter>& U = idealMHD2D.getURef();

        interface2D.sendPICtoMHD(U);

        boundaryMHD.periodicBoundaryX2nd_U(U);
        boundaryMHD.symmetricBoundaryY2nd_U(U);
        
        for (int count = 0; count < Interface2DConst::convolutionCount; count++) {
            interfaceNoiseRemover2D.convolveU(U);

            boundaryMHD.periodicBoundaryX2nd_U(U);
            boundaryMHD.symmetricBoundaryY2nd_U(U);
        }

        //when crashed 
        if (idealMHD2D.checkCalculationIsCrashed()) {
            logfile << std::setprecision(6) << PIC2DConst::totalTime << std::endl;
            pIC2D.saveFields(
                directoryName, filenameWithoutStep, step
            );
            pIC2D.saveZerothMoments(
                directoryName, filenameWithoutStep, step
            );
            pIC2D.saveFirstMoments(
                directoryName, filenameWithoutStep, step
            );
            idealMHD2D.save(
                directoryName, filenameWithoutStep + "_U", step
            );
            std::cout << "Calculation stopped! : " << step << " steps" << std::endl;
            break;
        }

        if (mPIInfoMHD.rank == 0) {
            IdealMHD2DConst::totalTime += IdealMHD2DConst::dt;
        }   
    }

    MPI_Finalize();

    if (mPIInfoMHD.rank == 0) {
        std::cout << "program was completed!" << std::endl;
    }

    return 0;
}



