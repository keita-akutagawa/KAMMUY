#include "hip/hip_runtime.h"
#include "mpi.hpp"


int PIC2DMPI::MPIInfo::getRank(int dx)
{
    int rankX = (localGridX + dx + gridX) % gridX;
    return rankX;
}


bool PIC2DMPI::MPIInfo::isInside(int globalX)
{
    int startX = localNx * localGridX;
    int endX = startX + localNx;

    if (globalX < startX) return false;
    if (globalX >= endX) return false;

    return true;
}


__device__
int PIC2DMPI::MPIInfo::globalToLocal(int globalX, int globalY)
{
    int startX = localNx * localGridX;
    int x = globalX - startX;

    int y = globalY;

    return y + (x + buffer) * PIC2DConst::device_ny;
}


void PIC2DMPI::setupInfo(MPIInfo& mPIInfo, int buffer, int mpiBufNumParticles)
{
    int rank = 0, procs = 0;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &procs);

    mPIInfo.rank = rank;
    mPIInfo.procs = procs;
    mPIInfo.gridX = procs;
    mPIInfo.localGridX = rank;
    mPIInfo.localNx = PIC2DConst::nx / mPIInfo.gridX;
    mPIInfo.buffer = buffer;
    mPIInfo.localSizeX = mPIInfo.localNx + 2 * mPIInfo.buffer;
    mPIInfo.mpiBufNumParticles = mpiBufNumParticles; 


    int block_lengths_particle[8] = {1, 1, 1, 1, 1, 1, 1, 1};
    MPI_Aint offsets_particle[8];
    offsets_particle[0]  = offsetof(Particle, x);
    offsets_particle[1]  = offsetof(Particle, y);
    offsets_particle[2]  = offsetof(Particle, z);
    offsets_particle[3]  = offsetof(Particle, vx);
    offsets_particle[4]  = offsetof(Particle, vy);
    offsets_particle[5]  = offsetof(Particle, vz);
    offsets_particle[6]  = offsetof(Particle, gamma);
    offsets_particle[7]  = offsetof(Particle, isExist);

    MPI_Datatype types_particle[8] = {
        MPI_FLOAT, MPI_FLOAT, MPI_FLOAT, 
        MPI_FLOAT, MPI_FLOAT, MPI_FLOAT, 
        MPI_FLOAT, MPI_C_BOOL
    };

    MPI_Type_create_struct(8, block_lengths_particle, offsets_particle, types_particle, &mPIInfo.mpi_particleType);
    MPI_Type_commit(&mPIInfo.mpi_particleType);

    // MagneticField, ElectricField, CurrentField共通
    int block_lengths_field[3] = {1, 1, 1};
    MPI_Aint offsets_field[3];
    offsets_field[0] = offsetof(MagneticField, bX);
    offsets_field[1] = offsetof(MagneticField, bY);
    offsets_field[2] = offsetof(MagneticField, bZ);
    MPI_Datatype types_field[3] = {MPI_FLOAT, MPI_FLOAT, MPI_FLOAT};
    MPI_Type_create_struct(3, block_lengths_field, offsets_field, types_field, &mPIInfo.mpi_fieldType);
    MPI_Type_commit(&mPIInfo.mpi_fieldType);

    int block_lengths_zerothMoment[1] = {1};
    MPI_Aint offsets_zerothMoment[1];
    offsets_zerothMoment[0] = offsetof(ZerothMoment, n);
    MPI_Datatype types_zerothMoment[1] = {MPI_FLOAT};
    MPI_Type_create_struct(1, block_lengths_zerothMoment, offsets_zerothMoment, types_zerothMoment, &mPIInfo.mpi_zerothMomentType);
    MPI_Type_commit(&mPIInfo.mpi_zerothMomentType);

    int block_lengths_firstMoment[3] = {1, 1, 1};
    MPI_Aint offsets_firstMoment[3];
    offsets_firstMoment[0] = offsetof(FirstMoment, x);
    offsets_firstMoment[1] = offsetof(FirstMoment, y);
    offsets_firstMoment[2] = offsetof(FirstMoment, z);
    MPI_Datatype types_firstMoment[3] = {MPI_FLOAT, MPI_FLOAT, MPI_FLOAT};
    MPI_Type_create_struct(3, block_lengths_firstMoment, offsets_firstMoment, types_firstMoment, &mPIInfo.mpi_firstMomentType);
    MPI_Type_commit(&mPIInfo.mpi_firstMomentType);
}


//////////////////////////////////////////////////

void PIC2DMPI::sendrecv_magneticField_x(
    thrust::device_vector<MagneticField>& B, 
    thrust::device_vector<MagneticField>& sendMagneticFieldLeft, 
    thrust::device_vector<MagneticField>& sendMagneticFieldRight, 
    thrust::device_vector<MagneticField>& recvMagneticFieldLeft, 
    thrust::device_vector<MagneticField>& recvMagneticFieldRight, 
    MPIInfo& mPIInfo
)
{
    PIC2DMPI::sendrecv_field_x(
        B, 
        sendMagneticFieldLeft, sendMagneticFieldRight, 
        recvMagneticFieldLeft, recvMagneticFieldRight, 
        mPIInfo, mPIInfo.mpi_fieldType
    );
}


void PIC2DMPI::sendrecv_electricField_x(
    thrust::device_vector<ElectricField>& E, 
    thrust::device_vector<ElectricField>& sendElectricFieldLeft, 
    thrust::device_vector<ElectricField>& sendElectricFieldRight, 
    thrust::device_vector<ElectricField>& recvElectricFieldLeft, 
    thrust::device_vector<ElectricField>& recvElectricFieldRight,
    MPIInfo& mPIInfo
)
{
    PIC2DMPI::sendrecv_field_x(
        E, 
        sendElectricFieldLeft, sendElectricFieldRight, 
        recvElectricFieldLeft, recvElectricFieldRight, 
        mPIInfo, mPIInfo.mpi_fieldType
    );
}


void PIC2DMPI::sendrecv_currentField_x(
    thrust::device_vector<CurrentField>& current,  
    thrust::device_vector<CurrentField>& sendCurrentFieldLeft, 
    thrust::device_vector<CurrentField>& sendCurrentFieldRight, 
    thrust::device_vector<CurrentField>& recvCurrentFieldLeft, 
    thrust::device_vector<CurrentField>& recvCurrentFieldRight, 
    MPIInfo& mPIInfo
)
{
    PIC2DMPI::sendrecv_field_x(
        current, 
        sendCurrentFieldLeft, sendCurrentFieldRight, 
        recvCurrentFieldLeft, recvCurrentFieldRight, 
        mPIInfo, mPIInfo.mpi_fieldType
    );
}


void PIC2DMPI::sendrecv_zerothMoment_x(
    thrust::device_vector<ZerothMoment>& zerothMoment,  
    thrust::device_vector<ZerothMoment>& sendZerothMomentLeft, 
    thrust::device_vector<ZerothMoment>& sendZerothMomentRight, 
    thrust::device_vector<ZerothMoment>& recvZerothMomentLeft, 
    thrust::device_vector<ZerothMoment>& recvZerothMomentRight, 
    MPIInfo& mPIInfo
)
{
    PIC2DMPI::sendrecv_field_x(
        zerothMoment, 
        sendZerothMomentLeft, sendZerothMomentRight, 
        recvZerothMomentLeft, recvZerothMomentRight, 
        mPIInfo, mPIInfo.mpi_zerothMomentType
    );
}


void PIC2DMPI::sendrecv_firstMoment_x(
    thrust::device_vector<FirstMoment>& firstMoment,  
    thrust::device_vector<FirstMoment>& sendFirstMomentLeft, 
    thrust::device_vector<FirstMoment>& sendFirstMomentRight, 
    thrust::device_vector<FirstMoment>& recvFirstMomentLeft, 
    thrust::device_vector<FirstMoment>& recvFirstMomentRight, 
    MPIInfo& mPIInfo
)
{
    PIC2DMPI::sendrecv_field_x(
        firstMoment, 
        sendFirstMomentLeft, sendFirstMomentRight, 
        recvFirstMomentLeft, recvFirstMomentRight, 
        mPIInfo, mPIInfo.mpi_firstMomentType
    );
}

//////////////////////////////////////////////////

void PIC2DMPI::sendrecv_numParticle_x(
    const unsigned int& numForSendParticlesSpeciesLeft, 
    const unsigned int& numForSendParticlesSpeciesRight, 
    unsigned int& numForRecvParticlesSpeciesLeft, 
    unsigned int& numForRecvParticlesSpeciesRight, 
    MPIInfo& mPIInfo
)
{
    int left  = mPIInfo.getRank(-1);
    int right = mPIInfo.getRank(1);
    MPI_Status st;

    MPI_Sendrecv(
        &(numForSendParticlesSpeciesLeft), 
        1, 
        MPI_UNSIGNED,  
        left, 0, 
        &(numForRecvParticlesSpeciesRight), 
        1, 
        MPI_UNSIGNED, 
        right, 0, 
        MPI_COMM_WORLD, &st
    );

    MPI_Sendrecv(
        &(numForSendParticlesSpeciesRight), 
        1, 
        MPI_UNSIGNED, 
        right, 0, 
        &(numForRecvParticlesSpeciesLeft), 
        1, 
        MPI_UNSIGNED, 
        left, 0, 
        MPI_COMM_WORLD, &st
    );
}


void PIC2DMPI::sendrecv_particle_x(
    thrust::device_vector<Particle>& sendParticlesSpeciesLeft, 
    thrust::device_vector<Particle>& sendParticlesSpeciesRight, 
    thrust::device_vector<Particle>& recvParticlesSpeciesLeft, 
    thrust::device_vector<Particle>& recvParticlesSpeciesRight, 
    const unsigned int& numForSendParticlesSpeciesLeft, 
    const unsigned int& numForSendParticlesSpeciesRight, 
    const unsigned int& numForRecvParticlesSpeciesLeft, 
    const unsigned int& numForRecvParticlesSpeciesRight, 
    MPIInfo& mPIInfo
)
{
    int left  = mPIInfo.getRank(-1);
    int right = mPIInfo.getRank(1);
    MPI_Status st;


    unsigned int maxNumSendLeftRecvRightForProcs = max(
        numForSendParticlesSpeciesLeft, 
        numForRecvParticlesSpeciesRight 
    );
    unsigned int maxNumSendRightRecvLeftForProcs = max(
        numForSendParticlesSpeciesRight, 
        numForRecvParticlesSpeciesLeft
    );

    unsigned int maxNumSendLeftRecvRight = 0, maxNumSendRightRecvLeft = 0;
    MPI_Allreduce(&maxNumSendLeftRecvRightForProcs, &maxNumSendLeftRecvRight, 1, MPI_UNSIGNED, MPI_MAX, MPI_COMM_WORLD);
    MPI_Allreduce(&maxNumSendRightRecvLeftForProcs, &maxNumSendRightRecvLeft, 1, MPI_UNSIGNED, MPI_MAX, MPI_COMM_WORLD);

    MPI_Sendrecv(
        thrust::raw_pointer_cast(sendParticlesSpeciesLeft.data()), 
        numForSendParticlesSpeciesLeft, 
        mPIInfo.mpi_particleType, 
        left, 0, 
        thrust::raw_pointer_cast(recvParticlesSpeciesRight.data()), 
        numForRecvParticlesSpeciesRight, 
        mPIInfo.mpi_particleType, 
        right, 0, 
        MPI_COMM_WORLD, &st
    );

    MPI_Sendrecv(
        thrust::raw_pointer_cast(sendParticlesSpeciesRight.data()), 
        numForSendParticlesSpeciesRight, 
        mPIInfo.mpi_particleType, 
        right, 0, 
        thrust::raw_pointer_cast(recvParticlesSpeciesLeft.data()),
        numForRecvParticlesSpeciesLeft, 
        mPIInfo.mpi_particleType, 
        left, 0, 
        MPI_COMM_WORLD, &st
    );
}

