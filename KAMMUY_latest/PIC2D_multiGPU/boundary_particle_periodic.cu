#include "hip/hip_runtime.h"
#include "boundary.hpp"


void BoundaryPIC::periodicBoundaryParticle_x(
    thrust::device_vector<Particle>& particlesIon, 
    thrust::device_vector<Particle>& particlesElectron
)
{   
    MPI_Barrier(MPI_COMM_WORLD); 
    periodicBoundaryParticleOfOneSpecies_x(
        particlesIon,
        mPIInfo.existNumIonPerProcs, 
        mPIInfo.numForSendParticlesIonLeft, 
        mPIInfo.numForSendParticlesIonRight, 
        mPIInfo.numForRecvParticlesIonLeft, 
        mPIInfo.numForRecvParticlesIonRight
    );
    MPI_Barrier(MPI_COMM_WORLD); 
    periodicBoundaryParticleOfOneSpecies_x(
        particlesElectron, 
        mPIInfo.existNumElectronPerProcs,  
        mPIInfo.numForSendParticlesElectronLeft, 
        mPIInfo.numForSendParticlesElectronRight, 
        mPIInfo.numForRecvParticlesElectronLeft, 
        mPIInfo.numForRecvParticlesElectronRight
    );
    MPI_Barrier(MPI_COMM_WORLD);
}


__global__ void periodicBoundaryParticle_x_kernel(
    Particle* particlesSpecies, 
    Particle* sendParticlesSpeciesLeft, 
    Particle* sendParticlesSpeciesRight, 
    unsigned int* countForSendParticlesSpeciesLeft, 
    unsigned int* countForSendParticlesSpeciesRight, 
    const unsigned long long existNumSpecies, 
    const float xminForProcs, const float xmaxForProcs, 
    const int buffer
)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < existNumSpecies) {

        float boundaryLeft  = xminForProcs + PIC2DConst::device_EPS; 
        float boundaryRight = xmaxForProcs - PIC2DConst::device_EPS; 
        
        if (x <= boundaryLeft) {
            particlesSpecies[i].isExist = false;
            return;
        }
        if (x >= boundaryRight) {
            particlesSpecies[i].isExist = false;
            return;
        }

        if (x > boundaryLeft && x < boundaryLeft + buffer * PIC2DConst::device_dx - PIC2DConst::device_EPS) {
            unsigned int particleIndex = atomicAdd(&(countForSendParticlesSpeciesRight[0]), 1);
            particlesSpecies[i].isMPISendRight = false;
            Particle sendParticle = particlesSpecies[i];
            if (sendParticle.x > PIC2DConst::device_xmax - buffer * PIC2DConst::device_dx + PIC2DConst::device_EPS) {
                sendParticle.x = sendParticle.x - PIC2DConst::device_xmax + PIC2DConst::device_EPS;
            }
            sendParticlesSpeciesRight[particleIndex] = sendParticle;
        }

        if (x < boundaryRight && x > boundaryRight - buffer * PIC2DConst::device_dx + PIC2DConst::device_EPS) {
            unsigned int particleIndex = atomicAdd(&(countForSendParticlesSpeciesLeft[0]), 1);
            particlesSpecies[i].isMPISendLeft = false;
            Particle sendParticle = particlesSpecies[i];
            if (sendParticle.x < PIC2DConst::device_xmin + buffer * PIC2DConst::device_dx - PIC2DConst::device_EPS) {
                sendParticle.x = sendParticle.x + PIC2DConst::device_xmax - PIC2DConst::device_EPS;
            }
            sendParticlesSpeciesLeft[particleIndex] = sendParticle;
        }
    }
}

void BoundaryPIC::periodicBoundaryParticleOfOneSpecies_x(
    thrust::device_vector<Particle>& particlesSpecies, 
    unsigned long long& existNumSpecies, 
    unsigned int& numForSendParticlesSpeciesLeft, 
    unsigned int& numForSendParticlesSpeciesRight, 
    unsigned int& numForRecvParticlesSpeciesLeft, 
    unsigned int& numForRecvParticlesSpeciesRight
)
{
    thrust::device_vector<unsigned int> countForSendParticlesSpeciesLeft(1, 0); 
    thrust::device_vector<unsigned int> countForSendParticlesSpeciesRight(1, 0); 

    dim3 threadsPerBlock(256);
    dim3 blocksPerGrid((existNumSpecies + threadsPerBlock.x - 1) / threadsPerBlock.x);

    periodicBoundaryParticle_x_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(particlesSpecies.data()), 
        thrust::raw_pointer_cast(sendParticlesSpeciesLeft.data()), 
        thrust::raw_pointer_cast(sendParticlesSpeciesRight.data()), 
        thrust::raw_pointer_cast(countForSendParticlesSpeciesLeft.data()), 
        thrust::raw_pointer_cast(countForSendParticlesSpeciesRight.data()), 
        existNumSpecies, 
        mPIInfo.xminForProcs, mPIInfo.xmaxForProcs, 
        mPIInfo.buffer
    );
    hipDeviceSynchronize();

    auto partitionEnd = thrust::partition(
        particlesSpecies.begin(), particlesSpecies.end(), 
        [] __device__ (const Particle& p) { return p.isExist; }
    );
    existNumSpecies = thrust::distance(particlesSpecies.begin(), partitionEnd);

    numForSendParticlesSpeciesLeft  = countForSendParticlesSpeciesLeft[0];
    numForSendParticlesSpeciesRight = countForSendParticlesSpeciesRight[0];

    PIC2DMPI::sendrecv_numParticle_x(
        numForSendParticlesSpeciesLeft, 
        numForSendParticlesSpeciesRight, 
        numForRecvParticlesSpeciesLeft, 
        numForRecvParticlesSpeciesRight, 
        mPIInfo
    );

    PIC2DMPI::sendrecv_particle_x(
        sendParticlesSpeciesLeft, 
        sendParticlesSpeciesRight,  
        recvParticlesSpeciesLeft, 
        recvParticlesSpeciesRight,  
        numForSendParticlesSpeciesLeft, 
        numForSendParticlesSpeciesRight, 
        numForRecvParticlesSpeciesLeft, 
        numForRecvParticlesSpeciesRight, 
        mPIInfo
    );

    thrust::copy(
        recvParticlesSpeciesLeft.begin(), 
        recvParticlesSpeciesLeft.begin() + numForRecvParticlesSpeciesLeft,
        particlesSpecies.begin() + existNumSpecies
    );
    existNumSpecies += numForRecvParticlesSpeciesLeft;
    thrust::copy(
        recvParticlesSpeciesRight.begin(), 
        recvParticlesSpeciesRight.begin() + numForRecvParticlesSpeciesRight,
        particlesSpecies.begin() + existNumSpecies
    );
    existNumSpecies += numForRecvParticlesSpeciesRight;

}


