#include "hip/hip_runtime.h"
#include "boundary.hpp"


void BoundaryPIC::periodicBoundaryParticle_x(
    thrust::device_vector<Particle>& particlesIon, 
    thrust::device_vector<Particle>& particlesElectron
)
{   
    MPI_Barrier(MPI_COMM_WORLD); 
    periodicBoundaryParticleOfOneSpecies_x(
        particlesIon,
        mPIInfo.existNumIonPerProcs, 
        mPIInfo.numForSendParticlesIonLeft, 
        mPIInfo.numForSendParticlesIonRight, 
        mPIInfo.numForRecvParticlesIonLeft, 
        mPIInfo.numForRecvParticlesIonRight
    );
    MPI_Barrier(MPI_COMM_WORLD); 
    periodicBoundaryParticleOfOneSpecies_x(
        particlesElectron, 
        mPIInfo.existNumElectronPerProcs,  
        mPIInfo.numForSendParticlesElectronLeft, 
        mPIInfo.numForSendParticlesElectronRight, 
        mPIInfo.numForRecvParticlesElectronLeft, 
        mPIInfo.numForRecvParticlesElectronRight
    );
    MPI_Barrier(MPI_COMM_WORLD);

    if (mPIInfo.existNumIonPerProcs > mPIInfo.totalNumIonPerProcs) std::cout << "BROKEN" << std::endl;
    if (mPIInfo.existNumElectronPerProcs > mPIInfo.totalNumElectronPerProcs) std::cout << "BROKEN" << std::endl;
}


__global__ void periodicBoundaryParticle_x_kernel(
    Particle* particlesSpecies, 
    Particle* sendParticlesSpeciesLeft, 
    Particle* sendParticlesSpeciesRight, 
    unsigned int* countForSendParticlesSpeciesLeft, 
    unsigned int* countForSendParticlesSpeciesRight, 
    const unsigned long long existNumSpecies, 
    const float xminForProcs, const float xmaxForProcs, 
    const int buffer
)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < existNumSpecies) {
        float x = particlesSpecies[i].x; 

        float boundaryLeft  = xminForProcs + PIC2DConst::device_EPS; 
        float boundaryRight = xmaxForProcs - PIC2DConst::device_EPS; 

        if (x <= boundaryLeft) {
            particlesSpecies[i].isExist = false;
            return;
        }
        if (x >= boundaryRight) {
            particlesSpecies[i].isExist = false;
            return;
        }

        if (x > boundaryLeft && x <= boundaryLeft + buffer * PIC2DConst::device_dx) {
            unsigned int particleIndex = atomicAdd(&(countForSendParticlesSpeciesLeft[0]), 1);
            Particle sendParticle = particlesSpecies[i];
            if (sendParticle.x < PIC2DConst::device_xmin + buffer * PIC2DConst::device_dx) {
                sendParticle.x = sendParticle.x + PIC2DConst::device_xmax;
            }
            sendParticlesSpeciesLeft[particleIndex] = sendParticle;
        }

        if (x < boundaryRight && x >= boundaryRight - buffer * PIC2DConst::device_dx) {
            unsigned int particleIndex = atomicAdd(&(countForSendParticlesSpeciesRight[0]), 1);
            Particle sendParticle = particlesSpecies[i];
            if (sendParticle.x > PIC2DConst::device_xmax - buffer * PIC2DConst::device_dx) {
                sendParticle.x = sendParticle.x - PIC2DConst::device_xmax;
            }
            sendParticlesSpeciesRight[particleIndex] = sendParticle;
        }
    }
}

void BoundaryPIC::periodicBoundaryParticleOfOneSpecies_x(
    thrust::device_vector<Particle>& particlesSpecies, 
    unsigned long long& existNumSpecies, 
    unsigned int& numForSendParticlesSpeciesLeft, 
    unsigned int& numForSendParticlesSpeciesRight, 
    unsigned int& numForRecvParticlesSpeciesLeft, 
    unsigned int& numForRecvParticlesSpeciesRight
)
{
    thrust::device_vector<unsigned int> countForSendParticlesSpeciesLeft(1, 0); 
    thrust::device_vector<unsigned int> countForSendParticlesSpeciesRight(1, 0); 

    dim3 threadsPerBlock(256);
    dim3 blocksPerGrid((existNumSpecies + threadsPerBlock.x - 1) / threadsPerBlock.x);

    periodicBoundaryParticle_x_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(particlesSpecies.data()), 
        thrust::raw_pointer_cast(sendParticlesSpeciesLeft.data()), 
        thrust::raw_pointer_cast(sendParticlesSpeciesRight.data()), 
        thrust::raw_pointer_cast(countForSendParticlesSpeciesLeft.data()), 
        thrust::raw_pointer_cast(countForSendParticlesSpeciesRight.data()), 
        existNumSpecies, 
        mPIInfo.xminForProcs, mPIInfo.xmaxForProcs, 
        mPIInfo.buffer
    );
    hipDeviceSynchronize();
    
    auto partitionEnd = thrust::partition(
        particlesSpecies.begin(), particlesSpecies.end(), 
        [] __device__ (const Particle& p) { return p.isExist; }
    );
    existNumSpecies = static_cast<unsigned long long>(thrust::distance(particlesSpecies.begin(), partitionEnd));

    numForSendParticlesSpeciesLeft  = countForSendParticlesSpeciesLeft[0];
    numForSendParticlesSpeciesRight = countForSendParticlesSpeciesRight[0];

    PIC2DMPI::sendrecv_numParticle_x(
        numForSendParticlesSpeciesLeft, 
        numForSendParticlesSpeciesRight, 
        numForRecvParticlesSpeciesLeft, 
        numForRecvParticlesSpeciesRight, 
        mPIInfo
    );

    PIC2DMPI::sendrecv_particle_x(
        sendParticlesSpeciesLeft, 
        sendParticlesSpeciesRight,  
        recvParticlesSpeciesLeft, 
        recvParticlesSpeciesRight,  
        numForSendParticlesSpeciesLeft, 
        numForSendParticlesSpeciesRight, 
        numForRecvParticlesSpeciesLeft, 
        numForRecvParticlesSpeciesRight, 
        mPIInfo
    );

    thrust::copy(
        recvParticlesSpeciesLeft.begin(), 
        recvParticlesSpeciesLeft.begin() + numForRecvParticlesSpeciesLeft,
        particlesSpecies.begin() + existNumSpecies
    );
    existNumSpecies += numForRecvParticlesSpeciesLeft;
    thrust::copy(
        recvParticlesSpeciesRight.begin(), 
        recvParticlesSpeciesRight.begin() + numForRecvParticlesSpeciesRight,
        particlesSpecies.begin() + existNumSpecies
    );
    existNumSpecies += numForRecvParticlesSpeciesRight;
}


