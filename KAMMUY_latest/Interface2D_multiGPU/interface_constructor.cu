#include "hip/hip_runtime.h"
#include "interface.hpp"


__global__ void initializeReloadParticlesSource_kernel(
    Particle* reloadParticlesSourceSpecies, 
    unsigned long long reloadParticlesNumSpecies, 
    int seed
)
{
    unsigned long long i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < reloadParticlesNumSpecies) {
        hiprandState state;
        hiprand_init(seed, i, 0, &state);

        float x, y, z, vx, vy, vz;
        
        x = hiprand_uniform(&state);
        y = hiprand_uniform(&state);
        z = 0.0f;
        vx = hiprand_normal(&state);
        vy = hiprand_normal(&state);
        vz = hiprand_normal(&state);

        x = min(max(x, PIC2DConst::device_EPS), 1.0f - PIC2DConst::device_EPS); 
        y = min(max(y, PIC2DConst::device_EPS), 1.0f - PIC2DConst::device_EPS); 

        reloadParticlesSourceSpecies[i].x  = x;
        reloadParticlesSourceSpecies[i].y  = y;
        reloadParticlesSourceSpecies[i].z  = z;
        reloadParticlesSourceSpecies[i].vx = vx;
        reloadParticlesSourceSpecies[i].vy = vy;
        reloadParticlesSourceSpecies[i].vz = vz;
    }
}


Interface2D::Interface2D(
    IdealMHD2DMPI::MPIInfo& mPIInfoMHD, 
    PIC2DMPI::MPIInfo& mPIInfoPIC, 
    Interface2DMPI::MPIInfo& mPIInfoInterface, 
    int indexOfInterfaceStartMHD, 
    thrust::host_vector<double>& host_interlockingFunctionY, 
    InterfaceNoiseRemover2D& interfaceNoiseRemover2D
)
    : mPIInfoMHD(mPIInfoMHD), 
      mPIInfoPIC(mPIInfoPIC), 
      mPIInfoInterface(mPIInfoInterface), 

      indexOfInterfaceStartInMHD(indexOfInterfaceStartMHD), 

      interlockingFunctionY(mPIInfoPIC.localSizeX * PIC2DConst::ny, 0.0), 

      B_timeAve                   (mPIInfoPIC.localSizeX * PIC2DConst::ny), 
      zerothMomentIon_timeAve     (mPIInfoPIC.localSizeX * PIC2DConst::ny), 
      zerothMomentElectron_timeAve(mPIInfoPIC.localSizeX * PIC2DConst::ny), 
      firstMomentIon_timeAve      (mPIInfoPIC.localSizeX * PIC2DConst::ny), 
      firstMomentElectron_timeAve (mPIInfoPIC.localSizeX * PIC2DConst::ny),

      restartParticlesIndexIon(0), 
      restartParticlesIndexElectron(0), 

      reloadParticlesSourceIon     (Interface2DConst::reloadParticlesTotalNum), 
      reloadParticlesSourceElectron(Interface2DConst::reloadParticlesTotalNum), 

      reloadParticlesDataIon     (mPIInfoPIC.localNx * PIC2DConst::ny), 
      reloadParticlesDataElectron(mPIInfoPIC.localNx * PIC2DConst::ny), 
      
      B_PICtoMHD                   (mPIInfoMHD.localNx * (PIC2DConst::ny / Interface2DConst::gridSizeRatio)), 
      zerothMomentIon_PICtoMHD     (mPIInfoMHD.localNx * (PIC2DConst::ny / Interface2DConst::gridSizeRatio)), 
      zerothMomentElectron_PICtoMHD(mPIInfoMHD.localNx * (PIC2DConst::ny / Interface2DConst::gridSizeRatio)), 
      firstMomentIon_PICtoMHD      (mPIInfoMHD.localNx * (PIC2DConst::ny / Interface2DConst::gridSizeRatio)), 
      firstMomentElectron_PICtoMHD (mPIInfoMHD.localNx * (PIC2DConst::ny / Interface2DConst::gridSizeRatio)), 

      USub (mPIInfoMHD.localSizeX * IdealMHD2DConst::ny), 
      UHalf(mPIInfoMHD.localSizeX * IdealMHD2DConst::ny), 

      momentCalculator(mPIInfoPIC), 
      boundaryPIC(mPIInfoPIC), 
      interfaceNoiseRemover2D(interfaceNoiseRemover2D)
{

    hipMalloc(&device_mPIInfoMHD, sizeof(IdealMHD2DMPI::MPIInfo));
    hipMemcpy(device_mPIInfoMHD, &mPIInfoMHD, sizeof(IdealMHD2DMPI::MPIInfo), hipMemcpyHostToDevice);
    hipMalloc(&device_mPIInfoPIC, sizeof(PIC2DMPI::MPIInfo));
    hipMemcpy(device_mPIInfoPIC, &mPIInfoPIC, sizeof(PIC2DMPI::MPIInfo), hipMemcpyHostToDevice);
    hipMalloc(&device_mPIInfoInterface, sizeof(Interface2DMPI::MPIInfo));
    hipMemcpy(device_mPIInfoInterface, &mPIInfoInterface, sizeof(Interface2DMPI::MPIInfo), hipMemcpyHostToDevice);

    interlockingFunctionY = host_interlockingFunctionY;
    

    dim3 threadsPerBlock(256);
    dim3 blocksPerGrid((Interface2DConst::reloadParticlesTotalNum + threadsPerBlock.x - 1) / threadsPerBlock.x);

    initializeReloadParticlesSource_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(reloadParticlesSourceIon.data()),
        Interface2DConst::reloadParticlesTotalNum, 
        10000000 + 100 * mPIInfoPIC.rank
    );
    hipDeviceSynchronize();

    initializeReloadParticlesSource_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(reloadParticlesSourceElectron.data()),
        Interface2DConst::reloadParticlesTotalNum, 
        20000000 + 100 * mPIInfoPIC.rank
    );
    hipDeviceSynchronize();
}

