#include "hip/hip_runtime.h"
#include "interface.hpp"


__global__ void setUHalf_kernel(
    const ConservationParameter* UPast, 
    const ConservationParameter* UNext, 
    ConservationParameter* UHalf, 
    int localSizeXMHD
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < localSizeXMHD && j < IdealMHD2DConst::device_ny) {
        int index = j + i * IdealMHD2DConst::device_ny;

        UHalf[index] = 0.5 * (UPast[index] + UNext[index]);
    }
}


__global__ void sendPICtoMHD_kernel(
    const double* interlockingFunctionY, 
    const ZerothMoment* zerothMomentIon, 
    const ZerothMoment* ZerothMomentElectron, 
    const FirstMoment* firstMomentIon, 
    const FirstMoment* firstMomentElectron, 
    const MagneticField* B, 
    ConservationParameter* U, 
    int indexOfInterfaceStartInMHD, 
    int indexOfInterfaceStartInPIC, 
    int buffer, 
    int localSizeXPIC, 
    int localSizeXMHD, 
    int interfaceSizeX
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (0 < i && i < interfaceSizeX - 1 && 0 < j && j < Interface2DConst::device_interfaceLength - 1) {
        int indexPIC = indexOfInterfaceStartInPIC + j + i * PIC2DConst::device_ny;
        int indexMHD = indexOfInterfaceStartInMHD + j + i * IdealMHD2DConst::device_ny;
        double rhoMHD, uMHD, vMHD, wMHD, bXMHD, bYMHD, bZMHD, eMHD, pMHD;
        double rhoPIC, uPIC, vPIC, wPIC, bXPIC, bYPIC, bZPIC;
        double niMHD, neMHD, tiMHD, teMHD;
        double mIon = PIC2DConst::device_mIon, mElectron = PIC2DConst::device_mElectron;

        //MHDのグリッドにPICを合わせる
        rhoMHD      = max(U[indexMHD].rho, IdealMHD2DConst::device_rho0 * 0.1);
        uMHD        = U[indexMHD].rhoU / (rhoMHD + IdealMHD2DConst::device_EPS);
        vMHD        = U[indexMHD].rhoV / (rhoMHD + IdealMHD2DConst::device_EPS);
        wMHD        = U[indexMHD].rhoW / (rhoMHD + IdealMHD2DConst::device_EPS);
        bXMHD       = U[indexMHD].bX;
        bYMHD       = U[indexMHD].bY;
        bZMHD       = U[indexMHD].bZ;
        eMHD        = U[indexMHD].e;
        pMHD        = (IdealMHD2DConst::device_gamma - 1.0)
                    * (eMHD - 0.5 * rhoMHD * (uMHD * uMHD + vMHD * vMHD + wMHD * wMHD)
                    - 0.5 * (bXMHD * bXMHD + bYMHD * bYMHD + bZMHD * bZMHD));
        pMHD        = max(pMHD, IdealMHD2DConst::device_p0 * 0.1);

        //tiMHD, teMHDはMHDの情報のままにするために、この計算が必要。
        niMHD = rhoMHD / (mIon + mElectron);
        neMHD = niMHD;
        tiMHD = pMHD / 2.0 / niMHD;
        teMHD = pMHD / 2.0 / neMHD;
        
        rhoPIC =  max(mIon * zerothMomentIon[indexPIC].n + mElectron * ZerothMomentElectron[indexPIC].n, IdealMHD2DConst::device_rho0 * 0.1);
        uPIC   = (mIon * firstMomentIon[indexPIC].x  + mElectron * firstMomentElectron[indexPIC].x) / (rhoPIC + PIC2DConst::device_EPS);
        vPIC   = (mIon * firstMomentIon[indexPIC].y  + mElectron * firstMomentElectron[indexPIC].y) / (rhoPIC + PIC2DConst::device_EPS);
        wPIC   = (mIon * firstMomentIon[indexPIC].z  + mElectron * firstMomentElectron[indexPIC].z) / (rhoPIC + PIC2DConst::device_EPS);
        bXPIC  = B[indexPIC].bX; 
        bYPIC  = B[indexPIC].bY; 
        bZPIC  = B[indexPIC].bZ; 

        rhoMHD = interlockingFunctionY[j] * rhoMHD + (1.0 - interlockingFunctionY[j]) * rhoPIC;
        uMHD   = interlockingFunctionY[j] * uMHD   + (1.0 - interlockingFunctionY[j]) * uPIC;
        vMHD   = interlockingFunctionY[j] * vMHD   + (1.0 - interlockingFunctionY[j]) * vPIC;
        wMHD   = interlockingFunctionY[j] * wMHD   + (1.0 - interlockingFunctionY[j]) * wPIC;
        bXMHD  = interlockingFunctionY[j] * bXMHD  + (1.0 - interlockingFunctionY[j]) * bXPIC;
        bYMHD  = interlockingFunctionY[j] * bYMHD  + (1.0 - interlockingFunctionY[j]) * bYPIC;
        bZMHD  = interlockingFunctionY[j] * bZMHD  + (1.0 - interlockingFunctionY[j]) * bZPIC;

        niMHD = rhoMHD / (mIon + mElectron);
        neMHD = niMHD;
        pMHD  = niMHD * tiMHD + neMHD * teMHD;


        U[indexMHD].rho  = rhoMHD;
        U[indexMHD].rhoU = rhoMHD * uMHD;
        U[indexMHD].rhoV = rhoMHD * vMHD;
        U[indexMHD].rhoW = rhoMHD * wMHD;
        U[indexMHD].bX   = bXMHD;
        U[indexMHD].bY   = bYMHD;
        U[indexMHD].bZ   = bZMHD;
        eMHD = pMHD / (IdealMHD2DConst::device_gamma - 1.0)
             + 0.5 * rhoMHD * (uMHD * uMHD + vMHD * vMHD + wMHD * wMHD)
             + 0.5 * (bXMHD * bXMHD + bYMHD * bYMHD + bZMHD * bZMHD);
        U[indexMHD].e = eMHD;
    }
}


//MHDのグリッドを整数格子点上に再配置してから使うこと
void Interface2D::sendPICtoMHD(
    thrust::device_vector<ConservationParameter>& U
)
{

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((localSizeXInterface + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (Interface2DConst::interfaceLength + threadsPerBlock.y - 1) / threadsPerBlock.y);

    sendPICtoMHD_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(interlockingFunctionY.data()), 
        thrust::raw_pointer_cast(zerothMomentIon_timeAve.data()), 
        thrust::raw_pointer_cast(zerothMomentElectron_timeAve.data()), 
        thrust::raw_pointer_cast(firstMomentIon_timeAve.data()), 
        thrust::raw_pointer_cast(firstMomentElectron_timeAve.data()), 
        thrust::raw_pointer_cast(B_timeAve.data()), 
        thrust::raw_pointer_cast(U.data()), 
        indexOfInterfaceStartInMHD, 
        indexOfInterfaceStartInPIC, 
        mPIInfoMHD.buffer, 
        localSizeXPIC, 
        localSizeXMHD, 
        localSizeXInterface
    );
}


