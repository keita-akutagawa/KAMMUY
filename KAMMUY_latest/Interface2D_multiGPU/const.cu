#include "const.hpp"


void Interface2DConst::initializeDeviceConstants()
{
    hipMemcpyToSymbol(HIP_SYMBOL(device_gridSizeRatio), &gridSizeRatio, sizeof(int));

    hipMemcpyToSymbol(HIP_SYMBOL(device_EPS), &EPS, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_PI), &PI, sizeof(double));

    hipMemcpyToSymbol(HIP_SYMBOL(device_interfaceLength), &interfaceLength, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(device_deltaForInterlockingFunction), &deltaForInterlockingFunction, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_indexOfInterfaceStartInMHD), &indexOfInterfaceStartInMHD, sizeof(int));

    hipMemcpyToSymbol(HIP_SYMBOL(device_nx), &nx, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(device_ny), &ny, sizeof(int));

    hipMemcpyToSymbol(HIP_SYMBOL(device_reloadParticlesTotalNum), &reloadParticlesTotalNum, sizeof(unsigned long long));
}


