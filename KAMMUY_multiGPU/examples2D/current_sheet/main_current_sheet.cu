#include "hip/hip_runtime.h"
#include "main_current_sheet_const.hpp"


// 別にinitializeUを作ることにする。
void IdealMHD2D::initializeU()
{
}


__global__ void initializeU_lower_kernel(
    ConservationParameter* U, 
    IdealMHD2DMPI::MPIInfo* device_mPIInfo
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < IdealMHD2DConst::device_nx && j < IdealMHD2DConst::device_ny) {
        IdealMHD2DMPI::MPIInfo mPIInfo = *device_mPIInfo;

        if (mPIInfo.isInside(i, j)) {
            int index = mPIInfo.globalToLocal(i, j);

            double rho, u, v, w, bX, bY, bZ, e, p;
            
            rho = IdealMHD2DConst::device_rho0 * sqrt(device_betaUpstream);
            u   = 0.0;
            v   = 0.0;
            w   = 0.0;
            bX  = -1.0 * IdealMHD2DConst::device_B0;
            bY  = 0.0;
            bZ  = 0.0;
            p   = IdealMHD2DConst::device_p0 * device_betaUpstream;
            e   = p / (IdealMHD2DConst::device_gamma - 1.0)
                + 0.5 * rho * (u * u + v * v + w * w)
                + 0.5 * (bX * bX + bY * bY + bZ * bZ);

            U[index].rho  = rho;
            U[index].rhoU = rho * u;
            U[index].rhoV = rho * v;
            U[index].rhoW = rho * w;
            U[index].bX   = bX;
            U[index].bY   = bY;
            U[index].bZ   = bZ;
            U[index].e    = e;
        }
    }
}


__global__ void initializeU_upper_kernel(
    ConservationParameter* U, 
    IdealMHD2DMPI::MPIInfo* device_mPIInfo
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < IdealMHD2DConst::device_nx && j < IdealMHD2DConst::device_ny) {
        IdealMHD2DMPI::MPIInfo mPIInfo = *device_mPIInfo;

        if (mPIInfo.isInside(i, j)) {
            int index = mPIInfo.globalToLocal(i, j);

            double rho, u, v, w, bX, bY, bZ, e, p;
            
            rho = IdealMHD2DConst::device_rho0 * sqrt(device_betaUpstream);
            u   = 0.0;
            v   = 0.0;
            w   = 0.0;
            bX  = 1.0 * IdealMHD2DConst::device_B0;
            bY  = 0.0;
            bZ  = 0.0;
            p   = IdealMHD2DConst::device_p0 * device_betaUpstream;
            e   = p / (IdealMHD2DConst::device_gamma - 1.0)
                + 0.5 * rho * (u * u + v * v + w * w)
                + 0.5 * (bX * bX + bY * bY + bZ * bZ);

            U[index].rho  = rho;
            U[index].rhoU = rho * u;
            U[index].rhoV = rho * v;
            U[index].rhoW = rho * w;
            U[index].bX   = bX;
            U[index].bY   = bY;
            U[index].bZ   = bZ;
            U[index].e    = e;
        }
    }
}


void initializeU(
    thrust::device_vector<ConservationParameter>& U_lower, 
    thrust::device_vector<ConservationParameter>& U_upper, 
    BoundaryMHD& boundaryMHD, 
    IdealMHD2DMPI::MPIInfo& mPIInfoMHD
)
{
    IdealMHD2DMPI::MPIInfo* device_mPIInfoMHD; 
    hipMalloc(&device_mPIInfoMHD, sizeof(IdealMHD2DMPI::MPIInfo));
    hipMemcpy(device_mPIInfoMHD, &mPIInfoMHD, sizeof(IdealMHD2DMPI::MPIInfo), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((IdealMHD2DConst::nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (IdealMHD2DConst::ny + threadsPerBlock.y - 1) / threadsPerBlock.y);

    initializeU_lower_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(U_lower.data()), 
        device_mPIInfoMHD
    );
    hipDeviceSynchronize();

    initializeU_upper_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(U_upper.data()), 
        device_mPIInfoMHD
    );
    hipDeviceSynchronize();

    MPI_Barrier(MPI_COMM_WORLD);

    IdealMHD2DMPI::sendrecv_U(U_lower, mPIInfoMHD);
    boundaryMHD.periodicBoundaryX2nd_U(U_lower);
    boundaryMHD.symmetricBoundaryY2nd_U(U_lower);
    IdealMHD2DMPI::sendrecv_U(U_upper, mPIInfoMHD);
    boundaryMHD.periodicBoundaryX2nd_U(U_upper);
    boundaryMHD.symmetricBoundaryY2nd_U(U_upper);

    MPI_Barrier(MPI_COMM_WORLD);
}


__global__ void initializePICField_kernel(
    ElectricField* E, MagneticField* B, 
    PIC2DMPI::MPIInfo* device_mPIInfo
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < PIC2DConst::device_nx && j < PIC2DConst::device_ny) {
        PIC2DMPI::MPIInfo mPIInfo = *device_mPIInfo;

        if (mPIInfo.isInside(i, j)) {
            int index = mPIInfo.globalToLocal(i, j);

            float bX, bY, bZ, eX, eY, eZ;
            float x = i * PIC2DConst::device_dx, y = j * PIC2DConst::device_dy;
            float xCenter = 0.5f * (PIC2DConst::device_xmax - PIC2DConst::device_xmin);
            float yCenter = 0.5f * (PIC2DConst::device_ymax - PIC2DConst::device_ymin);

            bX = PIC2DConst::device_B0 * tanh((y - yCenter) / device_sheatThickness)
            - PIC2DConst::device_B0 * device_triggerRatio * (y - yCenter) / device_sheatThickness
            * exp(-(pow((x - xCenter), 2) + pow((y - yCenter), 2))
            / pow(2.0f * device_sheatThickness, 2));;
            bY = PIC2DConst::device_B0 * device_triggerRatio * (x - xCenter) / device_sheatThickness
            * exp(-(pow((x - xCenter), 2) + pow((y - yCenter), 2))
            / pow(2.0f * device_sheatThickness, 2)); 
            bZ = 0.0f;
            eX = 0.0f;
            eY = 0.0f;
            eZ = 0.0f;

            E[index].eX = eX;
            E[index].eY = eY;
            E[index].eZ = eZ;
            B[index].bX = bX;
            B[index].bY = bY; 
            B[index].bZ = bZ;
        }
    }
}

void PIC2D::initialize()
{
    unsigned long long harrisNumIonPerProcs = harrisNumIon / mPIInfo.procs; 
    unsigned long long harrisNumElectronPerProcs = harrisNumElectron / mPIInfo.procs; 

    initializeParticle.uniformForPosition_x(
        0, mPIInfo.existNumIonPerProcs, 
        mPIInfo.xminForProcs, mPIInfo.xmaxForProcs, 
        0 + mPIInfo.rank, particlesIon
    );
    initializeParticle.uniformForPosition_x(
        0, mPIInfo.existNumElectronPerProcs, 
        mPIInfo.xminForProcs, mPIInfo.xmaxForProcs, 
        10000 + mPIInfo.rank, particlesElectron
    );

    initializeParticle.harrisForPosition_y(
        0, harrisNumIonPerProcs, 
        20000 + mPIInfo.rank, sheatThickness, particlesIon
    );
    initializeParticle.uniformForPosition_y(
        harrisNumIonPerProcs, mPIInfo.existNumIonPerProcs, 
        PIC2DConst::ymin, PIC2DConst::ymax, 
        30000 + mPIInfo.rank, particlesIon
    );
    initializeParticle.harrisForPosition_y(
        0, harrisNumElectronPerProcs, 
        40000 + mPIInfo.rank, sheatThickness, particlesElectron
    );
    initializeParticle.uniformForPosition_y(
        harrisNumElectronPerProcs, mPIInfo.existNumElectronPerProcs, 
        PIC2DConst::ymin, PIC2DConst::ymax, 
        50000 + mPIInfo.rank, particlesElectron
    );

    initializeParticle.maxwellDistributionForVelocity(
        PIC2DConst::bulkVxIon, PIC2DConst::bulkVyIon, PIC2DConst::bulkVzIon, 
        PIC2DConst::vThIon, PIC2DConst::vThIon, PIC2DConst::vThIon, 
        0, harrisNumIonPerProcs, 
        60000 + mPIInfo.rank, particlesIon
    );
    initializeParticle.maxwellDistributionForVelocity(
        bulkVxIonBackground, bulkVyIonBackground, bulkVzIonBackground, 
        vThIonBackground, vThIonBackground, vThIonBackground, 
        harrisNumIonPerProcs, mPIInfo.existNumIonPerProcs, 
        70000 + mPIInfo.rank, particlesIon
    );
    initializeParticle.maxwellDistributionForVelocity(
        PIC2DConst::bulkVxElectron, PIC2DConst::bulkVyElectron, PIC2DConst::bulkVzElectron, 
        PIC2DConst::vThElectron, PIC2DConst::vThElectron, PIC2DConst::vThElectron, 
        0, harrisNumElectronPerProcs, 
        80000 + mPIInfo.rank, particlesElectron
    );
    initializeParticle.maxwellDistributionForVelocity(
        bulkVxElectronBackground, bulkVyElectronBackground, bulkVzElectronBackground, 
        vThElectronBackground, vThElectronBackground, vThElectronBackground, 
        harrisNumElectronPerProcs, mPIInfo.existNumElectronPerProcs, 
        90000 + mPIInfo.rank, particlesElectron
    );
    

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((PIC2DConst::nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (PIC2DConst::ny + threadsPerBlock.y - 1) / threadsPerBlock.y);

    initializePICField_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(E.data()), 
        thrust::raw_pointer_cast(B.data()), 
        device_mPIInfo
    );
    hipDeviceSynchronize();

    MPI_Barrier(MPI_COMM_WORLD);

    PIC2DMPI::sendrecv_field(B, mPIInfo, mPIInfo.mpi_fieldType);
    PIC2DMPI::sendrecv_field(E, mPIInfo, mPIInfo.mpi_fieldType);
    PIC2DMPI::sendrecv_field(current, mPIInfo, mPIInfo.mpi_fieldType);

    boundaryPIC.periodicBoundaryB_x(B);
    boundaryPIC.freeBoundaryB_y(B);
    boundaryPIC.periodicBoundaryE_x(E);
    boundaryPIC.freeBoundaryE_y(E);
    boundaryPIC.periodicBoundaryCurrent_x(current);
    boundaryPIC.freeBoundaryCurrent_y(current);
    boundaryPIC.periodicBoundaryForInitializeParticle_x(particlesIon, particlesElectron);
    boundaryPIC.freeBoundaryForInitializeParticle_y(particlesIon, particlesElectron);
    
    MPI_Barrier(MPI_COMM_WORLD);
}



int main(int argc, char** argv)
{
    MPI_Init(&argc, &argv);

    int rank = 0, procs = 0;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &procs);
    //int d2[2] = {};
    //MPI_Dims_create(procs, 2, d2);
    int gridX = procs;
    int gridY = 1;

    PIC2DMPI::MPIInfo mPIInfoPIC;
    PIC2DMPI::setupInfo(mPIInfoPIC, buffer, gridX, gridY);
    IdealMHD2DMPI::MPIInfo mPIInfoMHD;
    IdealMHD2DMPI::setupInfo(mPIInfoMHD, buffer, gridX, gridY);
    Interface2DMPI::MPIInfo mPIInfoInterface; 
    Interface2DMPI::setupInfo(mPIInfoInterface, buffer, gridX, gridY); 

    if (mPIInfoPIC.rank == 0) {
        std::cout   << mPIInfoPIC.gridX << "," << mPIInfoPIC.gridY << std::endl;
        mpifile_PIC << mPIInfoPIC.gridX << "," << mPIInfoPIC.gridY << std::endl;
    }
    if (mPIInfoMHD.rank == 0) {
        std::cout   << mPIInfoMHD.gridX << "," << mPIInfoMHD.gridY << std::endl;
        mpifile_MHD << mPIInfoMHD.gridX << "," << mPIInfoMHD.gridY << std::endl;
    }
    if (mPIInfoInterface.rank == 0) {
        std::cout   << mPIInfoInterface.gridX << "," << mPIInfoInterface.gridY << std::endl;
        mpifile_Interface << mPIInfoInterface.gridX << "," << mPIInfoInterface.gridY << std::endl;
    }

    hipSetDevice(mPIInfoPIC.rank);

    PIC2DConst::initializeDeviceConstants();
    IdealMHD2DConst::initializeDeviceConstants();
    Interface2DConst::initializeDeviceConstants();
    hipMemcpyToSymbol(HIP_SYMBOL(device_sheatThickness), &sheatThickness, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_betaUpstream), &betaUpstream, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_triggerRatio), &triggerRatio, sizeof(float));
    
    mPIInfoPIC.existNumIonPerProcs      = static_cast<unsigned long long>(PIC2DConst::totalNumIon / mPIInfoPIC.procs);
    mPIInfoPIC.existNumElectronPerProcs = static_cast<unsigned long long>(PIC2DConst::totalNumElectron / mPIInfoPIC.procs);
    mPIInfoPIC.totalNumIonPerProcs = mPIInfoPIC.existNumIonPerProcs
                                   + PIC2DConst::numberDensityIon * (mPIInfoPIC.localSizeX + mPIInfoPIC.localSizeY) * (2 * mPIInfoPIC.buffer)
                                   + Interface2DConst::reloadParticlesTotalNum;
    mPIInfoPIC.totalNumElectronPerProcs = mPIInfoPIC.existNumElectronPerProcs
                                        + PIC2DConst::numberDensityElectron * (mPIInfoPIC.localSizeX + mPIInfoPIC.localSizeY) * (2 * mPIInfoPIC.buffer)
                                        + Interface2DConst::reloadParticlesTotalNum;

    mPIInfoPIC.xminForProcs = PIC2DConst::xmin + (PIC2DConst::xmax - PIC2DConst::xmin) / mPIInfoPIC.gridX * mPIInfoPIC.localGridX;
    mPIInfoPIC.xmaxForProcs = PIC2DConst::xmin + (PIC2DConst::xmax - PIC2DConst::xmin) / mPIInfoPIC.gridX * (mPIInfoPIC.localGridX + 1);
    mPIInfoPIC.yminForProcs = PIC2DConst::ymin + (PIC2DConst::ymax - PIC2DConst::ymin) / mPIInfoPIC.gridY * mPIInfoPIC.localGridY;
    mPIInfoPIC.ymaxForProcs = PIC2DConst::ymin + (PIC2DConst::ymax - PIC2DConst::ymin) / mPIInfoPIC.gridY * (mPIInfoPIC.localGridY + 1);

    for (int j = 0; j < Interface2DConst::ny; j++) {
        host_interlockingFunctionY_lower[j] = max(
            0.5 * (1.0 + cos(Interface2DConst::PI * (j - 0.0) / (Interface2DConst::ny - 0.0))), 
            Interface2DConst::EPS
        );
        host_interlockingFunctionY_upper[j] = max(
            0.5 * (1.0 - cos(Interface2DConst::PI * (j - 0.0) / (Interface2DConst::ny - 0.0))), 
            Interface2DConst::EPS
        );
    }
    for (int j = 0; j < Interface2DConst::ny; j++) {
        host_interlockingFunctionYHalf_lower[j] = max(
            0.5 * (1.0 + cos(Interface2DConst::PI * (j + 0.5 - 0.0) / (Interface2DConst::ny - 0.0))), 
            Interface2DConst::EPS
        );
        host_interlockingFunctionYHalf_upper[j] = max(
            0.5 * (1.0 - cos(Interface2DConst::PI * (j + 0.5 - 0.0) / (Interface2DConst::ny - 0.0))), 
            Interface2DConst::EPS
        );
    }

    IdealMHD2D idealMHD2D_lower(mPIInfoMHD);
    IdealMHD2D idealMHD2D_upper(mPIInfoMHD);
    PIC2D pIC2D(mPIInfoPIC); 
    InterfaceNoiseRemover2D interfaceNoiseRemover2D_lower( 
        mPIInfoMHD, mPIInfoPIC, 
        indexOfConvolutionStartInMHD_lowerInterface, 
        indexOfConvolutionStartInPIC_lowerInterface, 
        convolutionSizeX, convolutionSizeY 
    );
    InterfaceNoiseRemover2D interfaceNoiseRemover2D_upper( 
        mPIInfoMHD, mPIInfoPIC, 
        indexOfConvolutionStartInMHD_upperInterface, 
        indexOfConvolutionStartInPIC_upperInterface, 
        convolutionSizeX, convolutionSizeY 
    );
    Interface2D interface2D_lower(
        mPIInfoMHD, mPIInfoPIC, mPIInfoInterface, 
        indexOfInterfaceStartInMHD_lower, 
        indexOfInterfaceStartInPIC_lower, 
        host_interlockingFunctionY_lower, 
        host_interlockingFunctionYHalf_lower, 
        interfaceNoiseRemover2D_lower
    );
    Interface2D interface2D_upper(
        mPIInfoMHD, mPIInfoPIC, mPIInfoInterface, 
        indexOfInterfaceStartInMHD_upper, 
        indexOfInterfaceStartInPIC_upper, 
        host_interlockingFunctionY_upper, 
        host_interlockingFunctionYHalf_upper,
        interfaceNoiseRemover2D_upper
    );
    BoundaryMHD boundaryMHD(mPIInfoMHD);

    
    if (mPIInfoPIC.rank == 0) {
        size_t free_mem = 0;
        size_t total_mem = 0;
        hipError_t status = hipMemGetInfo(&free_mem, &total_mem);

        std::cout << "Free memory: " << free_mem / (1024 * 1024) << " MB" << std::endl;
        std::cout << "Total memory: " << total_mem / (1024 * 1024) << " MB" << std::endl;

        std::cout << "exist number of partices is " 
                  << mPIInfoPIC.procs * (mPIInfoPIC.existNumIonPerProcs + mPIInfoPIC.existNumElectronPerProcs) 
                  << std::endl;
        std::cout << "exist number of partices + buffer particles is " 
                  << mPIInfoPIC.procs * (mPIInfoPIC.totalNumIonPerProcs + mPIInfoPIC.totalNumElectronPerProcs) 
                  << std::endl;
    }

    thrust::device_vector<ConservationParameter>& U_lower = idealMHD2D_lower.getURef();
    thrust::device_vector<ConservationParameter>& U_upper = idealMHD2D_upper.getURef();

    initializeU(U_lower, U_upper, boundaryMHD, mPIInfoMHD);
    pIC2D.initialize();


    const int totalSubstep = int(round(sqrt(PIC2DConst::mRatio)));
    for (int step = 0; step < IdealMHD2DConst::totalStep + 1; step++) {
        MPI_Barrier(MPI_COMM_WORLD);

        if (mPIInfoPIC.rank == 0) {
            if (step % 10 == 0) {
                std::cout << std::to_string(step) << " step done : total time is "
                        << std::setprecision(4) << step * totalSubstep * PIC2DConst::dt * PIC2DConst::omegaPe
                        << " [omega_pe * t]"
                        << std::endl;
            }
        }

        if (step % recordStep == 0) {
            logfile << std::setprecision(6) << PIC2DConst::totalTime << std::endl;
            pIC2D.saveFields(
                directoryName, filenameWithoutStep, step
            );
            pIC2D.saveZerothMoments(
                directoryName, filenameWithoutStep, step
            );
            pIC2D.saveFirstMoments(
                directoryName, filenameWithoutStep, step
            );
            idealMHD2D_lower.save(
                directoryName, filenameWithoutStep + "_U_lower", step
            );
            idealMHD2D_upper.save(
                directoryName, filenameWithoutStep + "_U_upper", step
            );
        }
        if (isParticleRecord && step % particleRecordStep == 0) {
            pIC2D.saveParticle(
                directoryName, filenameWithoutStep, step
            );
        }


        // STEP1 : MHD - predictor
        
        idealMHD2D_lower.calculateDt();
        double dt_lower = IdealMHD2DConst::dt;
        idealMHD2D_upper.calculateDt();
        double dt_upper = IdealMHD2DConst::dt;
        double dtCommon = min(min(dt_lower / totalSubstep, dt_upper / totalSubstep), min(0.7 * PIC2DConst::c, 0.1 * 1.0 / PIC2DConst::omegaPe));
        PIC2DConst::dt = dtCommon;
        IdealMHD2DConst::dt = totalSubstep * dtCommon;

        idealMHD2D_lower.setPastU();
        idealMHD2D_upper.setPastU();
        thrust::device_vector<ConservationParameter>& UPast_lower = idealMHD2D_lower.getUPastRef();
        thrust::device_vector<ConservationParameter>& UPast_upper = idealMHD2D_upper.getUPastRef();

        idealMHD2D_lower.oneStepRK2_periodicXSymmetricY_predictor();
        idealMHD2D_upper.oneStepRK2_periodicXSymmetricY_predictor();

        thrust::device_vector<ConservationParameter>& UNext_lower = idealMHD2D_lower.getURef();
        thrust::device_vector<ConservationParameter>& UNext_upper = idealMHD2D_upper.getURef();


        // STEP2 : PIC

        interface2D_lower.resetTimeAveParameters();
        interface2D_upper.resetTimeAveParameters();

        for (int substep = 1; substep <= totalSubstep; substep++) {
            pIC2D.oneStep_periodicXFreeY(
                UPast_lower, UPast_upper, 
                UNext_lower, UNext_upper, 
                interface2D_lower, interface2D_upper, 
                interfaceNoiseRemover2D_lower, interfaceNoiseRemover2D_upper, 
                step, substep, totalSubstep
            );

            thrust::device_vector<MagneticField>& B = pIC2D.getBRef();
            thrust::device_vector<Particle>& particlesIon = pIC2D.getParticlesIonRef();
            thrust::device_vector<Particle>& particlesElectron = pIC2D.getParticlesElectronRef();

            interface2D_lower.sumUpTimeAveParameters(B, particlesIon, particlesElectron);
            interface2D_upper.sumUpTimeAveParameters(B, particlesIon, particlesElectron);
        }

        interface2D_lower.calculateTimeAveParameters(totalSubstep);
        interface2D_upper.calculateTimeAveParameters(totalSubstep);


        // STEP3 : MHD - corrector
        
        interface2D_lower.sendPICtoMHD(UPast_lower, UNext_lower);
        interface2D_upper.sendPICtoMHD(UPast_upper, UNext_upper);
        thrust::device_vector<ConservationParameter>& UHalf_lower = interface2D_lower.getUHalfRef();
        thrust::device_vector<ConservationParameter>& UHalf_upper = interface2D_upper.getUHalfRef();

        IdealMHD2DMPI::sendrecv_U(UHalf_lower, mPIInfoMHD);
        boundaryMHD.periodicBoundaryX2nd_U(UHalf_lower);
        boundaryMHD.symmetricBoundaryY2nd_U(UHalf_lower);
        IdealMHD2DMPI::sendrecv_U(UHalf_upper, mPIInfoMHD);
        boundaryMHD.periodicBoundaryX2nd_U(UHalf_upper);
        boundaryMHD.symmetricBoundaryY2nd_U(UHalf_upper);

        idealMHD2D_lower.oneStepRK2_periodicXSymmetricY_corrector(UHalf_lower);
        idealMHD2D_upper.oneStepRK2_periodicXSymmetricY_corrector(UHalf_upper);

        U_lower = idealMHD2D_lower.getURef();
        U_upper = idealMHD2D_upper.getURef();
        for (int count = 0; count < Interface2DConst::convolutionCount; count++) {
            interfaceNoiseRemover2D_lower.convolveU(U_lower);
            interfaceNoiseRemover2D_upper.convolveU(U_upper);

            IdealMHD2DMPI::sendrecv_U_x(U_lower, mPIInfoMHD);
            boundaryMHD.periodicBoundaryX2nd_U(U_lower);
            boundaryMHD.symmetricBoundaryY2nd_U(U_lower);
            IdealMHD2DMPI::sendrecv_U_x(U_upper, mPIInfoMHD);
            boundaryMHD.periodicBoundaryX2nd_U(U_upper);
            boundaryMHD.symmetricBoundaryY2nd_U(U_upper);
        }

        //when crashed 
        if (idealMHD2D_lower.checkCalculationIsCrashed() || idealMHD2D_upper.checkCalculationIsCrashed()) {
            logfile << std::setprecision(6) << PIC2DConst::totalTime << std::endl;
            pIC2D.saveFields(
                directoryName, filenameWithoutStep, step
            );
            pIC2D.saveZerothMoments(
                directoryName, filenameWithoutStep, step
            );
            pIC2D.saveFirstMoments(
                directoryName, filenameWithoutStep, step
            );
            idealMHD2D_lower.save(
                directoryName, filenameWithoutStep + "_U_lower", step
            );
            idealMHD2D_upper.save(
                directoryName, filenameWithoutStep + "_U_upper", step
            );
            std::cout << "Calculation stopped! : " << step << " steps" << std::endl;
            break;
        }

        if (mPIInfoMHD.rank == 0) {
            IdealMHD2DConst::totalTime += IdealMHD2DConst::dt;
        }   
    }

    MPI_Finalize();

    if (mPIInfoMHD.rank == 0) {
        std::cout << "program was completed!" << std::endl;
    }

    return 0;
}


