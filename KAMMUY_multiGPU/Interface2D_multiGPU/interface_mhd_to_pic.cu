#include "hip/hip_runtime.h"
#include "interface.hpp"


__global__ void sendMHDtoPIC_magneticField_y_kernel(
    const double* interlockingFunctionY, 
    const double* interlockingFunctionYHalf, 
    const ConservationParameter* U, 
    MagneticField* B, 
    int indexOfInterfaceStartInMHD, 
    int indexOfInterfaceStartInPIC, 
    int localSizeXPIC, int localSizeYPIC, 
    int localSizeXMHD, int localSizeYMHD, 
    int localSizeXInterface, int localSizeYInterface
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (0 < i && i < localSizeXInterface - 1 && 0 < j && j < localSizeYInterface - 1) {
        double bXPIC, bYPIC, bZPIC;
        double bXMHD, bYMHD, bZMHD;
        double bXInterface, bYInterface, bZInterface;

        int indexPIC = indexOfInterfaceStartInPIC + j + i * localSizeYPIC;
        int indexMHD = indexOfInterfaceStartInMHD + j + i * localSizeYMHD;

        //PICのグリッドにMHDを合わせる
        bXPIC = B[indexPIC].bX;
        bYPIC = B[indexPIC].bY;
        bZPIC = B[indexPIC].bZ;
        bXMHD = 0.25 * (U[indexMHD].bX + U[indexMHD - localSizeYMHD].bX + U[indexMHD + 1].bX + U[indexMHD + 1 - localSizeYMHD].bX);
        bYMHD = 0.25 * (U[indexMHD].bY + U[indexMHD + localSizeYMHD].bY + U[indexMHD - 1].bY + U[indexMHD - 1 + localSizeYMHD].bY);
        bZMHD = 0.25 * (U[indexMHD].bZ + U[indexMHD + localSizeYMHD].bZ + U[indexMHD + 1].bZ + U[indexMHD + 1 + localSizeYMHD].bZ);

        bXInterface = interlockingFunctionYHalf[j] * bXMHD + (1.0 - interlockingFunctionYHalf[j]) * bXPIC;
        bYInterface = interlockingFunctionY[j]     * bYMHD + (1.0 - interlockingFunctionY[j])     * bYPIC;
        bZInterface = interlockingFunctionYHalf[j] * bZMHD + (1.0 - interlockingFunctionYHalf[j]) * bZPIC;
        
        B[indexPIC].bX = bXInterface;
        B[indexPIC].bY = bYInterface;
        B[indexPIC].bZ = bZInterface;
    }
}

void Interface2D::sendMHDtoPIC_magneticField_y(
    const thrust::device_vector<ConservationParameter>& U, 
    thrust::device_vector<MagneticField>& B
)
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((localSizeXInterface + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (localSizeYInterface + threadsPerBlock.y - 1) / threadsPerBlock.y);

    sendMHDtoPIC_magneticField_y_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(interlockingFunctionY.data()),
        thrust::raw_pointer_cast(interlockingFunctionYHalf.data()),
        thrust::raw_pointer_cast(U.data()), 
        thrust::raw_pointer_cast(B.data()), 
        indexOfInterfaceStartInMHD, 
        indexOfInterfaceStartInPIC, 
        localSizeXPIC, localSizeYPIC,  
        localSizeXMHD, localSizeYMHD, 
        localSizeXInterface, localSizeYInterface
    );
    hipDeviceSynchronize();
}


__global__ void sendMHDtoPIC_electricField_y_kernel(
    const double* interlockingFunctionY, 
    const double* interlockingFunctionYHalf, 
    const ConservationParameter* U, 
    ElectricField* E, 
    int indexOfInterfaceStartInMHD, 
    int indexOfInterfaceStartInPIC, 
    int localSizeXPIC, int localSizeYPIC, 
    int localSizeXMHD, int localSizeYMHD, 
    int localSizeXInterface, int localSizeYInterface
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (0 < i && i < localSizeXInterface - 1 && 0 < j && j < localSizeYInterface - 1) {
        double eXPIC, eYPIC, eZPIC;
        double eXMHD, eYMHD, eZMHD;
        double eXPlusX1MHD;
        double eYPlusY1MHD;
        double rho, u, v, w;
        double bXMHD, bYMHD, bZMHD;
        double eXInterface, eYInterface, eZInterface;
        double mIon = PIC2DConst::device_mIon, mElectron = PIC2DConst::device_mElectron; 

        int indexPIC = indexOfInterfaceStartInPIC + j + i * localSizeYPIC;
        int indexMHD = indexOfInterfaceStartInMHD + j + i * localSizeYMHD;

        //PICのグリッドにMHDを合わせる
        eXPIC = E[indexPIC].eX;
        eYPIC = E[indexPIC].eY;
        eZPIC = E[indexPIC].eZ;

        rho = max(U[indexMHD].rho, mIon * 1 + mElectron * 1);
        u = U[indexMHD].rhoU / (rho + IdealMHD2DConst::device_EPS);
        v = U[indexMHD].rhoV / (rho + IdealMHD2DConst::device_EPS);
        w = U[indexMHD].rhoW / (rho + IdealMHD2DConst::device_EPS); 
        bXMHD = 0.5 * (U[indexMHD].bX + U[indexMHD - localSizeYMHD].bX);
        bYMHD = 0.5 * (U[indexMHD].bY + U[indexMHD - 1].bY);
        bZMHD = U[indexMHD].bZ;
        eXMHD = -(v * bZMHD - w * bYMHD);
        eYMHD = -(w * bXMHD - u * bZMHD);
        eZMHD = -(u * bYMHD - v * bXMHD);

        rho = max(U[indexMHD + localSizeYMHD].rho, mIon * 1 + mElectron * 1);
        u = U[indexMHD + localSizeYMHD].rhoU / (rho + IdealMHD2DConst::device_EPS);
        v = U[indexMHD + localSizeYMHD].rhoV / (rho + IdealMHD2DConst::device_EPS);
        w = U[indexMHD + localSizeYMHD].rhoW / (rho + IdealMHD2DConst::device_EPS); 
        bXMHD = 0.5 * (U[indexMHD + localSizeYMHD].bX + U[indexMHD].bX);
        bYMHD = 0.5 * (U[indexMHD + localSizeYMHD].bY + U[indexMHD - 1 + localSizeYMHD].bY);
        bZMHD = U[indexMHD + localSizeYMHD].bZ;
        eXPlusX1MHD = -(v * bZMHD - w * bYMHD);

        rho = max(U[indexMHD + 1].rho, mIon * 1 + mElectron * 1);
        u = U[indexMHD + 1].rhoU / (rho + IdealMHD2DConst::device_EPS);
        v = U[indexMHD + 1].rhoV / (rho + IdealMHD2DConst::device_EPS);
        w = U[indexMHD + 1].rhoW / (rho + IdealMHD2DConst::device_EPS); 
        bXMHD = 0.5 * (U[indexMHD + 1].bX + U[indexMHD + 1 - localSizeYMHD].bX);
        bYMHD = 0.5 * (U[indexMHD + 1].bY + U[indexMHD].bY);
        bZMHD = U[indexMHD + 1].bZ;
        eYPlusY1MHD = -(w * bXMHD - u * bZMHD);

        eXInterface = interlockingFunctionY[j]     * 0.5 * (eXMHD + eXPlusX1MHD) + (1.0 - interlockingFunctionY[j])     * eXPIC;
        eYInterface = interlockingFunctionYHalf[j] * 0.5 * (eYMHD + eYPlusY1MHD) + (1.0 - interlockingFunctionYHalf[j]) * eYPIC;
        eZInterface = interlockingFunctionY[j]     * eZMHD                       + (1.0 - interlockingFunctionY[j])     * eZPIC;
         
        E[indexPIC].eX = eXInterface;
        E[indexPIC].eY = eYInterface;
        E[indexPIC].eZ = eZInterface;
    }
}

void Interface2D::sendMHDtoPIC_electricField_y(
    const thrust::device_vector<ConservationParameter>& U, 
    thrust::device_vector<ElectricField>& E
)
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((localSizeXInterface + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (localSizeYInterface + threadsPerBlock.y - 1) / threadsPerBlock.y);

    sendMHDtoPIC_electricField_y_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(interlockingFunctionY.data()), 
        thrust::raw_pointer_cast(interlockingFunctionYHalf.data()), 
        thrust::raw_pointer_cast(U.data()), 
        thrust::raw_pointer_cast(E.data()), 
        indexOfInterfaceStartInMHD, 
        indexOfInterfaceStartInPIC, 
        localSizeXPIC, localSizeYPIC,  
        localSizeXMHD, localSizeYMHD, 
        localSizeXInterface, localSizeYInterface
    );
    hipDeviceSynchronize();
}


__global__ void sendMHDtoPIC_currentField_y_kernel(
    const double* interlockingFunctionY, 
    const double* interlockingFunctionYHalf, 
    const ConservationParameter* U, 
    CurrentField* current, 
    int indexOfInterfaceStartInMHD, 
    int indexOfInterfaceStartInPIC, 
    int localSizeXPIC, int localSizeYPIC, 
    int localSizeXMHD, int localSizeYMHD, 
    int localSizeXInterface, int localSizeYInterface
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (0 < i && i < localSizeXInterface - 1 && 0 < j && j < localSizeYInterface - 1) {
        double jXPIC, jYPIC, jZPIC;
        double jXMHD, jYMHD, jZMHD;
        double jXPlusX1MHD; 
        double jYPlusY1MHD; 
        double jXInterface, jYInterface, jZInterface;
        double dx = IdealMHD2DConst::device_dx, dy = IdealMHD2DConst::device_dy;

        int indexPIC = indexOfInterfaceStartInPIC + j + i * localSizeYPIC;
        int indexMHD = indexOfInterfaceStartInMHD + j + i * localSizeYMHD;

        //PICのグリッドにMHDを合わせる
        jXPIC = current[indexPIC].jX;
        jYPIC = current[indexPIC].jY;
        jZPIC = current[indexPIC].jZ;
        jXMHD = (U[indexMHD + 1].bZ - U[indexMHD - 1].bZ) / (2.0 * dy);
        jYMHD = -(U[indexMHD + localSizeYMHD].bZ - U[indexMHD - localSizeYMHD].bZ) / (2.0 * dx);
        jZMHD = 0.25 * (
                (U[indexMHD + localSizeYMHD].bY - U[indexMHD].bY) / dx - (U[indexMHD + 1].bX - U[indexMHD].bX) / dy 
              + (U[indexMHD].bY - U[indexMHD - localSizeYMHD].bY) / dx - (U[indexMHD + 1 - localSizeYMHD].bX - U[indexMHD - localSizeYMHD].bX) / dy
              + (U[indexMHD - 1 + localSizeYMHD].bY - U[indexMHD - 1].bY) / dx - (U[indexMHD].bX - U[indexMHD - 1].bX) / dy
              + (U[indexMHD - 1].bY - U[indexMHD - 1 - localSizeYMHD].bY) / dx - (U[indexMHD - localSizeYMHD].bX - U[indexMHD - 1 - localSizeYMHD].bX) / dy);

        jXPlusX1MHD = (U[indexMHD + 1 + localSizeYMHD].bZ - U[indexMHD - 1 + localSizeYMHD].bZ) / (2.0 * dy);
        jYPlusY1MHD = -(U[indexMHD + localSizeYMHD + 1].bZ - U[indexMHD - localSizeYMHD + 1].bZ) / (2.0 * dx);

        jXInterface = interlockingFunctionY[j]     * 0.5 * (jXMHD + jXPlusX1MHD) + (1.0 - interlockingFunctionY[j])     * jXPIC;
        jYInterface = interlockingFunctionYHalf[j] * 0.5 * (jYMHD + jYPlusY1MHD) + (1.0 - interlockingFunctionYHalf[j]) * jYPIC;
        jZInterface = interlockingFunctionY[j]     * jZMHD                       + (1.0 - interlockingFunctionY[j])     * jZPIC;
        
        current[indexPIC].jX = jXInterface;
        current[indexPIC].jY = jYInterface;
        current[indexPIC].jZ = jZInterface;
    }
}

void Interface2D::sendMHDtoPIC_currentField_y(
    const thrust::device_vector<ConservationParameter>& U, 
    thrust::device_vector<CurrentField>& current
)
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((localSizeXInterface + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (localSizeYInterface + threadsPerBlock.y - 1) / threadsPerBlock.y);

    sendMHDtoPIC_currentField_y_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(interlockingFunctionY.data()), 
        thrust::raw_pointer_cast(interlockingFunctionYHalf.data()), 
        thrust::raw_pointer_cast(U.data()), 
        thrust::raw_pointer_cast(current.data()), 
        indexOfInterfaceStartInMHD, 
        indexOfInterfaceStartInPIC, 
        localSizeXPIC, localSizeYPIC,  
        localSizeXMHD, localSizeYMHD, 
        localSizeXInterface, localSizeYInterface
    );
    hipDeviceSynchronize();
}


__global__ void sendMHDtoPIC_particle_y_kernel(
    const double* interlockingFunctionY, 
    const ZerothMoment* zerothMomentIon, 
    const ZerothMoment* zerothMomentElectron, 
    const FirstMoment* firstMomentIon, 
    const FirstMoment* firstMomentElectron, 
    const ConservationParameter* U, 
    ReloadParticlesData* reloadParticlesDataIon, 
    ReloadParticlesData* reloadParticlesDataElectron, 
    int indexOfInterfaceStartInMHD, 
    int indexOfInterfaceStartInPIC, 
    int localSizeXPIC, int localSizeYPIC, 
    int localSizeXMHD, int localSizeYMHD, 
    int localSizeXInterface, int localSizeYInterface
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (0 < i && i < localSizeXInterface - 1 && 0 < j && j < localSizeYInterface - 1) {
        int indexForReload = j + i * localSizeYInterface;  
        int indexPIC = indexOfInterfaceStartInPIC + j + i * localSizeYPIC;
        int indexMHD = indexOfInterfaceStartInMHD + j + i * localSizeYMHD;
        double rhoMHD, uMHD, vMHD, wMHD, bXMHD, bYMHD, bZMHD, eMHD, pMHD;
        double jXMHD, jYMHD, jZMHD, niMHD, neMHD, tiMHD, teMHD;
        double rhoPIC, uPIC, vPIC, wPIC;
        double jXPIC, jYPIC, jZPIC, niPIC, nePIC, vThiPIC, vThePIC;
        double dx = IdealMHD2DConst::device_dx, dy = IdealMHD2DConst::device_dy;
        double mIon = PIC2DConst::device_mIon, mElectron = PIC2DConst::device_mElectron;
        double qIon = PIC2DConst::device_qIon, qElectron = PIC2DConst::device_qElectron;

        //整数格子点上で計算する。リロードに使う。
        rhoMHD = max(U[indexMHD].rho, mIon * 1 + mElectron * 1);
        uMHD   = U[indexMHD].rhoU / (rhoMHD + IdealMHD2DConst::device_EPS);
        vMHD   = U[indexMHD].rhoV / (rhoMHD + IdealMHD2DConst::device_EPS);
        wMHD   = U[indexMHD].rhoW / (rhoMHD + IdealMHD2DConst::device_EPS);
        bXMHD  = 0.5 * (U[indexMHD].bX + U[indexMHD - localSizeYMHD].bX);
        bYMHD  = 0.5 * (U[indexMHD].bY + U[indexMHD - 1].bY);
        bZMHD  = U[indexMHD].bZ;
        eMHD   = U[indexMHD].e;
        pMHD   = (IdealMHD2DConst::device_gamma - 1.0)
               * (eMHD - 0.5 * rhoMHD * (uMHD * uMHD + vMHD * vMHD + wMHD * wMHD)
               - 0.5 * (bXMHD * bXMHD + bYMHD * bYMHD + bZMHD * bZMHD));
        pMHD   = max(pMHD, IdealMHD2DConst::device_EPS);
        jXMHD  = (U[indexMHD + 1].bZ - U[indexMHD - 1].bZ) / (2.0 * dy);
        jYMHD  = -(U[indexMHD + localSizeYMHD].bZ - U[indexMHD - localSizeYMHD].bZ) / (2.0 * dx);
        jZMHD  = 0.25 * (
                 (U[indexMHD + localSizeYMHD].bY - U[indexMHD].bY) / dx - (U[indexMHD + 1].bX - U[indexMHD].bX) / dy 
               + (U[indexMHD].bY - U[indexMHD - localSizeYMHD].bY) / dx - (U[indexMHD + 1 - localSizeYMHD].bX - U[indexMHD - localSizeYMHD].bX) / dy
               + (U[indexMHD - 1 + localSizeYMHD].bY - U[indexMHD - 1].bY) / dx - (U[indexMHD].bX - U[indexMHD - 1].bX) / dy
               + (U[indexMHD - 1].bY - U[indexMHD - 1 - localSizeYMHD].bY) / dx - (U[indexMHD - localSizeYMHD].bX - U[indexMHD - 1 - localSizeYMHD].bX) / dy);

        niMHD = rhoMHD / (mIon + mElectron);
        neMHD = niMHD;
        tiMHD = pMHD / 2.0 / niMHD;
        teMHD = pMHD / 2.0 / neMHD;

        rhoPIC =  max(mIon * zerothMomentIon[indexPIC].n + mElectron * zerothMomentElectron[indexPIC].n, mIon * 1 + mElectron * 1);
        uPIC   = (mIon * firstMomentIon[indexPIC].x  + mElectron * firstMomentElectron[indexPIC].x) / (rhoPIC + PIC2DConst::device_EPS);
        vPIC   = (mIon * firstMomentIon[indexPIC].y  + mElectron * firstMomentElectron[indexPIC].y) / (rhoPIC + PIC2DConst::device_EPS);
        wPIC   = (mIon * firstMomentIon[indexPIC].z  + mElectron * firstMomentElectron[indexPIC].z) / (rhoPIC + PIC2DConst::device_EPS);
        jXPIC  = qIon  * firstMomentIon[indexPIC].x  + qElectron * firstMomentElectron[indexPIC].x;
        jYPIC  = qIon  * firstMomentIon[indexPIC].y  + qElectron * firstMomentElectron[indexPIC].y;
        jZPIC  = qIon  * firstMomentIon[indexPIC].z  + qElectron * firstMomentElectron[indexPIC].z;

        rhoPIC = interlockingFunctionY[j] * rhoMHD + (1.0 - interlockingFunctionY[j]) * rhoPIC;
        uPIC   = interlockingFunctionY[j] * uMHD   + (1.0 - interlockingFunctionY[j]) * uPIC;
        vPIC   = interlockingFunctionY[j] * vMHD   + (1.0 - interlockingFunctionY[j]) * vPIC;
        wPIC   = interlockingFunctionY[j] * wMHD   + (1.0 - interlockingFunctionY[j]) * wPIC;
        jXPIC  = interlockingFunctionY[j] * jXMHD  + (1.0 - interlockingFunctionY[j]) * jXPIC;
        jYPIC  = interlockingFunctionY[j] * jYMHD  + (1.0 - interlockingFunctionY[j]) * jYPIC;
        jZPIC  = interlockingFunctionY[j] * jZMHD  + (1.0 - interlockingFunctionY[j]) * jZPIC;

        niPIC   = rhoPIC / (mIon + mElectron);
        nePIC   = niPIC;
        vThiPIC = sqrt(2.0 * tiMHD / mIon);
        vThePIC = sqrt(2.0 * teMHD / mElectron);


        reloadParticlesDataIon     [indexForReload].numAndIndex = max(static_cast<unsigned long long>(round(niPIC)), static_cast<unsigned long long>(1));
        reloadParticlesDataElectron[indexForReload].numAndIndex = max(static_cast<unsigned long long>(round(nePIC)), static_cast<unsigned long long>(1));
        reloadParticlesDataIon     [indexForReload].u              = uPIC;
        reloadParticlesDataIon     [indexForReload].v              = vPIC;
        reloadParticlesDataIon     [indexForReload].w              = wPIC;
        reloadParticlesDataElectron[indexForReload].u              = uPIC - jXPIC / max(round(nePIC), 1.0) / abs(qElectron);
        reloadParticlesDataElectron[indexForReload].v              = vPIC - jYPIC / max(round(nePIC), 1.0) / abs(qElectron);
        reloadParticlesDataElectron[indexForReload].w              = wPIC - jZPIC / max(round(nePIC), 1.0) / abs(qElectron);
        reloadParticlesDataIon     [indexForReload].vth            = vThiPIC;
        reloadParticlesDataElectron[indexForReload].vth            = vThePIC;

        if (j == 1) {
            reloadParticlesDataIon[indexForReload - 1]      = reloadParticlesDataIon[indexForReload];
            reloadParticlesDataElectron[indexForReload - 1] = reloadParticlesDataElectron[indexForReload];
        }
        if (j == localSizeYInterface - 2) {
            reloadParticlesDataIon[indexForReload + 1]      = reloadParticlesDataIon[indexForReload];
            reloadParticlesDataElectron[indexForReload + 1] = reloadParticlesDataElectron[indexForReload];
        }
    }
}


__global__ void deleteParticles_kernel(
    const double* interlockingFunctionY, 
    Particle* particlesSpecies, 
    const int indexOfInterfaceStartInPIC, 
    const unsigned long long existNumSpecies, 
    int seed, 
    const float xminForProcs, const float xmaxForProcs, 
    const int buffer, 
    int localSizeXInterface, int localSizeYInterface
)
{
    unsigned long long i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < existNumSpecies) {
        float x = particlesSpecies[i].x;
        float y = particlesSpecies[i].y;
        float deleteXMin = xminForProcs - buffer * PIC2DConst::device_dx;
        float deleteXMax = xmaxForProcs + buffer * PIC2DConst::device_dx;
        float deleteYMin = indexOfInterfaceStartInPIC * PIC2DConst::device_dy;
        float deleteYMax = (indexOfInterfaceStartInPIC + localSizeYInterface) * PIC2DConst::device_dy;

        if (deleteXMin < x && x < deleteXMax && deleteYMin < y && y < deleteYMax) {
            int j = floorf(y - deleteYMin);
            hiprandState state; 
            hiprand_init(seed, i, 0, &state);
            float randomValue = hiprand_uniform(&state);
            if (randomValue < interlockingFunctionY[j]) {
                particlesSpecies[i].isExist = false;
            }
        }
    }
}


void Interface2D::deleteParticlesSpecies(
    thrust::device_vector<Particle>& particlesSpecies, 
    unsigned long long& existNumSpeciesPerProcs, 
    int seed
)
{

    dim3 threadsPerBlock(256);
    dim3 blocksPerGrid((existNumSpeciesPerProcs + threadsPerBlock.x - 1) / threadsPerBlock.x);
    
    deleteParticles_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(interlockingFunctionY.data()), 
        thrust::raw_pointer_cast(particlesSpecies.data()),
        indexOfInterfaceStartInPIC, 
        existNumSpeciesPerProcs, 
        seed, 
        mPIInfoPIC.xminForProcs, mPIInfoPIC.xmaxForProcs, 
        mPIInfoPIC.buffer, 
        localSizeXInterface, localSizeYInterface
    );
    hipDeviceSynchronize();

    auto partitionEnd = thrust::partition(
        particlesSpecies.begin(), particlesSpecies.end(), 
        [] __device__ (const Particle& p) { return p.isExist; }
    );
    hipDeviceSynchronize();

    existNumSpeciesPerProcs = thrust::distance(particlesSpecies.begin(), partitionEnd);
}


__global__ void reloadParticlesSpecies_kernel(
    const double* interlockingFunctionY, 
    const ReloadParticlesData* reloadParticlesDataSpecies, 
    const Particle* reloadParticlesSpecies, 
    unsigned long long reloadParticlesTotalNumSpecies, 
    Particle* particlesSpecies, 
    unsigned long long restartParticlesIndexSpecies, 
    int indexOfInterfaceStartInPIC, 
    unsigned long long existNumSpecies, 
    int step, 
    const float xminForProcs, const float xmaxForProcs, 
    const float yminForProcs, const float ymaxForProcs, 
    int buffer, 
    int localSizeXInterface, int localSizeYInterface
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < localSizeXInterface && j < localSizeYInterface) {
        int index = j + i * localSizeYInterface; 
        float u = reloadParticlesDataSpecies[index].u;
        float v = reloadParticlesDataSpecies[index].v;
        float w = reloadParticlesDataSpecies[index].w;
        float vth = reloadParticlesDataSpecies[index].vth;
        Particle particleSource, particleReload;
        float x, y, z, vx, vy, vz, gamma;

        for (unsigned long long k = reloadParticlesDataSpecies[index].numAndIndex; k < reloadParticlesDataSpecies[index + 1].numAndIndex; k++) {
            hiprandState state; 
            hiprand_init(step, k, 0, &state);
            float randomValue = hiprand_uniform(&state);

            if (randomValue < interlockingFunctionY[j]) {
                particleSource = reloadParticlesSpecies[(restartParticlesIndexSpecies + k) % reloadParticlesTotalNumSpecies];

                x = particleSource.x; x += i * PIC2DConst::device_dx + (xminForProcs - buffer * PIC2DConst::device_dx);
                y = particleSource.y; y += (indexOfInterfaceStartInPIC + j) * PIC2DConst::device_dy;
                z = particleSource.z;
                vx = particleSource.vx; vx = u + vx * vth;
                vy = particleSource.vy; vy = v + vy * vth;
                vz = particleSource.vz; vz = w + vz * vth;
                if (1.0f - (vx * vx + vy * vy + vz * vz) / pow(PIC2DConst::device_c, 2) < 0.0f){
                    printf("particle exceeds light speed... ");
                    continue; //delete if particle speed exceeds light speed c. 
                };
                gamma = 1.0f / sqrt(1.0f - (vx * vx + vy * vy + vz * vz) / pow(PIC2DConst::device_c, 2));

                particleReload.x = x; particleReload.y = y; particleReload.z = z;
                particleReload.vx = vx * gamma; particleReload.vy = vy * gamma, particleReload.vz = vz * gamma; 
                particleReload.gamma = gamma;
                particleReload.isExist = true;

                particlesSpecies[existNumSpecies + k] = particleReload;
            } 
        }
    }
}


void Interface2D::reloadParticlesSpecies(
    thrust::device_vector<Particle>& particlesSpecies, 
    thrust::device_vector<ReloadParticlesData>& reloadParticlesDataSpecies, 
    thrust::device_vector<Particle>& reloadParticlesSourceSpecies, 
    unsigned long long& existNumSpeciesPerProcs, 
    int seed 
)
{
    std::mt19937 genSpecies(seed);
    std::uniform_int_distribution<unsigned long long> distSpecies(0, Interface2DConst::reloadParticlesTotalNum);
    unsigned long long restartParticlesIndexSpecies = distSpecies(genSpecies);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((localSizeXInterface + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (localSizeYInterface + threadsPerBlock.y - 1) / threadsPerBlock.y);

    reloadParticlesSpecies_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(interlockingFunctionY.data()), 
        thrust::raw_pointer_cast(reloadParticlesDataSpecies.data()), 
        thrust::raw_pointer_cast(reloadParticlesSourceSpecies.data()), 
        Interface2DConst::reloadParticlesTotalNum,   
        thrust::raw_pointer_cast(particlesSpecies.data()), 
        restartParticlesIndexSpecies, 
        indexOfInterfaceStartInPIC, 
        existNumSpeciesPerProcs, 
        seed, 
        mPIInfoPIC.xminForProcs, mPIInfoPIC.xmaxForProcs, 
        mPIInfoPIC.yminForProcs, mPIInfoPIC.ymaxForProcs, 
        mPIInfoPIC.buffer, 
        localSizeXInterface, localSizeYInterface
    );
    hipDeviceSynchronize();

    auto partitionEnd = thrust::partition(
        particlesSpecies.begin(), particlesSpecies.end(), 
        [] __device__ (const Particle& p) { return p.isExist; }
    );

    existNumSpeciesPerProcs = thrust::distance(particlesSpecies.begin(), partitionEnd);
}


void Interface2D::sendMHDtoPIC_particle(
    const thrust::device_vector<ConservationParameter>& U, 
    thrust::device_vector<Particle>& particlesIon, 
    thrust::device_vector<Particle>& particlesElectron, 
    int seed
)
{
    setMoments(particlesIon, particlesElectron); 

    for (int count = 0; count < Interface2DConst::convolutionCount; count++) {
        interfaceNoiseRemover2D.convolveMoments(
            zerothMomentIon, zerothMomentElectron, 
            firstMomentIon, firstMomentElectron
        );

        PIC2DMPI::sendrecv_field_x(zerothMomentIon, mPIInfoPIC, mPIInfoPIC.mpi_zerothMomentType);
        PIC2DMPI::sendrecv_field_x(zerothMomentElectron, mPIInfoPIC, mPIInfoPIC.mpi_zerothMomentType);
        PIC2DMPI::sendrecv_field_x(firstMomentIon, mPIInfoPIC, mPIInfoPIC.mpi_firstMomentType);
        PIC2DMPI::sendrecv_field_x(firstMomentElectron, mPIInfoPIC, mPIInfoPIC.mpi_firstMomentType);
    }

    thrust::fill(reloadParticlesDataIon.begin(), reloadParticlesDataIon.end(), ReloadParticlesData());
    thrust::fill(reloadParticlesDataElectron.begin(), reloadParticlesDataElectron.end(), ReloadParticlesData());

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((localSizeXInterface + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (localSizeYInterface + threadsPerBlock.y - 1) / threadsPerBlock.y);

    sendMHDtoPIC_particle_y_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(interlockingFunctionY.data()), 
        thrust::raw_pointer_cast(zerothMomentIon.data()), 
        thrust::raw_pointer_cast(zerothMomentElectron.data()), 
        thrust::raw_pointer_cast(firstMomentIon.data()), 
        thrust::raw_pointer_cast(firstMomentElectron.data()), 
        thrust::raw_pointer_cast(U.data()),  
        thrust::raw_pointer_cast(reloadParticlesDataIon.data()), 
        thrust::raw_pointer_cast(reloadParticlesDataElectron.data()), 
        indexOfInterfaceStartInMHD, 
        indexOfInterfaceStartInPIC, 
        localSizeXPIC, localSizeYPIC,  
        localSizeXMHD, localSizeYMHD, 
        localSizeXInterface, localSizeYInterface
    );
    hipDeviceSynchronize();

    Interface2DMPI::sendrecv_reloadParticlesData_x(reloadParticlesDataIon, mPIInfoInterface);
    Interface2DMPI::sendrecv_reloadParticlesData_x(reloadParticlesDataElectron, mPIInfoInterface);
    
    deleteParticlesSpecies(
        particlesIon, mPIInfoPIC.existNumIonPerProcs, seed
    );
    deleteParticlesSpecies(
        particlesElectron, mPIInfoPIC.existNumElectronPerProcs, seed + 1
    );
    
    host_reloadParticlesDataIon = reloadParticlesDataIon;
    host_reloadParticlesDataElectron = reloadParticlesDataElectron;
    
    for (int i = 0; i < localSizeXInterface; i++) {
        for (int j = 0; j < localSizeYInterface; j++) {
            int index;
            index = j + i * localSizeYInterface;
            host_reloadParticlesDataIon[index + 1].numAndIndex += host_reloadParticlesDataIon[index].numAndIndex;
            host_reloadParticlesDataElectron[index + 1].numAndIndex += host_reloadParticlesDataElectron[index].numAndIndex;
        }
    }
    reloadParticlesDataIon = host_reloadParticlesDataIon;
    reloadParticlesDataElectron = host_reloadParticlesDataElectron;

    reloadParticlesSpecies(
        particlesIon, reloadParticlesDataIon, reloadParticlesSourceIon, 
        mPIInfoPIC.existNumIonPerProcs, seed
    ); 
    reloadParticlesSpecies(
        particlesElectron, reloadParticlesDataElectron, reloadParticlesSourceElectron, 
        mPIInfoPIC.existNumElectronPerProcs, seed + 1
    ); 
}


