#include "const.hpp"


void Interface2DConst::initializeDeviceConstants()
{
    hipMemcpyToSymbol(HIP_SYMBOL(device_EPS), &EPS, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_PI), &PI, sizeof(double));

    hipMemcpyToSymbol(HIP_SYMBOL(device_interfaceLength), &interfaceLength, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(device_windowSizeForConvolution), &windowSizeForConvolution, sizeof(int));

    hipMemcpyToSymbol(HIP_SYMBOL(device_reloadParticlesTotalNum), &reloadParticlesTotalNum, sizeof(unsigned long long));
}


