#include "const.hpp"


void Interface2DConst::initializeDeviceConstants_Interface()
{
    hipMemcpyToSymbol(HIP_SYMBOL(device_EPS), &EPS, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_PI), &PI, sizeof(double));

    hipMemcpyToSymbol(HIP_SYMBOL(device_windowSizeForConvolution), &windowSizeForConvolution, sizeof(int));

    hipMemcpyToSymbol(HIP_SYMBOL(device_reloadParticlesTotalNumIon), &reloadParticlesTotalNumIon, sizeof(unsigned long long));
}


