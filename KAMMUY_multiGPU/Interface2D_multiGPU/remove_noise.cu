#include "hip/hip_runtime.h"
#include "remove_noise.hpp"


InterfaceNoiseRemover2D::InterfaceNoiseRemover2D(
    IdealMHD2DMPI::MPIInfo& mPIInfoMHD, 
    PIC2DMPI::MPIInfo& mPIInfoPIC, 
    int indexOfConvolutionStartInMHD, 
    int indexOfConvolutionStartInPIC, 
    int localSizeXConvolution, int localSizeYConvolution
)
    : mPIInfoMHD(mPIInfoMHD), 
      mPIInfoPIC(mPIInfoPIC), 

      indexOfConvolutionStartInMHD(indexOfConvolutionStartInMHD), 
      indexOfConvolutionStartInPIC(indexOfConvolutionStartInPIC), 
      localSizeXConvolution(localSizeXConvolution), 
      localSizeYConvolution(localSizeYConvolution), 

      tmpB(localSizeXConvolution * localSizeYConvolution), 
      tmpE(localSizeXConvolution * localSizeYConvolution), 
      tmpCurrent(localSizeXConvolution * localSizeYConvolution), 
      tmpZerothMoment(localSizeXConvolution * localSizeYConvolution), 
      tmpFirstMoment(localSizeXConvolution * localSizeYConvolution), 
      tmpU(localSizeXConvolution * localSizeYConvolution)
{

    hipMalloc(&device_mPIInfoMHD, sizeof(IdealMHD2DMPI::MPIInfo));
    hipMemcpy(device_mPIInfoMHD, &mPIInfoMHD, sizeof(IdealMHD2DMPI::MPIInfo), hipMemcpyHostToDevice);
    hipMalloc(&device_mPIInfoPIC, sizeof(PIC2DMPI::MPIInfo));
    hipMemcpy(device_mPIInfoPIC, &mPIInfoPIC, sizeof(PIC2DMPI::MPIInfo), hipMemcpyHostToDevice);
    
}


template <typename FieldType>
__global__ void copyFieldsPIC_kernel(
    const FieldType* field, 
    FieldType* tmpField, 
    int indexOfConvolutionStartInPIC, 
    int localSizeXConvolution, int localSizeYConvolution, 
    int localSizeXPIC, int localSizeYPIC
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < localSizeXConvolution && j < localSizeYConvolution) {
        int indexForCopy = j + i * localSizeYConvolution;
        int indexPIC = indexOfConvolutionStartInPIC + j + i * localSizeYPIC;

        tmpField[indexForCopy] = field[indexPIC];
    }
}


template <typename FieldType>
__global__ void convolveFields_kernel(
    const FieldType* tmpField, 
    FieldType* field, 
    int indexOfConvolutionStartInPIC, 
    int localSizeXConvolution, int localSizeYConvolution, 
    int localSizeXPIC, int localSizeYPIC
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (2 <= i && i <= localSizeXConvolution - 3 && 2 <= j && j <= localSizeYConvolution - 3) {
        int indexForCopy = j + i * localSizeYConvolution;
        int indexPIC = indexOfConvolutionStartInPIC + j + i * localSizeYPIC;
        
        FieldType convolvedField; 

        for (int windowX = -1; windowX <= 1; windowX++) {
            for (int windowY = -1; windowY <= 1; windowY++) {
                int localIndex; 
                localIndex = indexForCopy + windowY + windowX * localSizeYConvolution; 
                convolvedField = convolvedField + tmpField[localIndex];
            }
        }
        convolvedField = 1.0 / 9.0 * convolvedField; 
        
        field[indexPIC] = convolvedField;
    }
}


void InterfaceNoiseRemover2D::convolve_magneticField(
    thrust::device_vector<MagneticField>& B
)
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((localSizeXConvolution + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (localSizeYConvolution + threadsPerBlock.y - 1) / threadsPerBlock.y);

    copyFieldsPIC_kernel<MagneticField><<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(B.data()), 
        thrust::raw_pointer_cast(tmpB.data()), 
        indexOfConvolutionStartInPIC, 
        localSizeXConvolution, localSizeYConvolution, 
        mPIInfoPIC.localSizeX, mPIInfoPIC.localSizeY
    );
    hipDeviceSynchronize();
    
    convolveFields_kernel<MagneticField><<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(tmpB.data()), 
        thrust::raw_pointer_cast(B.data()), 
        indexOfConvolutionStartInPIC, 
        localSizeXConvolution, localSizeYConvolution, 
        mPIInfoPIC.localSizeX, mPIInfoPIC.localSizeY
    );
    hipDeviceSynchronize();

}


void InterfaceNoiseRemover2D::convolve_electricField(
    thrust::device_vector<ElectricField>& E
)
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((localSizeXConvolution + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (localSizeYConvolution + threadsPerBlock.y - 1) / threadsPerBlock.y);

    copyFieldsPIC_kernel<ElectricField><<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(E.data()), 
        thrust::raw_pointer_cast(tmpE.data()), 
        indexOfConvolutionStartInPIC, 
        localSizeXConvolution, localSizeYConvolution, 
        mPIInfoPIC.localSizeX, mPIInfoPIC.localSizeY
    );
    hipDeviceSynchronize();

    convolveFields_kernel<ElectricField><<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(tmpE.data()), 
        thrust::raw_pointer_cast(E.data()), 
        indexOfConvolutionStartInPIC, 
        localSizeXConvolution, localSizeYConvolution, 
        mPIInfoPIC.localSizeX, mPIInfoPIC.localSizeY
    );
    hipDeviceSynchronize();

}


void InterfaceNoiseRemover2D::convolve_currentField(
    thrust::device_vector<CurrentField>& current
)
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((localSizeXConvolution + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (localSizeYConvolution + threadsPerBlock.y - 1) / threadsPerBlock.y);

    copyFieldsPIC_kernel<CurrentField><<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(current.data()), 
        thrust::raw_pointer_cast(tmpCurrent.data()), 
        indexOfConvolutionStartInPIC, 
        localSizeXConvolution, localSizeYConvolution, 
        mPIInfoPIC.localSizeX, mPIInfoPIC.localSizeY
    );
    hipDeviceSynchronize();
    
    convolveFields_kernel<CurrentField><<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(tmpCurrent.data()), 
        thrust::raw_pointer_cast(current.data()), 
        indexOfConvolutionStartInPIC, 
        localSizeXConvolution, localSizeYConvolution, 
        mPIInfoPIC.localSizeX, mPIInfoPIC.localSizeY
    );
    hipDeviceSynchronize();

}


void InterfaceNoiseRemover2D::convolveMomentsOfOneSpecies(
    thrust::device_vector<ZerothMoment>& zerothMomentOfOneSpecies, 
    thrust::device_vector<FirstMoment>& firstMomentOfOneSpecies
)
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((localSizeXConvolution + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (localSizeYConvolution + threadsPerBlock.y - 1) / threadsPerBlock.y);

    copyFieldsPIC_kernel<ZerothMoment><<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(zerothMomentOfOneSpecies.data()), 
        thrust::raw_pointer_cast(tmpZerothMoment.data()), 
        indexOfConvolutionStartInPIC, 
        localSizeXConvolution, localSizeYConvolution, 
        mPIInfoPIC.localSizeX, mPIInfoPIC.localSizeY
    );
    hipDeviceSynchronize();

    convolveFields_kernel<ZerothMoment><<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(tmpZerothMoment.data()), 
        thrust::raw_pointer_cast(zerothMomentOfOneSpecies.data()), 
        indexOfConvolutionStartInPIC, 
        localSizeXConvolution, localSizeYConvolution, 
        mPIInfoPIC.localSizeX, mPIInfoPIC.localSizeY
    );
    hipDeviceSynchronize();

    copyFieldsPIC_kernel<FirstMoment><<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(firstMomentOfOneSpecies.data()), 
        thrust::raw_pointer_cast(tmpFirstMoment.data()), 
        indexOfConvolutionStartInPIC, 
        localSizeXConvolution, localSizeYConvolution, 
        mPIInfoPIC.localSizeX, mPIInfoPIC.localSizeY
    );
    hipDeviceSynchronize();

    convolveFields_kernel<FirstMoment><<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(tmpFirstMoment.data()), 
        thrust::raw_pointer_cast(firstMomentOfOneSpecies.data()), 
        indexOfConvolutionStartInPIC, 
        localSizeXConvolution, localSizeYConvolution, 
        mPIInfoPIC.localSizeX, mPIInfoPIC.localSizeY
    );
    hipDeviceSynchronize();

}


void InterfaceNoiseRemover2D::convolveMoments(
    thrust::device_vector<ZerothMoment>& zerothMomentIon, 
    thrust::device_vector<ZerothMoment>& zerothMomentElectron, 
    thrust::device_vector<FirstMoment>& firstMomentIon, 
    thrust::device_vector<FirstMoment>& firstMomentElectron
)
{
    convolveMomentsOfOneSpecies(
        zerothMomentIon, firstMomentIon
    );
    convolveMomentsOfOneSpecies(
        zerothMomentElectron, firstMomentElectron
    );
}



__global__ void copyU_kernel(
    const ConservationParameter* U, 
    ConservationParameter* tmpU, 
    int indexOfConvolutionStartInMHD, 
    int localSizeXConvolution, int localSizeYConvolution, 
    int localSizeXMHD, int localSizeYMHD
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < localSizeXConvolution && j < localSizeYConvolution) {
        int indexForCopy = j + i * localSizeYConvolution;
        int indexMHD = indexOfConvolutionStartInMHD + j + i * localSizeYMHD;

        tmpU[indexForCopy] = U[indexMHD];
    }
}


__global__ void convolveU_kernel(
    const ConservationParameter* tmpU, 
    ConservationParameter* U, 
    int indexOfConvolutionStartInMHD, 
    int localSizeXConvolution, int localSizeYConvolution, 
    int localSizeXMHD, int localSizeYMHD
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (0 < i && i < localSizeXConvolution - 1 && 0 < j && j < localSizeYConvolution - 1) {
        int indexForCopy = j + i * localSizeYConvolution;
        int indexMHD = indexOfConvolutionStartInMHD + j + i * localSizeYMHD;
        
        ConservationParameter convolvedU;

        for (int windowX = -1; windowX <= 1; windowX++) {
            for (int windowY = -1; windowY <= 1; windowY++) {
                int localIndex; 
                localIndex = indexForCopy + windowY + windowX * localSizeYConvolution; 
                convolvedU = convolvedU + tmpU[localIndex];
            }
        }
        convolvedU = 1.0 / 9.0 * convolvedU; 
        
        U[indexMHD] = convolvedU;
    }
}


void InterfaceNoiseRemover2D::convolveU(
    thrust::device_vector<ConservationParameter>& U
)
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((localSizeXConvolution + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (localSizeYConvolution + threadsPerBlock.y - 1) / threadsPerBlock.y);

    copyU_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(U.data()), 
        thrust::raw_pointer_cast(tmpU.data()),
        indexOfConvolutionStartInMHD, 
        localSizeXConvolution, localSizeYConvolution, 
        mPIInfoMHD.localSizeX, mPIInfoMHD.localSizeY
    );
    hipDeviceSynchronize();

    convolveU_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(tmpU.data()), 
        thrust::raw_pointer_cast(U.data()), 
        indexOfConvolutionStartInMHD, 
        localSizeXConvolution, localSizeYConvolution, 
        mPIInfoMHD.localSizeX, mPIInfoMHD.localSizeY
    );
    hipDeviceSynchronize();
}

