#include "hip/hip_runtime.h"
#include "remove_noise.hpp"


InterfaceNoiseRemover2D::InterfaceNoiseRemover2D(
    IdealMHD2DMPI::MPIInfo& mPIInfoMHD, 
    PIC2DMPI::MPIInfo& mPIInfoPIC, 
    int indexOfInterfaceStartInMHD, 
    int indexOfInterfaceStartInPIC, 
    int localSizeXInterface, int localSizeYInterface
)
    : mPIInfoMHD(mPIInfoMHD), 
      mPIInfoPIC(mPIInfoPIC), 

      indexOfInterfaceStartInMHD(indexOfInterfaceStartInMHD), 
      indexOfInterfaceStartInPIC(indexOfInterfaceStartInPIC), 
      localSizeXInterface(localSizeXInterface), 
      localSizeYInterface(localSizeYInterface), 

      tmpB(localSizeXInterface * localSizeYInterface), 
      tmpE(localSizeXInterface * localSizeYInterface), 
      tmpCurrent(localSizeXInterface * localSizeYInterface), 
      tmpZerothMoment(localSizeXInterface * localSizeYInterface), 
      tmpFirstMoment(localSizeXInterface * localSizeYInterface), 
      tmpU(localSizeXInterface * localSizeYInterface)
{

    hipMalloc(&device_mPIInfoMHD, sizeof(IdealMHD2DMPI::MPIInfo));
    hipMemcpy(device_mPIInfoMHD, &mPIInfoMHD, sizeof(IdealMHD2DMPI::MPIInfo), hipMemcpyHostToDevice);
    hipMalloc(&device_mPIInfoPIC, sizeof(PIC2DMPI::MPIInfo));
    hipMemcpy(device_mPIInfoPIC, &mPIInfoPIC, sizeof(PIC2DMPI::MPIInfo), hipMemcpyHostToDevice);
    
}


template <typename FieldType>
__global__ void copyFieldsPIC_kernel(
    const FieldType* field, 
    FieldType* tmpField, 
    int indexOfInterfaceStartInPIC, 
    int localSizeXInterface, int localSizeYInterface, 
    int localSizeXPIC, int localSizeYPIC
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < localSizeXInterface && j < localSizeYInterface) {
        int indexForCopy = j + i * localSizeYInterface;
        int indexPIC = indexOfInterfaceStartInPIC + j + i * localSizeYPIC;

        tmpField[indexForCopy] = field[indexPIC];
    }
}


template <typename FieldType>
__global__ void convolveFields_kernel(
    const FieldType* tmpField, 
    FieldType* field, 
    int indexOfInterfaceStartInPIC, 
    int localSizeXInterface, int localSizeYInterface, 
    int localSizeXPIC, int localSizeYPIC
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (0 < i && i < localSizeXInterface - 1 && 0 < j && j < localSizeYInterface - 1) {
        int indexForCopy = j + i * localSizeYInterface;
        int indexPIC = indexOfInterfaceStartInPIC + j + i * localSizeYPIC;
        
        FieldType convolvedField; 

        convolvedField = 0.5  * (
                       + 0.5  * tmpField[indexForCopy]
                       + 0.25 * (tmpField[indexForCopy + localSizeYInterface] + tmpField[indexForCopy - localSizeYInterface])
                       + 0.5  * tmpField[indexForCopy]
                       + 0.25 * (tmpField[indexForCopy + 1] + tmpField[indexForCopy - 1])
        );
        
        field[indexPIC] = convolvedField;
    }
}


void InterfaceNoiseRemover2D::convolve_magneticField(
    thrust::device_vector<MagneticField>& B
)
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((localSizeXInterface + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (localSizeYInterface + threadsPerBlock.y - 1) / threadsPerBlock.y);

    copyFieldsPIC_kernel<MagneticField><<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(B.data()), 
        thrust::raw_pointer_cast(tmpB.data()), 
        indexOfInterfaceStartInPIC, 
        localSizeXInterface, localSizeYInterface, 
        mPIInfoPIC.localSizeX, mPIInfoPIC.localSizeY
    );
    hipDeviceSynchronize();
    
    convolveFields_kernel<MagneticField><<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(tmpB.data()), 
        thrust::raw_pointer_cast(B.data()), 
        indexOfInterfaceStartInPIC, 
        localSizeXInterface, localSizeYInterface, 
        mPIInfoPIC.localSizeX, mPIInfoPIC.localSizeY
    );
    hipDeviceSynchronize();

}


void InterfaceNoiseRemover2D::convolve_electricField(
    thrust::device_vector<ElectricField>& E
)
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((localSizeXInterface + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (localSizeYInterface + threadsPerBlock.y - 1) / threadsPerBlock.y);

    copyFieldsPIC_kernel<ElectricField><<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(E.data()), 
        thrust::raw_pointer_cast(tmpE.data()), 
        indexOfInterfaceStartInPIC, 
        localSizeXInterface, localSizeYInterface, 
        mPIInfoPIC.localSizeX, mPIInfoPIC.localSizeY
    );
    hipDeviceSynchronize();

    convolveFields_kernel<ElectricField><<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(tmpE.data()), 
        thrust::raw_pointer_cast(E.data()), 
        indexOfInterfaceStartInPIC, 
        localSizeXInterface, localSizeYInterface, 
        mPIInfoPIC.localSizeX, mPIInfoPIC.localSizeY
    );
    hipDeviceSynchronize();

}


void InterfaceNoiseRemover2D::convolve_currentField(
    thrust::device_vector<CurrentField>& current
)
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((localSizeXInterface + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (localSizeYInterface + threadsPerBlock.y - 1) / threadsPerBlock.y);

    copyFieldsPIC_kernel<CurrentField><<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(current.data()), 
        thrust::raw_pointer_cast(tmpCurrent.data()), 
        indexOfInterfaceStartInPIC, 
        localSizeXInterface, localSizeYInterface, 
        mPIInfoPIC.localSizeX, mPIInfoPIC.localSizeY
    );
    hipDeviceSynchronize();
    
    convolveFields_kernel<CurrentField><<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(tmpCurrent.data()), 
        thrust::raw_pointer_cast(current.data()), 
        indexOfInterfaceStartInPIC, 
        localSizeXInterface, localSizeYInterface, 
        mPIInfoPIC.localSizeX, mPIInfoPIC.localSizeY
    );
    hipDeviceSynchronize();

}


void InterfaceNoiseRemover2D::convolveMomentsOfOneSpecies(
    thrust::device_vector<ZerothMoment>& zerothMomentOfOneSpecies, 
    thrust::device_vector<FirstMoment>& firstMomentOfOneSpecies
)
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((localSizeXInterface + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (localSizeYInterface + threadsPerBlock.y - 1) / threadsPerBlock.y);

    copyFieldsPIC_kernel<ZerothMoment><<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(zerothMomentOfOneSpecies.data()), 
        thrust::raw_pointer_cast(tmpZerothMoment.data()), 
        indexOfInterfaceStartInPIC, 
        localSizeXInterface, localSizeYInterface, 
        mPIInfoPIC.localSizeX, mPIInfoPIC.localSizeY
    );
    hipDeviceSynchronize();

    convolveFields_kernel<ZerothMoment><<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(tmpZerothMoment.data()), 
        thrust::raw_pointer_cast(zerothMomentOfOneSpecies.data()), 
        indexOfInterfaceStartInPIC, 
        localSizeXInterface, localSizeYInterface, 
        mPIInfoPIC.localSizeX, mPIInfoPIC.localSizeY
    );
    hipDeviceSynchronize();

    copyFieldsPIC_kernel<FirstMoment><<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(firstMomentOfOneSpecies.data()), 
        thrust::raw_pointer_cast(tmpFirstMoment.data()), 
        indexOfInterfaceStartInPIC, 
        localSizeXInterface, localSizeYInterface, 
        mPIInfoPIC.localSizeX, mPIInfoPIC.localSizeY
    );
    hipDeviceSynchronize();

    convolveFields_kernel<FirstMoment><<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(tmpFirstMoment.data()), 
        thrust::raw_pointer_cast(firstMomentOfOneSpecies.data()), 
        indexOfInterfaceStartInPIC, 
        localSizeXInterface, localSizeYInterface, 
        mPIInfoPIC.localSizeX, mPIInfoPIC.localSizeY
    );
    hipDeviceSynchronize();

}


void InterfaceNoiseRemover2D::convolveMoments(
    thrust::device_vector<ZerothMoment>& zerothMomentIon, 
    thrust::device_vector<ZerothMoment>& zerothMomentElectron, 
    thrust::device_vector<FirstMoment>& firstMomentIon, 
    thrust::device_vector<FirstMoment>& firstMomentElectron
)
{
    convolveMomentsOfOneSpecies(
        zerothMomentIon, firstMomentIon
    );
    convolveMomentsOfOneSpecies(
        zerothMomentElectron, firstMomentElectron
    );
}



__global__ void copyU_kernel(
    const ConservationParameter* U, 
    ConservationParameter* tmpU, 
    int indexOfInterfaceStartInMHD, 
    int localSizeXInterface, int localSizeYInterface, 
    int localSizeXMHD, int localSizeYMHD
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < localSizeXInterface && j < localSizeYInterface) {
        int indexForCopy = j + i * localSizeYInterface;
        int indexMHD = indexOfInterfaceStartInMHD + j + i * localSizeYMHD;

        tmpU[indexForCopy] = U[indexMHD];
    }
}


__global__ void convolveU_kernel(
    const ConservationParameter* tmpU, 
    ConservationParameter* U, 
    int indexOfInterfaceStartInMHD, 
    int localSizeXInterface, int localSizeYInterface, 
    int localSizeXMHD, int localSizeYMHD
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (0 < i && i < localSizeXInterface - 1 && 0 < j && j < localSizeYInterface - 1) {
        int indexForCopy = j + i * localSizeYInterface;
        int indexMHD = indexOfInterfaceStartInMHD + j + i * localSizeYMHD;
        
        ConservationParameter convolvedU;

        convolvedU = 0.5  * (
                   + 0.5  * tmpU[indexForCopy]
                   + 0.25 * (tmpU[indexForCopy + localSizeYInterface] + tmpU[indexForCopy - localSizeYInterface])
                   + 0.5  * tmpU[indexForCopy]
                   + 0.25 * (tmpU[indexForCopy + 1] + tmpU[indexForCopy - 1])
        );
        
        U[indexMHD] = convolvedU;
    }
}


void InterfaceNoiseRemover2D::convolveU(
    thrust::device_vector<ConservationParameter>& U
)
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((localSizeXInterface + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (localSizeYInterface + threadsPerBlock.y - 1) / threadsPerBlock.y);

    copyU_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(U.data()), 
        thrust::raw_pointer_cast(tmpU.data()),
        indexOfInterfaceStartInMHD, 
        localSizeXInterface, localSizeYInterface, 
        mPIInfoMHD.localSizeX, mPIInfoMHD.localSizeY
    );
    hipDeviceSynchronize();

    convolveU_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(tmpU.data()), 
        thrust::raw_pointer_cast(U.data()), 
        indexOfInterfaceStartInMHD, 
        localSizeXInterface, localSizeYInterface, 
        mPIInfoMHD.localSizeX, mPIInfoMHD.localSizeY
    );
    hipDeviceSynchronize();
}

