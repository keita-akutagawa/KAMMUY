#include "hip/hip_runtime.h"
#include "interface.hpp"


__global__ void initializeReloadParticlesSource_kernel(
    Particle* reloadParticlesSourceSpecies, 
    unsigned long long reloadParticlesNumSpecies, 
    int seed
)
{
    unsigned long long i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < reloadParticlesNumSpecies) {
        hiprandState stateX; 
        hiprandState stateY;
        hiprandState stateVx; 
        hiprandState stateVy; 
        hiprandState stateVz;  
        hiprand_init(seed + 0, i, 0, &stateX);
        hiprand_init(seed + 1, i, 0, &stateY);
        hiprand_init(seed + 2, i, 0, &stateVx);
        hiprand_init(seed + 3, i, 0, &stateVy);
        hiprand_init(seed + 4, i, 0, &stateVz);

        reloadParticlesSourceSpecies[i].x  = hiprand_uniform_double(&stateX);
        reloadParticlesSourceSpecies[i].y  = hiprand_uniform_double(&stateY);
        reloadParticlesSourceSpecies[i].z  = 0.0;
        reloadParticlesSourceSpecies[i].vx = hiprand_normal_double(&stateVx);
        reloadParticlesSourceSpecies[i].vy = hiprand_normal_double(&stateVy);
        reloadParticlesSourceSpecies[i].vz = hiprand_normal_double(&stateVz);
        reloadParticlesSourceSpecies[i].gamma = 0.0;
        reloadParticlesSourceSpecies[i].isExist = false;
    }
}

Interface2D::Interface2D(
    IdealMHD2DMPI::MPIInfo& mPIInfoMHD, 
    PIC2DMPI::MPIInfo& mPIInfoPIC, 
    int indexStartMHD, 
    int indexStartPIC, 
    int length, 
    thrust::host_vector<double>& host_interlockingFunctionY, 
    thrust::host_vector<double>& host_interlockingFunctionYHalf, 
    InterfaceNoiseRemover2D& interfaceNoiseRemover2D
)
    : mPIInfoMHD(mPIInfoMHD), 
      mPIInfoPIC(mPIInfoPIC), 

      indexOfInterfaceStartInMHD(indexStartMHD), 
      indexOfInterfaceStartInPIC(indexStartPIC), 
      interfaceLength(length), 
      indexOfInterfaceEndInMHD(indexStartMHD + length), 
      indexOfInterfaceEndInPIC(indexStartPIC + length), 

      interlockingFunctionY    (interfaceLength, 0.0), 
      interlockingFunctionYHalf(interfaceLength, 0.0),

      zerothMomentIon     (mPIInfoPIC.localSizeX * mPIInfoPIC.localSizeY), 
      zerothMomentElectron(mPIInfoPIC.localSizeX * mPIInfoPIC.localSizeY), 
      firstMomentIon      (mPIInfoPIC.localSizeX * mPIInfoPIC.localSizeY), 
      firstMomentElectron (mPIInfoPIC.localSizeX * mPIInfoPIC.localSizeY),

      restartParticlesIndexIon(0), 
      restartParticlesIndexElectron(0), 

      reloadParticlesSourceIon     (Interface2DConst::reloadParticlesTotalNum), 
      reloadParticlesSourceElectron(Interface2DConst::reloadParticlesTotalNum), 

      reloadParticlesDataIon            (mPIInfoPIC.localSizeX * interfaceLength + 1), 
      reloadParticlesDataElectron       (mPIInfoPIC.localSizeX * interfaceLength + 1), 
      host_reloadParticlesDataIon       (mPIInfoPIC.localSizeX * interfaceLength + 1), 
      host_reloadParticlesDataElectron  (mPIInfoPIC.localSizeX * interfaceLength + 1), 

      B_timeAve                   (mPIInfoPIC.localSizeX * mPIInfoPIC.localSizeY), 
      zerothMomentIon_timeAve     (mPIInfoPIC.localSizeX * mPIInfoPIC.localSizeY), 
      zerothMomentElectron_timeAve(mPIInfoPIC.localSizeX * mPIInfoPIC.localSizeY), 
      firstMomentIon_timeAve      (mPIInfoPIC.localSizeX * mPIInfoPIC.localSizeY), 
      firstMomentElectron_timeAve (mPIInfoPIC.localSizeX * mPIInfoPIC.localSizeY), 

      USub (mPIInfoMHD.localSizeX * mPIInfoMHD.localSizeY), 
      UHalf(mPIInfoMHD.localSizeX * mPIInfoMHD.localSizeY), 

      momentCalculater(mPIInfoPIC), 
      interfaceNoiseRemover2D(interfaceNoiseRemover2D)
{

    hipMalloc(&device_mPIInfoMHD, sizeof(IdealMHD2DMPI::MPIInfo));
    hipMemcpy(device_mPIInfoMHD, &mPIInfoMHD, sizeof(IdealMHD2DMPI::MPIInfo), hipMemcpyHostToDevice);
    hipMalloc(&device_mPIInfoPIC, sizeof(PIC2DMPI::MPIInfo));
    hipMemcpy(device_mPIInfoPIC, &mPIInfoPIC, sizeof(PIC2DMPI::MPIInfo), hipMemcpyHostToDevice);

    interlockingFunctionY = host_interlockingFunctionY;
    interlockingFunctionYHalf = host_interlockingFunctionYHalf;

    dim3 threadsPerBlock(256);
    dim3 blocksPerGrid((Interface2DConst::reloadParticlesTotalNum + threadsPerBlock.x - 1) / threadsPerBlock.x);

    initializeReloadParticlesSource_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(reloadParticlesSourceIon.data()),
        Interface2DConst::reloadParticlesTotalNum, 
        100000
    );
    hipDeviceSynchronize();

    initializeReloadParticlesSource_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(reloadParticlesSourceElectron.data()),
        Interface2DConst::reloadParticlesTotalNum, 
        200000
    );
    hipDeviceSynchronize();
}

