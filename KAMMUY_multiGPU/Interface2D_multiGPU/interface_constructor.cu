#include "hip/hip_runtime.h"
#include "interface.hpp"


__global__ void initializeReloadParticlesSource_kernel(
    Particle* reloadParticlesSourceSpecies, 
    unsigned long long reloadParticlesNumSpecies, 
    int seed
)
{
    unsigned long long i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < reloadParticlesNumSpecies) {
        hiprandState stateX; 
        hiprandState stateY;
        hiprandState stateVx; 
        hiprandState stateVy; 
        hiprandState stateVz;  
        hiprand_init(seed + 0, 100 * i, 0, &stateX);
        hiprand_init(seed + 1, 100 * i, 0, &stateY);
        hiprand_init(seed + 2, 100 * i, 0, &stateVx);
        hiprand_init(seed + 3, 100 * i, 0, &stateVy);
        hiprand_init(seed + 4, 100 * i, 0, &stateVz);

        float x, y, z, vx, vy, vz;
        float EPS = 0.001f;
        while (true) {
            x  = hiprand_uniform(&stateX);
            y  = hiprand_uniform(&stateY);
            z  = 0.0f;

            if (EPS < x && x < 1.0f - EPS && EPS < y && y < 1.0f - EPS) break;
        }
        
        vx = hiprand_normal(&stateVx);
        vy = hiprand_normal(&stateVy);
        vz = hiprand_normal(&stateVz);

        reloadParticlesSourceSpecies[i].x  = x;
        reloadParticlesSourceSpecies[i].y  = y;
        reloadParticlesSourceSpecies[i].z  = z;
        reloadParticlesSourceSpecies[i].vx = vx;
        reloadParticlesSourceSpecies[i].vy = vy;
        reloadParticlesSourceSpecies[i].vz = vz;
    }
}

Interface2D::Interface2D(
    IdealMHD2DMPI::MPIInfo& mPIInfoMHD, 
    PIC2DMPI::MPIInfo& mPIInfoPIC, 
    Interface2DMPI::MPIInfo& mPIInfoInterface, 
    int indexOfInterfaceStartMHD, 
    int indexOfInterfaceStartPIC, 
    thrust::host_vector<double>& host_interlockingFunctionY, 
    thrust::host_vector<double>& host_interlockingFunctionYHalf, 
    InterfaceNoiseRemover2D& interfaceNoiseRemover2D
)
    : mPIInfoMHD(mPIInfoMHD), 
      mPIInfoPIC(mPIInfoPIC), 
      mPIInfoInterface(mPIInfoInterface), 

      indexOfInterfaceStartInMHD(indexOfInterfaceStartMHD), 
      indexOfInterfaceStartInPIC(indexOfInterfaceStartPIC),

      localSizeXPIC(mPIInfoPIC.localSizeX), 
      localSizeYPIC(mPIInfoPIC.localSizeY), 
      localSizeXMHD(mPIInfoMHD.localSizeX), 
      localSizeYMHD(mPIInfoMHD.localSizeY), 
      localSizeXInterface(mPIInfoInterface.localSizeX), 
      localSizeYInterface(mPIInfoInterface.localSizeY), 

      interlockingFunctionY    (localSizeYInterface, 0.0), 
      interlockingFunctionYHalf(localSizeYInterface, 0.0),

      zerothMomentIon     (localSizeXPIC * localSizeYPIC), 
      zerothMomentElectron(localSizeXPIC * localSizeYPIC), 
      firstMomentIon      (localSizeXPIC * localSizeYPIC), 
      firstMomentElectron (localSizeXPIC * localSizeYPIC),

      restartParticlesIndexIon(0), 
      restartParticlesIndexElectron(0), 

      reloadParticlesSourceIon     (Interface2DConst::reloadParticlesTotalNum), 
      reloadParticlesSourceElectron(Interface2DConst::reloadParticlesTotalNum), 

      reloadParticlesDataIon     (localSizeXInterface * localSizeYInterface), 
      reloadParticlesDataElectron(localSizeXInterface * localSizeYInterface), 
      
      B_timeAve                   (localSizeXPIC * localSizeYPIC), 
      zerothMomentIon_timeAve     (localSizeXPIC * localSizeYPIC), 
      zerothMomentElectron_timeAve(localSizeXPIC * localSizeYPIC), 
      firstMomentIon_timeAve      (localSizeXPIC * localSizeYPIC), 
      firstMomentElectron_timeAve (localSizeXPIC * localSizeYPIC), 

      USub (localSizeXMHD * localSizeYMHD), 
      UHalf(localSizeXMHD * localSizeYMHD), 

      momentCalculater(mPIInfoPIC), 
      interfaceNoiseRemover2D(interfaceNoiseRemover2D)
{

    hipMalloc(&device_mPIInfoMHD, sizeof(IdealMHD2DMPI::MPIInfo));
    hipMemcpy(device_mPIInfoMHD, &mPIInfoMHD, sizeof(IdealMHD2DMPI::MPIInfo), hipMemcpyHostToDevice);
    hipMalloc(&device_mPIInfoPIC, sizeof(PIC2DMPI::MPIInfo));
    hipMemcpy(device_mPIInfoPIC, &mPIInfoPIC, sizeof(PIC2DMPI::MPIInfo), hipMemcpyHostToDevice);
    hipMalloc(&device_mPIInfoInterface, sizeof(Interface2DMPI::MPIInfo));
    hipMemcpy(device_mPIInfoInterface, &mPIInfoInterface, sizeof(Interface2DMPI::MPIInfo), hipMemcpyHostToDevice);

    interlockingFunctionY = host_interlockingFunctionY;
    interlockingFunctionYHalf = host_interlockingFunctionYHalf;

    dim3 threadsPerBlock(256);
    dim3 blocksPerGrid((Interface2DConst::reloadParticlesTotalNum + threadsPerBlock.x - 1) / threadsPerBlock.x);

    initializeReloadParticlesSource_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(reloadParticlesSourceIon.data()),
        Interface2DConst::reloadParticlesTotalNum, 
        10000000 + 100 * mPIInfoPIC.rank
    );
    hipDeviceSynchronize();

    initializeReloadParticlesSource_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(reloadParticlesSourceElectron.data()),
        Interface2DConst::reloadParticlesTotalNum, 
        20000000 + 100 * mPIInfoPIC.rank
    );
    hipDeviceSynchronize();
}

