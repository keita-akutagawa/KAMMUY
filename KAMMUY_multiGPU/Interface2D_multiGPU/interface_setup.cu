#include "hip/hip_runtime.h"
#include "interface.hpp"


__global__ void calculateSubU_kernel(
    const ConservationParameter* UPast, 
    const ConservationParameter* UNext, 
    ConservationParameter* USub, 
    double mixingRatio, 
    int localSizeXPIC, int localSizeYPIC, 
    int localSizeXMHD, int localSizeYMHD
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < localSizeXMHD && j < localSizeYMHD) {
        int index = j + i * localSizeYMHD;

        USub[index].rho  = mixingRatio * UPast[index].rho  + (1.0 - mixingRatio) * UNext[index].rho;
        USub[index].rhoU = mixingRatio * UPast[index].rhoU + (1.0 - mixingRatio) * UNext[index].rhoU;
        USub[index].rhoV = mixingRatio * UPast[index].rhoV + (1.0 - mixingRatio) * UNext[index].rhoV;
        USub[index].rhoW = mixingRatio * UPast[index].rhoW + (1.0 - mixingRatio) * UNext[index].rhoW;
        USub[index].bX   = mixingRatio * UPast[index].bX   + (1.0 - mixingRatio) * UNext[index].bX;
        USub[index].bY   = mixingRatio * UPast[index].bY   + (1.0 - mixingRatio) * UNext[index].bY;
        USub[index].bZ   = mixingRatio * UPast[index].bZ   + (1.0 - mixingRatio) * UNext[index].bZ;
        USub[index].e    = mixingRatio * UPast[index].e    + (1.0 - mixingRatio) * UNext[index].e;
    }
}

thrust::device_vector<ConservationParameter>& Interface2D::calculateAndGetSubU(
    const thrust::device_vector<ConservationParameter>& UPast, 
    const thrust::device_vector<ConservationParameter>& UNext, 
    double mixingRatio
)
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((mPIInfoMHD.localSizeX + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (mPIInfoMHD.localSizeY + threadsPerBlock.y - 1) / threadsPerBlock.y);

    calculateSubU_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(UPast.data()), 
        thrust::raw_pointer_cast(UNext.data()), 
        thrust::raw_pointer_cast(USub.data()), 
        mixingRatio, 
        mPIInfoPIC.localSizeX, mPIInfoPIC.localSizeY, 
        mPIInfoMHD.localSizeX, mPIInfoMHD.localSizeY
    );
    hipDeviceSynchronize();

    return USub;
}


void Interface2D::resetTimeAveParameters()
{
    thrust::fill(
        B_timeAve.begin(), 
        B_timeAve.end(), 
        MagneticField()
    );

    thrust::fill(
        zerothMomentIon_timeAve.begin(), 
        zerothMomentIon_timeAve.end(), 
        ZerothMoment()
    );
    thrust::fill(
        zerothMomentElectron_timeAve.begin(), 
        zerothMomentElectron_timeAve.end(), 
        ZerothMoment()
    );

    thrust::fill(
        firstMomentIon_timeAve.begin(), 
        firstMomentIon_timeAve.end(), 
        FirstMoment()
    );
    thrust::fill(
        firstMomentElectron_timeAve.begin(), 
        firstMomentElectron_timeAve.end(), 
        FirstMoment()
    );
}


void Interface2D::setMoments(
    const thrust::device_vector<Particle>& particlesIon, 
    const thrust::device_vector<Particle>& particlesElectron
)
{
    momentCalculater.calculateZerothMomentOfOneSpecies(
        zerothMomentIon, particlesIon, mPIInfoPIC.existNumIonPerProcs
    );
    momentCalculater.calculateZerothMomentOfOneSpecies(
        zerothMomentElectron, particlesElectron, mPIInfoPIC.existNumElectronPerProcs
    );

    momentCalculater.calculateFirstMomentOfOneSpecies(
        firstMomentIon, particlesIon, mPIInfoPIC.existNumIonPerProcs
    );
    momentCalculater.calculateFirstMomentOfOneSpecies(
        firstMomentElectron, particlesElectron, mPIInfoPIC.existNumElectronPerProcs
    );

    PIC2DMPI::sendrecv_field_x(zerothMomentIon, mPIInfoPIC, mPIInfoPIC.mpi_zerothMomentType);
    PIC2DMPI::sendrecv_field_x(zerothMomentElectron, mPIInfoPIC, mPIInfoPIC.mpi_zerothMomentType);
    PIC2DMPI::sendrecv_field_x(firstMomentIon, mPIInfoPIC, mPIInfoPIC.mpi_firstMomentType);
    PIC2DMPI::sendrecv_field_x(firstMomentElectron, mPIInfoPIC, mPIInfoPIC.mpi_firstMomentType);
}


void Interface2D::sumUpTimeAveParameters(
    const thrust::device_vector<MagneticField>& B, 
    const thrust::device_vector<Particle>& particlesIon, 
    const thrust::device_vector<Particle>& particlesElectron
)
{
    thrust::transform(
        B_timeAve.begin(), B_timeAve.end(), B.begin(), 
        B_timeAve.begin(), thrust::plus<MagneticField>()
    );
    
    setMoments(particlesIon, particlesElectron);

    thrust::transform(
        zerothMomentIon_timeAve.begin(), zerothMomentIon_timeAve.end(), zerothMomentIon.begin(), 
        zerothMomentIon_timeAve.begin(), thrust::plus<ZerothMoment>()
    );
    thrust::transform(
        zerothMomentElectron_timeAve.begin(), zerothMomentElectron_timeAve.end(), zerothMomentElectron.begin(), 
        zerothMomentElectron_timeAve.begin(), thrust::plus<ZerothMoment>()
    );
    thrust::transform(
        firstMomentIon_timeAve.begin(), firstMomentIon_timeAve.end(), firstMomentIon.begin(), 
        firstMomentIon_timeAve.begin(), thrust::plus<FirstMoment>()
    );
    thrust::transform(
        firstMomentElectron_timeAve.begin(), firstMomentElectron_timeAve.end(), firstMomentElectron.begin(), 
        firstMomentElectron_timeAve.begin(), thrust::plus<FirstMoment>()
    );
}


__global__ void calculateTimeAveParameters_kernel(
    MagneticField* B_timeAve, 
    ZerothMoment* zerothMomentIon_timeAve, 
    ZerothMoment* zerothMomentElectron_timeAve, 
    FirstMoment* firstMomentIon_timeAve, 
    FirstMoment* firstMomentElectron_timeAve, 
    int substeps, 
    int localSizeXPIC, int localSizeYPIC, 
    int localSizeXMHD, int localSizeYMHD
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < localSizeXPIC && j < localSizeYPIC) {
        int index = j + i * localSizeYPIC;

        B_timeAve[index].bX                   /= static_cast<double>(substeps);
        B_timeAve[index].bY                   /= static_cast<double>(substeps);
        B_timeAve[index].bZ                   /= static_cast<double>(substeps);
        zerothMomentIon_timeAve[index].n      /= static_cast<double>(substeps);
        zerothMomentElectron_timeAve[index].n /= static_cast<double>(substeps);
        firstMomentIon_timeAve[index].x       /= static_cast<double>(substeps);
        firstMomentIon_timeAve[index].y       /= static_cast<double>(substeps);
        firstMomentIon_timeAve[index].z       /= static_cast<double>(substeps);
        firstMomentElectron_timeAve[index].x  /= static_cast<double>(substeps);
        firstMomentElectron_timeAve[index].y  /= static_cast<double>(substeps);
        firstMomentElectron_timeAve[index].z  /= static_cast<double>(substeps);
    }
}

void Interface2D::calculateTimeAveParameters(int substeps)
{
    interfaceNoiseRemover2D.convolve_magneticField(B_timeAve, isLower, isUpper);
    interfaceNoiseRemover2D.convolveMoments(
        zerothMomentIon_timeAve, zerothMomentElectron_timeAve, 
        firstMomentIon_timeAve, firstMomentElectron_timeAve, 
        isLower, isUpper
    );

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((mPIInfoPIC.localSizeX + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (mPIInfoPIC.localSizeY + threadsPerBlock.y - 1) / threadsPerBlock.y);

    calculateTimeAveParameters_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(B_timeAve.data()), 
        thrust::raw_pointer_cast(zerothMomentIon_timeAve.data()), 
        thrust::raw_pointer_cast(zerothMomentElectron_timeAve.data()), 
        thrust::raw_pointer_cast(firstMomentIon_timeAve.data()), 
        thrust::raw_pointer_cast(firstMomentElectron_timeAve.data()), 
        substeps, 
        mPIInfoPIC.localSizeX, mPIInfoPIC.localSizeY, 
        mPIInfoMHD.localSizeX, mPIInfoMHD.localSizeY
    );
    hipDeviceSynchronize();
}


thrust::device_vector<ConservationParameter>& Interface2D::getUHalfRef()
{
    return UHalf;
}


