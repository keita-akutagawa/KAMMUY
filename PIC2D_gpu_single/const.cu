#include "const.hpp"


using namespace PIC2DConst;

void initializeDeviceConstants()
{
    hipMemcpyToSymbol(HIP_SYMBOL(device_c_PIC), &c_PIC, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_epsilon0_PIC), &epsilon0_PIC, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_mu0_PIC), &mu0_PIC, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_dOfLangdonMarderTypeCorrection_PIC), &dOfLangdonMarderTypeCorrection_PIC, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_EPS_PIC), &EPS_PIC, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(device_nx_PIC), &nx_PIC, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(device_dx_PIC), &dx_PIC, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_xmin_PIC), &xmin_PIC, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_xmax_PIC), &xmax_PIC, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(device_ny_PIC), &ny_PIC, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(device_dy_PIC), &dy_PIC, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_ymin_PIC), &ymin_PIC, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_ymax_PIC), &ymax_PIC, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(device_dt_PIC), &dt_PIC, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(device_numberDensityIon_PIC), &numberDensityIon_PIC, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(device_numberDensityElectron_PIC), &numberDensityElectron_PIC, sizeof(int));

    hipMemcpyToSymbol(HIP_SYMBOL(device_totalNumIon_PIC), &totalNumIon_PIC, sizeof(unsigned long long));
    hipMemcpyToSymbol(HIP_SYMBOL(device_totalNumElectron_PIC), &totalNumElectron_PIC, sizeof(unsigned long long));
    hipMemcpyToSymbol(HIP_SYMBOL(device_totalNumParticles_PIC), &totalNumParticles_PIC, sizeof(unsigned long long));
    hipMemcpyToSymbol(HIP_SYMBOL(device_existNumIon_PIC), &existNumIon_PIC, sizeof(unsigned long long));
    hipMemcpyToSymbol(HIP_SYMBOL(device_existNumElectron_PIC), &existNumElectron_PIC, sizeof(unsigned long long));

    hipMemcpyToSymbol(HIP_SYMBOL(device_B0_PIC), &B0_PIC, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(device_mRatio_PIC), &mRatio_PIC, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_mIon_PIC), &mIon_PIC, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_mElectron_PIC), &mElectron_PIC, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(device_tRatio_PIC), &tRatio_PIC, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_tIon_PIC), &tIon_PIC, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_tElectron_PIC), &tElectron_PIC, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(device_qRatio_PIC), &qRatio_PIC, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_qIon_PIC), &qIon_PIC, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_qElectron_PIC), &qElectron_PIC, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(device_omegaPe_PIC), &omegaPe_PIC, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_omegaPi_PIC), &omegaPi_PIC, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_omegaCe_PIC), &omegaCe_PIC, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_omegaCi_PIC), &omegaCi_PIC, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(device_debyeLength_PIC), &debyeLength_PIC, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_ionInertialLength_PIC), &ionInertialLength_PIC, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(device_vThIon_PIC), &vThIon_PIC, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_vThElectron_PIC), &vThElectron_PIC, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_bulkVxIon_PIC), &bulkVxIon_PIC, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_bulkVyIon_PIC), &bulkVyIon_PIC, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_bulkVzIon_PIC), &bulkVzIon_PIC, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_bulkVxElectron_PIC), &bulkVxElectron_PIC, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_bulkVyElectron_PIC), &bulkVyElectron_PIC, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_bulkVzElectron_PIC), &bulkVzElectron_PIC, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(device_totalStep_PIC), &totalStep_PIC, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(device_totalTime_PIC), &totalTime_PIC, sizeof(float));
}

