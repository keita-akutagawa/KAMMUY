#include "hip/hip_runtime.h"
#include "main_alfven_const.hpp"
#include <string>
#include <fstream>
#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>


// 別にinitializeUを作ることにする。
void IdealMHD2D::initializeU()
{
}


__global__ void initializeU_Lower_kernel(
    ConservationParameter* U
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < device_nx_MHD && j < device_ny_MHD) {
        double rho, u, v, w, bX, bY, bZ, e, p;
        
        rho = device_rho0_MHD;
        u   = device_waveAmp * device_VA * cos(device_waveNumber * j * IdealMHD2DConst::device_dy_MHD);
        v   = 0.0;
        w   = -device_waveAmp * device_VA * sin(device_waveNumber * j * IdealMHD2DConst::device_dy_MHD);
        bX  = -device_waveAmp * device_b0_MHD * cos(device_waveNumber * j * IdealMHD2DConst::device_dy_MHD);
        bY  = device_b0_MHD;
        bZ  = device_waveAmp * device_b0_MHD * sin(device_waveNumber * j * IdealMHD2DConst::device_dy_MHD);
        p   = device_p0_MHD;
        e   = p / (IdealMHD2DConst::device_gamma_MHD - 1.0)
            + 0.5 * rho * (u * u + v * v + w * w)
            + 0.5 * (bX * bX + bY * bY + bZ * bZ);

        U[j + i * device_ny_MHD].rho  = rho;
        U[j + i * device_ny_MHD].rhoU = rho * u;
        U[j + i * device_ny_MHD].rhoV = rho * v;
        U[j + i * device_ny_MHD].rhoW = rho * w;
        U[j + i * device_ny_MHD].bX   = bX;
        U[j + i * device_ny_MHD].bY   = bY;
        U[j + i * device_ny_MHD].bZ   = bZ;
        U[j + i * device_ny_MHD].e    = e;
    }
}


__global__ void initializeU_Upper_kernel(
    ConservationParameter* U
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < device_nx_MHD && j < device_ny_MHD) {
        double rho, u, v, w, bX, bY, bZ, e, p;
        double y = j * PIC2DConst::device_dy_PIC + 9500 * IdealMHD2DConst::device_dy_MHD + 950 * PIC2DConst::device_dy_PIC;
        
        rho = device_rho0_MHD;
        u   = device_waveAmp * device_VA * cos(device_waveNumber * y);
        v   = 0.0;
        w   = -device_waveAmp * device_VA * sin(device_waveNumber * y);
        bX  = -device_waveAmp * device_b0_MHD * cos(device_waveNumber * y);
        bY  = device_b0_MHD;
        bZ  = device_waveAmp * device_b0_MHD * sin(device_waveNumber * y);
        p   = device_p0_MHD;
        e   = p / (IdealMHD2DConst::device_gamma_MHD - 1.0)
            + 0.5 * rho * (u * u + v * v + w * w)
            + 0.5 * (bX * bX + bY * bY + bZ * bZ);

        U[j + i * device_ny_MHD].rho  = rho;
        U[j + i * device_ny_MHD].rhoU = rho * u;
        U[j + i * device_ny_MHD].rhoV = rho * v;
        U[j + i * device_ny_MHD].rhoW = rho * w;
        U[j + i * device_ny_MHD].bX   = bX;
        U[j + i * device_ny_MHD].bY   = bY;
        U[j + i * device_ny_MHD].bZ   = bZ;
        U[j + i * device_ny_MHD].e    = e;
    }
}


void initializeU(
    thrust::device_vector<ConservationParameter>& U_Lower, 
    thrust::device_vector<ConservationParameter>& U_Upper
)
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((nx_MHD + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (ny_MHD + threadsPerBlock.y - 1) / threadsPerBlock.y);

    initializeU_Lower_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(U_Lower.data())
    );

    hipDeviceSynchronize();


    initializeU_Upper_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(U_Upper.data())
    );

    hipDeviceSynchronize();
}


__global__ void initializePICField_kernel(
    ElectricField* E, MagneticField* B
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < device_nx_PIC && j < device_ny_PIC) {
        double u, v, w, bX, bY, bZ, eX, eY, eZ;
        double y = j * PIC2DConst::device_dy_PIC + 9500 * IdealMHD2DConst::device_dy_MHD;

        bX = -device_waveAmp * device_b0_PIC * cos(device_waveNumber * y);
        bY = device_b0_PIC; 
        bZ = device_waveAmp * device_b0_PIC * sin(device_waveNumber * y);
        u = device_waveAmp * device_VA * cos(device_waveNumber * y);
        v = 0.0;
        w = -device_waveAmp * device_VA * sin(device_waveNumber * y);
        eX = -(v * bZ - w * bY);
        eY = -(w * bX - u * bZ);
        eZ = -(u * bY - v * bX);

        E[j + i * device_ny_PIC].eX = eX;
        E[j + i * device_ny_PIC].eY = eY;
        E[j + i * device_ny_PIC].eZ = eZ;
        B[j + i * device_ny_PIC].bX = bX;
        B[j + i * device_ny_PIC].bY = bY; 
        B[j + i * device_ny_PIC].bZ = bZ;
    }
}

void PIC2D::initialize()
{
    initializeParticle.uniformForPositionX(
        0, existNumIon_PIC, 0, particlesIon
    );
    initializeParticle.uniformForPositionX(
        0, existNumElectron_PIC, 100, particlesElectron
    );
    initializeParticle.uniformForPositionY(
        0, existNumIon_PIC, 200, particlesIon
    );
    initializeParticle.uniformForPositionY(
        0, existNumElectron_PIC, 300, particlesElectron
    );

    for (int j = 0; j < PIC2DConst::ny_PIC; j++) {
        double u, v, w;
        u = waveAmp * VA * cos(waveNumber * (j * PIC2DConst::dy_PIC + 950 * IdealMHD2DConst::dy_MHD));
        v = 0.0;
        w = -waveAmp * VA * sin(waveNumber * (j * PIC2DConst::dy_PIC + 950 * IdealMHD2DConst::dy_MHD));

        initializeParticle.maxwellDistributionForVelocity(
            u, v, w, vThIon_PIC, vThIon_PIC, vThIon_PIC, 
            j * PIC2DConst::nx_PIC * numberDensityIon_PIC, (j + 1) * PIC2DConst::nx_PIC * numberDensityIon_PIC, j * 100 + 400, particlesIon
        );
        initializeParticle.maxwellDistributionForVelocity(
            u, v, w, vThElectron_PIC, vThElectron_PIC, vThElectron_PIC, 
            j * PIC2DConst::nx_PIC * numberDensityElectron_PIC, (j + 1) * PIC2DConst::nx_PIC * numberDensityElectron_PIC, j * 100 + 400 + totalNumIon_PIC, particlesElectron
        );
    }
    

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((nx_PIC + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (ny_PIC + threadsPerBlock.y - 1) / threadsPerBlock.y);

    initializePICField_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(E.data()), thrust::raw_pointer_cast(B.data())
    );

    hipDeviceSynchronize();
}



int main()
{
    hipMemcpyToSymbol(HIP_SYMBOL(device_VA), &VA, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_waveAmp), &waveAmp, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_waveLength), &waveLength, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_waveNumber), &waveNumber, sizeof(double));

    initializeDeviceConstants_PIC();
    initializeDeviceConstants_MHD();
    initializeDeviceConstants_Interface();
    for (int i = 0; i < interfaceLength; i++) {
        host_interlockingFunctionY_Lower[i] = max(
            0.5 * (1.0 + cos(Interface2DConst::PI * (i - 0.0) / (interfaceLength - 0.0))), 
            1e-20
        );
        host_interlockingFunctionY_Upper[i] = max(
            0.5 * (1.0 - cos(Interface2DConst::PI * (i - 0.0) / (interfaceLength - 0.0))), 
            Interface2DConst::EPS
        );
    }
    for (int i = 0; i < interfaceLength - 1; i++) {
        host_interlockingFunctionYHalf_Lower[i] = max(
            0.5 * (1.0 + cos(Interface2DConst::PI * (i + 0.5 - 0.0) / (interfaceLength - 0.0))), 
            1e-20
        );
        host_interlockingFunctionYHalf_Upper[i] = max(
            0.5 * (1.0 - cos(Interface2DConst::PI * (i + 0.5 - 0.0) / (interfaceLength - 0.0))), 
            Interface2DConst::EPS
        );
    }


    IdealMHD2D idealMHD2D_Lower;
    IdealMHD2D idealMHD2D_Upper;
    PIC2D pIC2D;
    InterfaceNoiseRemover2D interfaceNoiseRemover2D_Lower(
        indexOfInterfaceStartInMHD_Lower, 
        indexOfInterfaceStartInPIC_Lower, 
        interfaceLength, 
        windowSizeForConvolution
    );
    InterfaceNoiseRemover2D interfaceNoiseRemover2D_Upper(
        indexOfInterfaceStartInMHD_Upper, 
        indexOfInterfaceStartInPIC_Upper, 
        interfaceLength, 
        windowSizeForConvolution
    );
    Interface2D interface2D_Lower(
        indexOfInterfaceStartInMHD_Lower, 
        indexOfInterfaceStartInPIC_Lower, 
        interfaceLength, 
        host_interlockingFunctionY_Lower, 
        host_interlockingFunctionYHalf_Lower, 
        interfaceNoiseRemover2D_Lower
    );
    Interface2D interface2D_Upper(
        indexOfInterfaceStartInMHD_Upper, 
        indexOfInterfaceStartInPIC_Upper, 
        interfaceLength, 
        host_interlockingFunctionY_Upper, 
        host_interlockingFunctionYHalf_Upper, 
        interfaceNoiseRemover2D_Upper
    );
    BoundaryPIC boundaryPIC;
    BoundaryMHD boundaryMHD;
    

    size_t free_mem = 0;
    size_t total_mem = 0;
    hipError_t status = hipMemGetInfo(&free_mem, &total_mem);

    std::cout << "Free memory: " << free_mem / (1024 * 1024) << " MB" << std::endl;
    std::cout << "Total memory: " << total_mem / (1024 * 1024) << " MB" << std::endl;

    std::cout << "total number of particles is " 
              << PIC2DConst::totalNumIon_PIC + PIC2DConst::totalNumElectron_PIC << std::endl;



    thrust::device_vector<ConservationParameter>& U_Lower = idealMHD2D_Lower.getURef();
    thrust::device_vector<ConservationParameter>& U_Upper = idealMHD2D_Upper.getURef();
    initializeU(U_Lower, U_Upper);
    pIC2D.initialize();

    const int substeps = int(round(sqrt(PIC2DConst::mRatio_PIC)));
    for (int step = 0; step < IdealMHD2DConst::totalStep_MHD + 1; step++) {
        if (step % recordStep == 0) {
            std::cout << std::to_string(step) << " step done : total time is "
                      << std::setprecision(4) << step * substeps * PIC2DConst::dt_PIC * PIC2DConst::omegaPe_PIC
                      << " [omega_pe * t]"
                      << std::endl;
            logfile << std::setprecision(6) << PIC2DConst::totalTime_PIC << std::endl;
            pIC2D.saveFields(
                directoryname, filenameWithoutStep, step
            );
            pIC2D.saveZerothMoments(
                directoryname, filenameWithoutStep, step
            );
            pIC2D.saveFirstMoments(
                directoryname, filenameWithoutStep, step
            );
            idealMHD2D_Lower.save(
                directoryname, filenameWithoutStep + "_lower", step
            );
            idealMHD2D_Upper.save(
                directoryname, filenameWithoutStep + "_upper", step
            );
        }
        if (isParticleRecord && step % particleRecordStep == 0) {
            pIC2D.saveParticle(
                directoryname, filenameWithoutStep, step
            );
        }

        
        idealMHD2D_Lower.calculateDt();
        double dt_Lower_MHD = IdealMHD2DConst::dt_MHD;
        idealMHD2D_Upper.calculateDt();
        double dt_Upper_MHD = IdealMHD2DConst::dt_MHD;
        double dtCommon = min(min(dt_Lower_MHD / substeps, dt_Upper_MHD / substeps), min(0.7 * PIC2DConst::c_PIC, 0.1 * 1.0 / PIC2DConst::omegaPe_PIC));
        PIC2DConst::dt_PIC = dtCommon;
        IdealMHD2DConst::dt_MHD = substeps * dtCommon;

        idealMHD2D_Lower.setPastU();
        idealMHD2D_Upper.setPastU();
        thrust::device_vector<ConservationParameter>& UPast_Lower = idealMHD2D_Lower.getUPastRef();
        thrust::device_vector<ConservationParameter>& UPast_Upper = idealMHD2D_Upper.getUPastRef();
        idealMHD2D_Lower.oneStepRK2_predictor();
        idealMHD2D_Upper.oneStepRK2_predictor();
        thrust::device_vector<ConservationParameter>& UNext_Lower = idealMHD2D_Lower.getURef();
        thrust::device_vector<ConservationParameter>& UNext_Upper = idealMHD2D_Upper.getURef();


        interface2D_Lower.resetTimeAveParameters();
        interface2D_Upper.resetTimeAveParameters();
        for (int substep = 1; substep <= substeps; substep++) {
            pIC2D.oneStepPeriodicXFreeY();

            thrust::device_vector<MagneticField>& B = pIC2D.getBRef();
            thrust::device_vector<ElectricField>& E = pIC2D.getERef();
            thrust::device_vector<CurrentField>& current = pIC2D.getCurrentRef();
            thrust::device_vector<Particle>& particlesIon = pIC2D.getParticlesIonRef();
            thrust::device_vector<Particle>& particlesElectron = pIC2D.getParticlesElectronRef();

            double mixingRatio = (substeps - substep) / substeps;
            thrust::device_vector<ConservationParameter>& USub_Lower = interface2D_Lower.calculateAndGetSubU(UPast_Lower, UNext_Lower, mixingRatio);
            thrust::device_vector<ConservationParameter>& USub_Upper = interface2D_Upper.calculateAndGetSubU(UPast_Upper, UNext_Upper, mixingRatio);
            
            interface2D_Lower.sendMHDtoPIC_magneticField_yDirection(USub_Lower, B);
            interface2D_Lower.sendMHDtoPIC_electricField_yDirection(USub_Lower, E);
            interface2D_Lower.sendMHDtoPIC_currentField_yDirection(USub_Lower, current);
            interface2D_Lower.sendMHDtoPIC_particle(USub_Lower, particlesIon, particlesElectron, step * substeps + substep);
            interface2D_Upper.sendMHDtoPIC_magneticField_yDirection(USub_Upper, B);
            interface2D_Upper.sendMHDtoPIC_electricField_yDirection(USub_Upper, E);
            interface2D_Upper.sendMHDtoPIC_currentField_yDirection(USub_Upper, current);
            interface2D_Upper.sendMHDtoPIC_particle(USub_Upper, particlesIon, particlesElectron, step * substeps + substep);

            interfaceNoiseRemover2D_Lower.convolveFields(B, E, current);
            interfaceNoiseRemover2D_Upper.convolveFields(B, E, current);

            boundaryPIC.freeBoundaryBY(B);
            boundaryPIC.freeBoundaryEY(E);
            boundaryPIC.freeBoundaryCurrentY(current); 
            boundaryPIC.periodicBoundaryParticleX(particlesIon, particlesElectron);
            boundaryPIC.openBoundaryParticleY(particlesIon, particlesElectron);

            interface2D_Lower.sumUpTimeAveParameters(B, particlesIon, particlesElectron);
            interface2D_Upper.sumUpTimeAveParameters(B, particlesIon, particlesElectron);
        }

        interface2D_Lower.calculateTimeAveParameters(substeps);
        interface2D_Upper.calculateTimeAveParameters(substeps);


        interface2D_Lower.sendPICtoMHD(UPast_Lower, UNext_Lower);
        interface2D_Upper.sendPICtoMHD(UPast_Upper, UNext_Upper);
        thrust::device_vector<ConservationParameter>& UHalf_Lower = interface2D_Lower.getUHalfRef();
        thrust::device_vector<ConservationParameter>& UHalf_Upper = interface2D_Upper.getUHalfRef();
        boundaryMHD.periodicBoundaryX2nd(UHalf_Lower);
        boundaryMHD.symmetricBoundaryY2nd(UHalf_Lower);
        boundaryMHD.periodicBoundaryX2nd(UHalf_Upper);
        boundaryMHD.symmetricBoundaryY2nd(UHalf_Upper);

        idealMHD2D_Lower.oneStepRK2_corrector(UHalf_Lower);
        idealMHD2D_Upper.oneStepRK2_corrector(UHalf_Upper);
        U_Lower = idealMHD2D_Lower.getURef();
        U_Upper = idealMHD2D_Upper.getURef();
        interfaceNoiseRemover2D_Lower.convolveU(U_Lower);
        interfaceNoiseRemover2D_Upper.convolveU(U_Upper);
        boundaryMHD.periodicBoundaryX2nd(U_Lower);
        boundaryMHD.symmetricBoundaryY2nd(U_Lower);
        boundaryMHD.periodicBoundaryX2nd(U_Upper);
        boundaryMHD.symmetricBoundaryY2nd(U_Upper);

        if (idealMHD2D_Lower.checkCalculationIsCrashed() || idealMHD2D_Upper.checkCalculationIsCrashed()) {
            std::cout << "Calculation stopped! : " << step << " steps" << std::endl;
            return 0;
        }

        hipError_t error = hipGetLastError();
        if (error != hipSuccess) {
            printf("CUDA Error: %s\n", hipGetErrorString(error));
            exit(-1);
        }

        IdealMHD2DConst::totalTime_MHD += IdealMHD2DConst::dt_MHD;
    }

    return 0;
}


