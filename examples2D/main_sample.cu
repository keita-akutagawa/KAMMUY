#include "../IdealMHD2D_gpu/IdealMHD2D.hpp"
#include "../IdealMHD2D_gpu/IdealMHD2D.cu"
#include "../PIC2D_gpu_single/PIC2D.hpp"
#include "../PIC2D_gpu_single/PIC2D.cu"
#include "../Interface2D/interface.hpp"
#include "../Interface2D/interface.cu"
#include "main_sample_const.cu"
#include <string>
#include <fstream>
#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>
#include "main_sample_const.cu"



void initialize()
{

}



int main()
{
    PIC2DConst::initializeDeviceConstants();
    IdealMHD2DConst::initializeDeviceConstants();

    size_t free_mem = 0;
    size_t total_mem = 0;
    hipError_t status = hipMemGetInfo(&free_mem, &total_mem);

    std::cout << "Free memory: " << free_mem / (1024 * 1024) << " MB" << std::endl;
    std::cout << "Total memory: " << total_mem / (1024 * 1024) << " MB" << std::endl;


    for (int step = 0; step < IdealMHD2DConst::totalStep; step++) {
        
    }

    return 0;
}


