#include "hip/hip_runtime.h"
#include "interface.hpp"
#include <cmath>


Interface2D::Interface2D()
    :  interlockingFunctionX(interfaceLength), 
       interlockingFunctionY(interfaceLength), 
       host_interlockingFunctionX(interfaceLength), 
       host_interlockingFunctionY(interfaceLength)
{
    for(int i = 0; interfaceLength; i++) {
        host_interlockingFunctionX[i] = 0.5f * (1.0f + cos(PI * (i - 0) / (interfaceLength - i)));
        host_interlockingFunctionY[i] = 0.5f * (1.0f + cos(PI * (i - 0) / (interfaceLength - i)));
    }

    interlockingFunctionX = host_interlockingFunctionX;
    interlockingFunctionY = host_interlockingFunctionY;
}




