#include "const.hpp"


using namespace IdealMHD2DConst;

void initializeDeviceConstants_MHD() {
    hipMemcpyToSymbol(HIP_SYMBOL(device_EPS_MHD), &EPS_MHD, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_PI_MHD), &PI_MHD, sizeof(double));

    hipMemcpyToSymbol(HIP_SYMBOL(device_b0_MHD), &b0_MHD, sizeof(double));

    hipMemcpyToSymbol(HIP_SYMBOL(device_rho0_MHD), &rho0_MHD, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_u0_MHD), &u0_MHD, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_v0_MHD), &v0_MHD, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_w0_MHD), &w0_MHD, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_bX0_MHD), &bX0_MHD, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_bY0_MHD), &bY0_MHD, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_bZ0_MHD), &bZ0_MHD, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_p0_MHD), &p0_MHD, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_e0_MHD), &e0_MHD, sizeof(double));

    hipMemcpyToSymbol(HIP_SYMBOL(device_dx_MHD), &dx_MHD, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_xmin_MHD), &xmin_MHD, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_xmax_MHD), &xmax_MHD, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_nx_MHD), &nx_MHD, sizeof(int));

    hipMemcpyToSymbol(HIP_SYMBOL(device_dy_MHD), &dy_MHD, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_ymin_MHD), &ymin_MHD, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_ymax_MHD), &ymax_MHD, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_ny_MHD), &ny_MHD, sizeof(int));

    hipMemcpyToSymbol(HIP_SYMBOL(device_CFL_MHD), &CFL_MHD, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_gamma_MHD), &gamma_MHD, sizeof(double));

    hipMemcpyToSymbol(HIP_SYMBOL(device_dt_MHD), &dt_MHD, sizeof(double));
}
